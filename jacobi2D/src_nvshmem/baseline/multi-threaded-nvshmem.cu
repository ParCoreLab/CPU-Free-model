#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
// Adapted from
// https://github.com/NVIDIA/multi-gpu-programming-models/blob/master/nvshmem/jacobi.cu
#include <cmath>
#include <cstdio>
#include <cstdlib>

#include "../../include_nvshmem/baseline/multi-threaded-nvshmem.cuh"
#include <nvshmem.h>
#include <nvshmemx.h>
namespace BaselineMultiThreadedNvshmem
{
    template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
    __global__ void jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                                  const int iy_start, const int iy_end, const int nx,
                                  const int top_pe, const int top_iy, const int bottom_pe, const int bottom_iy)
    {

        int iy = blockIdx.y * blockDim.y + threadIdx.y + 1;
        int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;

        if (iy < iy_end && ix < (nx - 1))
        {
            const real new_val = (real(1) / real(4)) *
                                 (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                  a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);

            a_new[iy * nx + ix] = new_val;
            if (iy_start == iy)
            {
                nvshmem_float_p(a_new + top_iy * nx + ix, new_val, top_pe);
            }
            if ((iy_end - 1) == iy)
            {
                nvshmem_float_p(a_new + bottom_iy * nx + ix, new_val, bottom_pe);
            }
        }
    }

} // namespace BaselineMultiThreadedNvshmem

int BaselineMultiThreadedNvshmem::init(int argc, char *argv[])
{
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare");

    real *a;
    real *a_new;

    real *a_ref_h;
    real *a_h;

    int rank = 0, size = 1;
    MPI_CALL(MPI_Init(&argc, &argv));
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &size));

    int num_devices;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    int local_rank = -1;
    int local_size = 1;
    {
        MPI_Comm local_comm;
        MPI_CALL(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
                                     &local_comm));

        MPI_CALL(MPI_Comm_rank(local_comm, &local_rank));
        MPI_CALL(MPI_Comm_size(local_comm, &local_size));

        MPI_CALL(MPI_Comm_free(&local_comm));
    }
    if (1 < num_devices && num_devices < local_size)
    {
        fprintf(stderr, "ERROR Number of visible devices (%d) is less than number of ranks on the node (%d)!\n", num_devices, local_size);
        MPI_CALL(MPI_Finalize());
        return 1;
    }
    if (1 == num_devices)
    {
        // Only 1 device visible, assuming GPU affinity is handled via CUDA_VISIBLE_DEVICES
        CUDA_RT_CALL(hipSetDevice(0));
    }
    else
    {
        CUDA_RT_CALL(hipSetDevice(local_rank));
    }
    CUDA_RT_CALL(hipFree(0));

    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr;

    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;
    // Set symmetric heap size for nvshmem based on problem size
    // Its default value in nvshmem is 1 GB which is not sufficient
    // for large mesh sizes
    long long unsigned int mesh_size_per_rank = nx * (((ny - 2) + size - 1) / size + 2);
    long long unsigned int required_symmetric_heap_size =
        2 * mesh_size_per_rank * sizeof(real) *
        1.1; // Factor 2 is because 2 arrays are allocated - a and a_new
             // 1.1 factor is just for alignment or other usage

    char *value = getenv("NVSHMEM_SYMMETRIC_SIZE");
    if (value)
    { /* env variable is set */
        long long unsigned int size_env = parse_nvshmem_symmetric_size(value);
        if (size_env < required_symmetric_heap_size)
        {
            fprintf(stderr, "ERROR: Minimum NVSHMEM_SYMMETRIC_SIZE = %lluB, Current NVSHMEM_SYMMETRIC_SIZE = %s\n", required_symmetric_heap_size, value);
            MPI_CALL(MPI_Finalize());
            return -1;
        }
    }
    else
    {
        char symmetric_heap_size_str[100];
        sprintf(symmetric_heap_size_str, "%llu", required_symmetric_heap_size);
        setenv("NVSHMEM_SYMMETRIC_SIZE", symmetric_heap_size_str, 1);
    }
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

    int npes = nvshmem_n_pes();
    int mype = nvshmem_my_pe();

    nvshmem_barrier_all();

    hipStream_t compute_stream;
    hipEvent_t compute_done[2];

    double runtime_serial_non_persistent = 0.0;
    if (compare_to_single_gpu)
    {
        CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * sizeof(real)));
        CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * sizeof(real)));

        runtime_serial_non_persistent = single_gpu(nx, ny, iter_max, a_ref_h, 0, true);
    }

    nvshmem_barrier_all();
    // ny - 2 rows are distributed amongst `size` ranks in such a way
    // that each rank gets either (ny - 2) / size or (ny - 2) / size + 1 rows.
    // This optimizes load balancing when (ny - 2) % size != 0
    int chunk_size;
    int chunk_size_low = (ny - 2) / npes;
    int chunk_size_high = chunk_size_low + 1;
    // To calculate the number of ranks that need to compute an extra row,
    // the following formula is derived from this equation:
    // num_ranks_low * chunk_size_low + (size - num_ranks_low) * (chunk_size_low + 1) = ny - 2
    int num_ranks_low = npes * chunk_size_low + npes -
                        (ny - 2); // Number of ranks with chunk_size = chunk_size_low
    if (mype < num_ranks_low)
        chunk_size = chunk_size_low;
    else
        chunk_size = chunk_size_high;

    a = (real *)nvshmem_malloc(
        nx * (chunk_size_high + 2) *
        sizeof(real)); // Using chunk_size_high so that it is same across all PEs
    a_new = (real *)nvshmem_malloc(nx * (chunk_size_high + 2) * sizeof(real));

    hipMemset(a, 0, nx * (chunk_size + 2) * sizeof(real));
    hipMemset(a_new, 0, nx * (chunk_size + 2) * sizeof(real));

    // Calculate local domain boundaries
    int iy_start_global; // My start index in the global array
    if (mype < num_ranks_low)
    {
        iy_start_global = mype * chunk_size_low + 1;
    }
    else
    {
        iy_start_global =
            num_ranks_low * chunk_size_low + (mype - num_ranks_low) * chunk_size_high + 1;
    }
    int iy_end_global = iy_start_global + chunk_size - 1; // My last index in the global array
    // do not process boundaries
    iy_end_global = std::min(iy_end_global, ny - 4);

    int iy_start = 1;
    int iy_end = (iy_end_global - iy_start_global + 1) + iy_start;

    // calculate boundary indices for top and bottom boundaries
    int top_pe = mype > 0 ? mype - 1 : (npes - 1);
    int bottom_pe = (mype + 1) % npes;

    int iy_end_top = (top_pe < num_ranks_low) ? chunk_size_low + 1 : chunk_size_high + 1;
    int iy_start_bottom = 0;

    // Set diriclet boundary conditions on left and right boundary
    initialize_boundaries<<<(ny / npes) / 128 + 1, 128>>>(
        a_new, a, PI, iy_start_global - 1, nx, chunk_size + 2, ny);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    CUDA_RT_CALL(hipStreamCreateWithFlags(&compute_stream, hipStreamNonBlocking));
    CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done[0], hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done[1], hipEventDisableTiming));

    nvshmemx_barrier_all_on_stream(compute_stream);
    CUDA_RT_CALL(hipDeviceSynchronize());

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;

    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x,
                  (chunk_size + dim_block_y - 1) / dim_block_y);

    int iter = 0;

    nvshmem_barrier_all();

    double start = MPI_Wtime();
    PUSH_RANGE("Jacobi solve", 0)

    hipStreamSynchronize(compute_stream);

    while (iter < iter_max)
    {
        // on new iteration: old current vars are now previous vars, old
        // previous vars are no longer needed
        // int prev = iter % 2;
        // int curr = (iter + 1) % 2;

        jacobi_kernel<dim_block_x, dim_block_y>
            <<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, compute_stream>>>(
                a_new, a, iy_start, iy_end, nx, top_pe, iy_end_top, bottom_pe,
                iy_start_bottom);
        CUDA_RT_CALL(hipGetLastError());

        nvshmemx_barrier_all_on_stream(compute_stream);

        std::swap(a_new, a);
        iter++;
    }

    CUDA_RT_CALL(hipDeviceSynchronize());

    nvshmem_barrier_all();
    double stop = MPI_Wtime();
    nvshmem_barrier_all();

    bool result_correct = true;
    if (compare_to_single_gpu)
    {

        CUDA_RT_CALL(hipMemcpy(
            a_h + iy_start_global * nx, a + nx,
            std::min(ny - iy_start_global, chunk_size) * nx * sizeof(real),
            hipMemcpyDeviceToHost));

        for (int iy = iy_start_global; result_correct && (iy < iy_end_global); ++iy)
        {
            for (int ix = 1; result_correct && (ix < (nx - 1)); ++ix)
            {
                if (std::fabs(a_h[iy * nx + ix] -
                              a_ref_h[iy * nx + ix]) > tol)
                {
                    fprintf(stderr,
                            "ERROR on rank %d: a[ %d * %d + %d] = %f does "
                            "not match %f "
                            "(reference)\n",
                            rank, iy, nx, ix, a_h[iy * nx + ix],
                            a_ref_h[iy * nx + ix]);
                    result_correct = 0;
                }
            }
        }
    }
    int global_result_correct = 1;
    MPI_CALL(MPI_Allreduce(&result_correct, &global_result_correct, 1, MPI_INT, MPI_MIN,
                           MPI_COMM_WORLD));

    if (!mype && global_result_correct)
    {
        // printf("Num GPUs: %d.\n", num_devices);
        printf("Execution time: %8.4f s\n", (stop - start));

        if (compare_to_single_gpu)
        {
            printf(
                "Non-persistent kernel - %dx%d: 1 GPU: %8.4f s, %d GPUs: "
                "%8.4f "
                "s, speedup: "
                "%8.2f, "
                "efficiency: %8.2f \n",
                nx, ny, runtime_serial_non_persistent, npes, (stop - start),
                runtime_serial_non_persistent / (stop - start),
                runtime_serial_non_persistent / (npes * (stop - start)) * 100);
        }
    }

    nvshmem_free(a);
    nvshmem_free(a_new);

    CUDA_RT_CALL(hipEventDestroy(compute_done[1]));
    CUDA_RT_CALL(hipEventDestroy(compute_done[0]));
    CUDA_RT_CALL(hipStreamDestroy(compute_stream));

    if (compare_to_single_gpu)
    {
        CUDA_RT_CALL(hipHostFree(a_h));
        CUDA_RT_CALL(hipHostFree(a_ref_h));
    }

    nvshmem_finalize();
    MPI_CALL(MPI_Finalize());

    return (result_correct == 1) ? 0 : 1;
}
