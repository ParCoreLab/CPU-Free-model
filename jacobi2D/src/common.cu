#include "hip/hip_runtime.h"
#include "../include/common.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

bool get_arg(char **begin, char **end, const std::string &arg)
{
    char **itr = std::find(begin, end, arg);
    if (itr != end)
    {
        return true;
    }
    return false;
}

__global__ void initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                      const real pi, const int offset, const int nx,
                                      const int my_ny, const int ny)
{
    for (unsigned int iy = blockIdx.x * blockDim.x + threadIdx.x +1; iy < my_ny-1;
         iy += blockDim.x * gridDim.x)
    {
        const real y0 = sin(2.0 * pi * (offset + iy) / (ny - 1));

        for (unsigned int ix = 0; ix < nx; ix++)
        {
            a[iy * nx + ix] = y0;
            a_new[iy * nx + ix] = y0; 
        }
    }
}

__global__ void jacobi_kernel_single_gpu(real *__restrict__ const a_new,
                                         const real *__restrict__ const a,
                                         real *__restrict__ const l2_norm, const int iy_start,
                                         const int iy_end, const int nx,
                                         const bool calculate_norm)
{
    int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (iy < iy_end && ix < (nx - 1))
    {
        const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                     a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);

        a_new[iy * nx + ix] = new_val;
    }
}

// I changed the kernel, switch it back later
__global__ void jacobi_kernel_single_gpu_perks(real *__restrict__ const a_new,
                                               const real *__restrict__ const a,
                                               real *__restrict__ const l2_norm, const int iy_start,
                                               const int iy_end, const int nx,
                                               const bool calculate_norm)
{
    int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;
    //    real local_l2_norm = 0.0;

    if (iy < iy_end && ix < (nx - 1))
    {
        // const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
        //                              a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);

        const real new_val =
            ((5 * a[(iy - 1) * nx + ix]) + (12 * a[iy * nx + ix + 1]) + (15 * a[iy * nx + ix]) +
             (12 * a[iy * nx + ix - 1]) + (5 * a[(iy + 1) * nx + ix])) /
            118;

        a_new[iy * nx + ix] = new_val;

        //        if (calculate_norm) {
        //            real residue = new_val - a[iy * nx + ix];
        //            local_l2_norm += residue * residue;
        //        }
    }
    //    if (calculate_norm) {
    //        atomicAdd(l2_norm, local_l2_norm);
    //    }
}

__global__ void jacobi_kernel_single_gpu_persistent(real *a_new, real *a, const int iy_start,
                                                    const int iy_end, const int nx,
                                                    const bool calculate_norm, const int iter_max)
{
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;

    //    real local_l2_norm = 0.0;

    int iter = 0;

    while (iter < iter_max)
    {
        if (iy < iy_end && ix < (nx - 1))
        {
            const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                         a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
            a_new[iy * nx + ix] = new_val;

            if (iy_start == iy)
            {
                a_new[iy_end * nx + ix] = new_val;
            }

            if ((iy_end - 1) == iy)
            {
                a_new[(iy_start - 1) * nx + ix] = new_val;
            }

            //        if (calculate_norm) {
            //            real residue = new_val - a[iy * nx + ix];
            //            local_l2_norm += residue * residue;
            //        }
        }

        iter++;

        real *temp_pointer = a_new;
        a_new = a;
        a = temp_pointer;

        cg::sync(grid);
    }

    //    if (calculate_norm) {
    //        atomicAdd(l2_norm, local_l2_norm);
    //    }
}

/*double single_cpu(real *a_h_input, const int nx, const int ny, const int iter_max,
                  real *const a_ref_h, const int nccheck, const bool print)
{
    double start = omp_get_wtime();
    jacobi_gold_iterative(a_h_input, ny, nx, a_ref_h, iter_max);
    return omp_get_wtime() - start;
}*/

double single_gpu(const int nx, const int ny, const int iter_max, real *const a_ref_h,
                  const int nccheck, const bool print)
{
    real *a;
    real *a_new;

    hipStream_t compute_stream;
    hipStream_t push_top_stream;
    hipStream_t push_bottom_stream;
    hipEvent_t compute_done;
    hipEvent_t push_top_done;
    hipEvent_t push_bottom_done;

    //    real* l2_norm_d;
    //    real* l2_norm_h;

    int iy_start = 1;
    int iy_end = (ny - 1);

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * sizeof(real)));

    // Set diriclet boundary conditions on left and right boarder
    initialize_boundaries<<<ny / 128 + 1, 128>>>(a, a_new, PI, 0, nx, ny, ny);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    CUDA_RT_CALL(hipStreamCreate(&compute_stream));
    CUDA_RT_CALL(hipStreamCreate(&push_top_stream));
    CUDA_RT_CALL(hipStreamCreate(&push_bottom_stream));
    CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&push_top_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&push_bottom_done, hipEventDisableTiming));

    //    CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(real)));
    //    CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(real)));

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (print)
        printf(
            "Single GPU jacobi relaxation (non-persistent kernel): %d iterations on %d x %d "
            "mesh "
            "with "
            "norm "
            "check every %d iterations\n",
            iter_max, ny, nx, nccheck);

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x, (ny + dim_block_y - 1) / dim_block_y, 1);

    int iter = 0;
    bool calculate_norm = false;
    //    real l2_norm = 1.0;

    double start = omp_get_wtime();
    PUSH_RANGE("Jacobi solve", 0)
    while (iter < iter_max)
    {
        //        CUDA_RT_CALL(hipMemsetAsync(l2_norm_d, 0, sizeof(real), compute_stream));

        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_top_done, 0));
        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_bottom_done, 0));

        //        calculate_norm = (iter % nccheck) == 0 || (print && ((iter % 100) == 0));
        jacobi_kernel_single_gpu<<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, compute_stream>>>(
            a_new, a, nullptr, iy_start, iy_end, nx, calculate_norm);
        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipEventRecord(compute_done, compute_stream));

        //        if (calculate_norm) {
        //            CUDA_RT_CALL(hipMemcpyAsync(l2_norm_h, l2_norm_d, sizeof(real),
        //            hipMemcpyDeviceToHost,
        //                                         compute_stream));
        //        }

        // Apply periodic boundary conditions

        CUDA_RT_CALL(hipStreamWaitEvent(push_top_stream, compute_done, 0));
        CUDA_RT_CALL(hipMemcpyAsync(a_new, a_new + (iy_end - 1) * nx, nx * sizeof(real),
                                     hipMemcpyDeviceToDevice, push_top_stream));
        CUDA_RT_CALL(hipEventRecord(push_top_done, push_top_stream));

        CUDA_RT_CALL(hipStreamWaitEvent(push_bottom_stream, compute_done, 0));
        CUDA_RT_CALL(hipMemcpyAsync(a_new + iy_end * nx, a_new + iy_start * nx, nx * sizeof(real),
                                     hipMemcpyDeviceToDevice, compute_stream));
        CUDA_RT_CALL(hipEventRecord(push_bottom_done, push_bottom_stream));

        //        if (calculate_norm) {
        //            CUDA_RT_CALL(hipStreamSynchronize(compute_stream));
        //            l2_norm = *l2_norm_h;
        //            l2_norm = std::sqrt(l2_norm);
        //            if (print && (iter % 100) == 0) printf("%5d, %0.6f\n", iter, l2_norm);
        //        }

        std::swap(a_new, a);
        iter++;
    }
    POP_RANGE
    double stop = omp_get_wtime();

    CUDA_RT_CALL(hipMemcpy(a_ref_h, a, nx * ny * sizeof(real), hipMemcpyDeviceToHost));

    CUDA_RT_CALL(hipEventDestroy(push_bottom_done));
    CUDA_RT_CALL(hipEventDestroy(push_top_done));
    CUDA_RT_CALL(hipEventDestroy(compute_done));
    CUDA_RT_CALL(hipStreamDestroy(push_bottom_stream));
    CUDA_RT_CALL(hipStreamDestroy(push_top_stream));
    CUDA_RT_CALL(hipStreamDestroy(compute_stream));

    //    CUDA_RT_CALL(hipHostFree(l2_norm_h));
    //    CUDA_RT_CALL(hipFree(l2_norm_d));

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
    return (stop - start);
}

double single_gpu(real *a_h, const int nx, const int ny, const int iter_max, real *const a_ref_h,
                  const int nccheck, const bool print)
{
    real *a;
    real *a_new;

    hipStream_t compute_stream;
    hipStream_t push_top_stream;
    hipStream_t push_bottom_stream;
    hipEvent_t compute_done;
    hipEvent_t push_top_done;
    hipEvent_t push_bottom_done;

    //    real* l2_norm_d;
    //    real* l2_norm_h;

    int iy_start = 1;
    int iy_end = (ny - 1);

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * sizeof(real)));

    CUDA_RT_CALL(hipMemcpy(a, a_h, nx * ny * sizeof(real), hipMemcpyHostToDevice));

    // CUDA_RT_CALL(hipMemset(a, 0, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * sizeof(real)));

    // Set diriclet boundary conditions on left and right boarder
    // initialize_boundaries<<<ny / 128 + 1, 128>>>(a, a_new, PI, 0, nx, ny, ny);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    CUDA_RT_CALL(hipStreamCreate(&compute_stream));
    CUDA_RT_CALL(hipStreamCreate(&push_top_stream));
    CUDA_RT_CALL(hipStreamCreate(&push_bottom_stream));
    CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&push_top_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&push_bottom_done, hipEventDisableTiming));

    //    CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(real)));
    //    CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(real)));

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (print)
        printf(
            "Single GPU jacobi relaxation (non-persistent kernel): %d iterations on %d x %d "
            "mesh "
            "with "
            "norm "
            "check every %d iterations\n",
            iter_max, ny, nx, nccheck);

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x, (ny + dim_block_y - 1) / dim_block_y, 1);

    int iter = 0;
    bool calculate_norm = false;
    //    real l2_norm = 1.0;

    double start = omp_get_wtime();
    PUSH_RANGE("Jacobi solve", 0)
    while (iter < iter_max)
    {
        //        CUDA_RT_CALL(hipMemsetAsync(l2_norm_d, 0, sizeof(real), compute_stream));

        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_top_done, 0));
        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_bottom_done, 0));

        //        calculate_norm = (iter % nccheck) == 0 || (print && ((iter % 100) == 0));
        jacobi_kernel_single_gpu<<<dim_grid, {dim_block_x, dim_block_y, 1}, 0, compute_stream>>>(
            a_new, a, nullptr, iy_start, iy_end, nx, calculate_norm);
        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipEventRecord(compute_done, compute_stream));

        //        if (calculate_norm) {
        //            CUDA_RT_CALL(hipMemcpyAsync(l2_norm_h, l2_norm_d, sizeof(real),
        //            hipMemcpyDeviceToHost,
        //                                         compute_stream));
        //        }

        // Apply periodic boundary conditions

        CUDA_RT_CALL(hipStreamWaitEvent(push_top_stream, compute_done, 0));
        CUDA_RT_CALL(hipMemcpyAsync(a_new, a_new + (iy_end - 1) * nx, nx * sizeof(real),
                                     hipMemcpyDeviceToDevice, push_top_stream));
        CUDA_RT_CALL(hipEventRecord(push_top_done, push_top_stream));

        CUDA_RT_CALL(hipStreamWaitEvent(push_bottom_stream, compute_done, 0));
        CUDA_RT_CALL(hipMemcpyAsync(a_new + iy_end * nx, a_new + iy_start * nx, nx * sizeof(real),
                                     hipMemcpyDeviceToDevice, compute_stream));
        CUDA_RT_CALL(hipEventRecord(push_bottom_done, push_bottom_stream));

        //        if (calculate_norm) {
        //            CUDA_RT_CALL(hipStreamSynchronize(compute_stream));
        //            l2_norm = *l2_norm_h;
        //            l2_norm = std::sqrt(l2_norm);
        //            if (print && (iter % 100) == 0) printf("%5d, %0.6f\n", iter, l2_norm);
        //        }

        std::swap(a_new, a);
        iter++;
    }
    POP_RANGE
    double stop = omp_get_wtime();

    CUDA_RT_CALL(hipMemcpy(a_ref_h, a, nx * ny * sizeof(real), hipMemcpyDeviceToHost));

    CUDA_RT_CALL(hipEventDestroy(push_bottom_done));
    CUDA_RT_CALL(hipEventDestroy(push_top_done));
    CUDA_RT_CALL(hipEventDestroy(compute_done));
    CUDA_RT_CALL(hipStreamDestroy(push_bottom_stream));
    CUDA_RT_CALL(hipStreamDestroy(push_top_stream));
    CUDA_RT_CALL(hipStreamDestroy(compute_stream));

    //    CUDA_RT_CALL(hipHostFree(l2_norm_h));
    //    CUDA_RT_CALL(hipFree(l2_norm_d));

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
    return (stop - start);
}

double single_gpu_persistent(const int nx, const int ny, const int iter_max, real *const a_ref_h,
                             const int nccheck, const bool print)
{
    real *a;
    real *a_new;

    // Skipping l2-norm calculation for now
    //    real* l2_norm_d;
    //    real* l2_norm_h;

    int iy_start = 1;
    int iy_end = (ny - 1);

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * sizeof(real)));

    // Set diriclet boundary conditions on left and right boarder
    initialize_boundaries<<<ny / 128 + 1, 128>>>(a, a_new, PI, 0, nx, ny, ny);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    //    CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(real)));
    //    CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(real)));

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (print)
        printf(
            "Single GPU jacobi relaxation (persistent kernel): %d iterations on %d x %d mesh "
            "with "
            "norm "
            "check every %d iterations\n",
            iter_max, ny, nx, nccheck);

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;

    dim3 dim_block(dim_block_x, dim_block_y);
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x, (ny + dim_block_y - 1) / dim_block_y, 1);

    bool calculate_norm = false;
    //    real l2_norm = 1.0;

    void *kernelArgs[] = {(void *)&a_new, (void *)&a, (void *)&iy_start,
                          (void *)&iy_end, (void *)&nx, (void *)&calculate_norm,
                          (void *)&iter_max};

    double start = omp_get_wtime();

    CUDA_RT_CALL(hipLaunchCooperativeKernel((void *)jacobi_kernel_single_gpu_persistent, dim_grid,
                                             dim_block, kernelArgs, 0, nullptr));

    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    double stop = omp_get_wtime();

    CUDA_RT_CALL(hipMemcpy(a_ref_h, a, nx * ny * sizeof(real), hipMemcpyDeviceToHost));

    //    CUDA_RT_CALL(hipHostFree(l2_norm_h));
    //    CUDA_RT_CALL(hipFree(l2_norm_d));

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
    return (stop - start);
}

void report_results(const int ny, const int nx, real *a_ref_h, real *a_h, const int num_devices,
                    const double runtime_serial_non_persistent, const double start,
                    const double stop, const bool compare_to_single_gpu)
{
    bool result_correct = true;

    if (compare_to_single_gpu)
    {
        for (int iy = 1; result_correct && (iy < (ny - 1)); ++iy)
        {
            for (int ix = 1; result_correct && (ix < (nx - 1)); ++ix)
            {
                if (std::fabs(a_ref_h[iy * nx + ix] - a_h[iy * nx + ix]) > tol)
                {
                    fprintf(stderr,
                            "ERROR: a[%d * %d + %d] = %.8f does not match %.8f "
                            "(reference)\n",
                            iy, nx, ix, a_h[iy * nx + ix], a_ref_h[iy * nx + ix]);
                    // result_correct = false;
                }
            }
        }
    }

    if (result_correct)
    {
        // printf("Num GPUs: %d.\n", num_devices);
        printf("Execution time: %8.4f s\n", (stop - start));

        if (compare_to_single_gpu)
        {
            printf(
                "Non-persistent kernel - %dx%d: 1 GPU: %8.4f s, %d GPUs: %8.4f s, speedup: "
                "%8.2f, "
                "efficiency: %8.2f \n",
                ny, nx, runtime_serial_non_persistent, num_devices, (stop - start),
                runtime_serial_non_persistent / (stop - start),
                runtime_serial_non_persistent / (num_devices * (stop - start)) * 100);
        }
    }
}
