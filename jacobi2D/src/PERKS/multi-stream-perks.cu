#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <cmath>
#include <cstdio>
#include <iostream>

#include <omp.h>

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>

#include "../../include/PERKS/multi-stream-perks.cuh"
#include "../../include/common.h"

// perks stuff
#include "./common/common.hpp"
#include "./common/cuda_common.cuh"
#include "./common/cuda_computation.cuh"
#include "./common/jacobi_cuda.cuh"
#include "./common/jacobi_reference.hpp"
#include "./common/types.hpp"
#include "./perksconfig.cuh"
#include "config.cuh"

namespace cg = cooperative_groups;

namespace MultiStreamPERKS {
__global__ void __launch_bounds__(1024, 1)
    boundary_sync_kernel(real *a_new, real *a, const int iy_start, const int iy_end, const int nx,
                         const int comm_tile_size, const int num_comm_tiles, const int iter_max,
                         volatile real *local_halo_buffer_for_top_neighbor,
                         volatile real *local_halo_buffer_for_bottom_neighbor,
                         const volatile real *remote_my_halo_buffer_on_top_neighbor,
                         const volatile real *remote_my_halo_buffer_on_bottom_neighbor,
                         const volatile int *local_is_top_neighbor_done_writing_to_me,
                         const volatile int *local_is_bottom_neighbor_done_writing_to_me,
                         volatile int *remote_am_done_writing_to_top_neighbor,
                         volatile int *remote_am_done_writing_to_bottom_neighbor,
                         volatile int *iteration_done) {
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    int num_flags = 2 * num_comm_tiles;

    int iter = 0;

    int cur_iter_mod = 0;
    int next_iter_mod = 1;
    int temp_iter_mod = 0;

    int cur_iter_comm_tile_flag_idx;
    int next_iter_comm_tile_flag_idx;

    int comm_tile_idx;

    int comm_tile_start;
    int comm_tile_end;

    while (iter < iter_max) {
        while (iteration_done[1] != iter) {
        }

        if (blockIdx.x == gridDim.x - 1) {
            for (comm_tile_idx = 0; comm_tile_idx < num_comm_tiles; comm_tile_idx++) {
                comm_tile_start = (comm_tile_idx == 0) ? 1 : comm_tile_idx * comm_tile_size;
                comm_tile_end = (comm_tile_idx == (num_comm_tiles - 1))
                                    ? nx - 1
                                    : (comm_tile_idx + 1) * comm_tile_size;

                int col = threadIdx.y * blockDim.x + threadIdx.x + comm_tile_start;

                cur_iter_comm_tile_flag_idx = comm_tile_idx + cur_iter_mod * num_flags;
                next_iter_comm_tile_flag_idx =
                    (num_comm_tiles + comm_tile_idx) + next_iter_mod * num_flags;

                if (cta.thread_rank() == 0) {
                    while (local_is_top_neighbor_done_writing_to_me[cur_iter_comm_tile_flag_idx] !=
                           iter) {
                    }
                }

                cg::sync(cta);

                if (col < comm_tile_end) {
                    const real first_row_val =
                        0.25 * (a[iy_start * nx + col + 1] + a[iy_start * nx + col - 1] +
                                a[(iy_start + 1) * nx + col] +
                                remote_my_halo_buffer_on_top_neighbor[nx * cur_iter_mod + col]);

                    a_new[iy_start * nx + col] = first_row_val;
                    local_halo_buffer_for_top_neighbor[nx * next_iter_mod + col] = first_row_val;
                }

                cg::sync(cta);

                if (cta.thread_rank() == 0) {
                    remote_am_done_writing_to_top_neighbor[next_iter_comm_tile_flag_idx] = iter + 1;
                }
            }
        } else if (blockIdx.x == gridDim.x - 2) {
            for (comm_tile_idx = 0; comm_tile_idx < num_comm_tiles; comm_tile_idx++) {
                comm_tile_start = (comm_tile_idx == 0) ? 1 : comm_tile_idx * comm_tile_size;
                comm_tile_end = (comm_tile_idx == (num_comm_tiles - 1))
                                    ? nx - 1
                                    : (comm_tile_idx + 1) * comm_tile_size;

                int col = threadIdx.y * blockDim.x + threadIdx.x + comm_tile_start;

                cur_iter_comm_tile_flag_idx =
                    (num_comm_tiles + comm_tile_idx) + cur_iter_mod * num_flags;
                next_iter_comm_tile_flag_idx = comm_tile_idx + next_iter_mod * num_flags;

                if (cta.thread_rank() == 0) {
                    while (
                        local_is_bottom_neighbor_done_writing_to_me[cur_iter_comm_tile_flag_idx] !=
                        iter) {
                    }
                }

                cg::sync(cta);

                if (col < comm_tile_end) {
                    const real last_row_val =
                        0.25 * (a[(iy_end - 1) * nx + col + 1] + a[(iy_end - 1) * nx + col - 1] +
                                remote_my_halo_buffer_on_bottom_neighbor[nx * cur_iter_mod + col] +
                                a[(iy_end - 2) * nx + col]);

                    a_new[(iy_end - 1) * nx + col] = last_row_val;
                    local_halo_buffer_for_bottom_neighbor[nx * next_iter_mod + col] = last_row_val;
                }

                cg::sync(cta);

                if (cta.thread_rank() == 0) {
                    remote_am_done_writing_to_bottom_neighbor[next_iter_comm_tile_flag_idx] =
                        iter + 1;
                }
            }
        }

        real *temp_pointer_first = a_new;
        a_new = a;
        a = temp_pointer_first;

        iter++;

        temp_iter_mod = cur_iter_mod;
        cur_iter_mod = next_iter_mod;
        next_iter_mod = temp_iter_mod;

        cg::sync(grid);

        if (threadIdx.x == 0 && threadIdx.y == 0) {
            iteration_done[0] = iter;
        }

        cg::sync(grid);
    }
}
}  // namespace MultiStreamPERKS

#pragma clang diagnostic push
#pragma ide diagnostic ignored "openmp-use-default-none"
int MultiStreamPERKS::init(int argc, char *argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare");

    real *a[MAX_NUM_DEVICES];
    real *a_new[MAX_NUM_DEVICES];
    int iy_end[MAX_NUM_DEVICES];

    real *halo_buffer_for_top_neighbor[MAX_NUM_DEVICES];
    real *halo_buffer_for_bottom_neighbor[MAX_NUM_DEVICES];

    int *is_top_done_computing_flags[MAX_NUM_DEVICES];
    int *is_bottom_done_computing_flags[MAX_NUM_DEVICES];

    real *a_ref_h;
    real *a_h;

    double runtime_serial_non_persistent = 0.0;

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));
    num_devices = 1;

    // ------------------------------------
    // PERKS config
    // ------------------------------------

    // Buffers
    real(*output)[nx] = (real(*)[nx])getZero2DArray(ny, nx);
    real(*output_gold)[nx] = (real(*)[nx])getZero2DArray(ny, nx);

    // 128 or 256
    int bdimx = 256;
    int blkpsm = 0;

    // damnit
    if (blkpsm <= 0) blkpsm = 100;

    bool async = false;
    bool useSM = true;
    bool usewarmup = false;
    int warmupiteration = -1;
    bool isDoubleTile = true;

    // Change this later
    int ptx = 800;

    int REG_FOLDER_Y = 0;

    if (blkpsm * bdimx >= 2 * 256) {
        if (useSM) {
            if (ptx == 800)
                REG_FOLDER_Y = isDoubleTile
                                   ? (regfolder<HALO, true, 128, 800, true, real, 2 * RTILE_Y>::val)
                                   : (regfolder<HALO, true, 128, 800, true, real>::val);
            if (ptx == 700)
                REG_FOLDER_Y = isDoubleTile
                                   ? (regfolder<HALO, true, 128, 700, true, real, 2 * RTILE_Y>::val)
                                   : (regfolder<HALO, true, 128, 700, true, real>::val);
        } else {
            if (ptx == 800)
                REG_FOLDER_Y =
                    isDoubleTile ? (regfolder<HALO, true, 128, 800, false, real, 2 * RTILE_Y>::val)
                                 : (regfolder<HALO, true, 128, 800, false, real>::val);
            if (ptx == 700)
                REG_FOLDER_Y =
                    isDoubleTile ? (regfolder<HALO, true, 128, 700, false, real, 2 * RTILE_Y>::val)
                                 : (regfolder<HALO, true, 128, 700, false, real>::val);
        }
    } else {
        if (useSM) {
            if (ptx == 800)
                REG_FOLDER_Y = isDoubleTile
                                   ? (regfolder<HALO, true, 256, 800, true, real, 2 * RTILE_Y>::val)
                                   : (regfolder<HALO, true, 256, 800, true, real>::val);
            if (ptx == 700)
                REG_FOLDER_Y = isDoubleTile
                                   ? (regfolder<HALO, true, 256, 700, true, real, 2 * RTILE_Y>::val)
                                   : (regfolder<HALO, true, 256, 700, true, real>::val);
        } else {
            if (ptx == 800)
                REG_FOLDER_Y =
                    isDoubleTile ? (regfolder<HALO, true, 256, 800, false, real, 2 * RTILE_Y>::val)
                                 : (regfolder<HALO, true, 256, 800, false, real>::val);
            if (ptx == 700)
                REG_FOLDER_Y =
                    isDoubleTile ? (regfolder<HALO, true, 256, 700, false, real, 2 * RTILE_Y>::val)
                                 : (regfolder<HALO, true, 256, 700, false, real>::val);
        }
    }

    auto execute_kernel =
        isDoubleTile ? (blkpsm * bdimx >= 2 * 256
                            ? (useSM ? kernel_general_wrapper<real, 2 * RTILE_Y, HALO, 128, true>
                                     : kernel_general_wrapper<real, 2 * RTILE_Y, HALO, 128, false>)
                            : (useSM ? kernel_general_wrapper<real, 2 * RTILE_Y, HALO, 256, true>
                                     : kernel_general_wrapper<real, 2 * RTILE_Y, HALO, 256, false>))
                     : (blkpsm * bdimx >= 2 * 256
                            ? (useSM ? kernel_general_wrapper<real, RTILE_Y, HALO, 128, true>
                                     : kernel_general_wrapper<real, RTILE_Y, HALO, 128, false>)
                            : (useSM ? kernel_general_wrapper<real, RTILE_Y, HALO, 256, true>
                                     : kernel_general_wrapper<real, RTILE_Y, HALO, 256, false>));

    real(*input_h)[nx] = (real(*)[nx])getRandom2DArray(ny, nx);

#pragma omp parallel num_threads(num_devices)
    {
        int dev_id = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(nullptr));

        // Taken from PERKS
        if (compare_to_single_gpu && 0 == dev_id) {
            std::cout << "Running single gpu" << std::endl;

            jacobi_gold_iterative((real *)input_h, ny, nx, (real *)output_gold, iter_max);
        }

#pragma omp barrier

        int chunk_size;
        int chunk_size_low = (ny - 2) / num_devices;
        int chunk_size_high = chunk_size_low + 1;

        int height_per_gpu = ny / num_devices;

        hipDeviceProp_t deviceProp{};
        CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, dev_id));
        int numSms = deviceProp.multiProcessorCount;

        constexpr int dim_block_x = 32;
        constexpr int dim_block_y = 32;

        int comp_tile_size_x = 256;
        int comp_tile_size_y;

        int grid_dim_x = (comp_tile_size_x + dim_block_x - 1) / dim_block_x;
        int max_thread_blocks_y = (numSms - 2) / grid_dim_x;

        comp_tile_size_y = dim_block_y * max_thread_blocks_y;

        // printf("Computation tile dimensions: %dx%d\n", comp_tile_size_x, comp_tile_size_y);

        int num_comp_tiles_x = nx / comp_tile_size_x + (nx % comp_tile_size_x != 0);
        int num_comp_tiles_y =
            height_per_gpu / comp_tile_size_y + (height_per_gpu % comp_tile_size_y != 0);

        int comm_tile_size = dim_block_x * dim_block_y;
        int num_comm_tiles = nx / comm_tile_size + (nx % comm_tile_size != 0);
        int num_flags = 4 * num_comm_tiles;

        // printf("Number of communication tiles: %d\n", num_comm_tiles);

        int num_ranks_low = num_devices * chunk_size_low + num_devices - (ny - 2);
        if (dev_id < num_ranks_low)
            chunk_size = chunk_size_low;
        else
            chunk_size = chunk_size_high;

        const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
        const int bottom = (dev_id + 1) % num_devices;

        if (top != dev_id) {
            int canAccessPeer = 0;
            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));
            if (canAccessPeer) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
            } else {
                std::cerr << "P2P access required from " << dev_id << " to " << top << std::endl;
            }
            if (top != bottom) {
                canAccessPeer = 0;
                CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, bottom));
                if (canAccessPeer) {
                    CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom, 0));
                } else {
                    std::cerr << "P2P access required from " << dev_id << " to " << bottom
                              << std::endl;
                }
            }
        }

#pragma omp barrier

        int *iteration_done_flags[2];

        CUDA_RT_CALL(hipMalloc(iteration_done_flags, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(iteration_done_flags, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMalloc(iteration_done_flags + 1, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(iteration_done_flags + 1, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMemset(iteration_done_flags[0], 0, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMemset(iteration_done_flags[1], 0, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMalloc(a + dev_id, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(a_new + dev_id, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMemset(a[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMemset(a_new[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(halo_buffer_for_top_neighbor + dev_id, 2 * nx * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(halo_buffer_for_bottom_neighbor + dev_id, 2 * nx * sizeof(real)));

        CUDA_RT_CALL(hipMemset(halo_buffer_for_top_neighbor[dev_id], 0, 2 * nx * sizeof(real)));
        CUDA_RT_CALL(hipMemset(halo_buffer_for_bottom_neighbor[dev_id], 0, 2 * nx * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(is_top_done_computing_flags + dev_id, num_flags * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(is_bottom_done_computing_flags + dev_id, num_flags * sizeof(int)));

        CUDA_RT_CALL(hipMemset(is_top_done_computing_flags[dev_id], 0, num_flags * sizeof(int)));
        CUDA_RT_CALL(
            hipMemset(is_bottom_done_computing_flags[dev_id], 0, num_flags * sizeof(int)));

        // Calculate local domain boundaries
        int iy_start_global;  // My start index in the global array
        if (dev_id < num_ranks_low) {
            iy_start_global = dev_id * chunk_size_low + 1;
        } else {
            iy_start_global =
                num_ranks_low * chunk_size_low + (dev_id - num_ranks_low) * chunk_size_high + 1;
        }
        int iy_end_global = iy_start_global + chunk_size - 1;  // My last index in the global array

        int iy_start = 1;
        iy_end[dev_id] = (iy_end_global - iy_start_global + 1) + iy_start;

        // Set diriclet boundary conditions on left and right border
        initialize_boundaries<<<(ny / num_devices) / 128 + 1, 128>>>(
            a[dev_id], a_new[dev_id], PI, iy_start_global - 1, nx, chunk_size + 2, ny);
        CUDA_RT_CALL(hipGetLastError());

        CUDA_RT_CALL(hipDeviceSynchronize());

        dim3 dim_grid(numSms - 2, 1, 1);
        dim3 dim_block(dim_block_x, dim_block_y);

        void *kernelArgsInner[] = {(void *)&a_new[dev_id],
                                   (void *)&a[dev_id],
                                   (void *)&iy_start,
                                   (void *)&iy_end[dev_id],
                                   (void *)&nx,
                                   (void *)&comp_tile_size_x,
                                   (void *)&comp_tile_size_y,
                                   (void *)&num_comp_tiles_x,
                                   (void *)&num_comp_tiles_y,
                                   (void *)&iter_max,
                                   (void *)&halo_buffer_for_top_neighbor[dev_id],
                                   (void *)&halo_buffer_for_bottom_neighbor[dev_id],
                                   (void *)&halo_buffer_for_bottom_neighbor[top],
                                   (void *)&halo_buffer_for_top_neighbor[bottom],
                                   (void *)&is_top_done_computing_flags[dev_id],
                                   (void *)&is_bottom_done_computing_flags[dev_id],
                                   (void *)&is_bottom_done_computing_flags[top],
                                   (void *)&is_top_done_computing_flags[bottom],
                                   (void *)&iteration_done_flags[0]};

        void *kernelArgsBoundary[] = {(void *)&a_new[dev_id],
                                      (void *)&a[dev_id],
                                      (void *)&iy_start,
                                      (void *)&iy_end[dev_id],
                                      (void *)&nx,
                                      (void *)&comm_tile_size,
                                      (void *)&num_comm_tiles,
                                      (void *)&iter_max,
                                      (void *)&halo_buffer_for_top_neighbor[dev_id],
                                      (void *)&halo_buffer_for_bottom_neighbor[dev_id],
                                      (void *)&halo_buffer_for_bottom_neighbor[top],
                                      (void *)&halo_buffer_for_top_neighbor[bottom],
                                      (void *)&is_top_done_computing_flags[dev_id],
                                      (void *)&is_bottom_done_computing_flags[dev_id],
                                      (void *)&is_bottom_done_computing_flags[top],
                                      (void *)&is_top_done_computing_flags[bottom],
                                      (void *)&iteration_done_flags[0]};

#pragma omp barrier
        double start = omp_get_wtime();

        hipStream_t inner_domain_stream;
        hipStream_t boundary_sync_stream;

        CUDA_RT_CALL(hipStreamCreate(&inner_domain_stream));
        CUDA_RT_CALL(hipStreamCreate(&boundary_sync_stream));

        const int LOCAL_RTILE_Y = isDoubleTile ? RTILE_Y * 2 : RTILE_Y;

        int sm_count;
        hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, 0);

        // initialization input and output space
        real *input;
        CUDA_RT_CALL(hipMalloc(&input, sizeof(real) * ((ny - 0) * (nx - 0))));
        CUDA_RT_CALL(hipMemcpy(input, input_h, sizeof(real) * ((ny - 0) * (nx - 0)),
                                hipMemcpyHostToDevice));
        real *__var_1__;
        CUDA_RT_CALL(hipMalloc(&__var_1__, sizeof(real) * ((ny - 0) * (nx - 0))));
        real *__var_2__;
        CUDA_RT_CALL(hipMalloc(&__var_2__, sizeof(real) * ((ny - 0) * (nx - 0))));

        real *L2_cache3;
        real *L2_cache4;
        size_t L2_utage_2 = sizeof(real) * (ny)*2 * (nx / bdimx) * HALO;

        CUDA_RT_CALL(hipMalloc(&L2_cache3, L2_utage_2 * 2));
        L2_cache4 = L2_cache3 + (ny)*2 * (nx / bdimx) * HALO;

        // initialize shared memory
        int maxSharedMemory;
        CUDA_RT_CALL(hipDeviceGetAttribute(&maxSharedMemory,
                                            hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, 0));

        int SharedMemoryUsed = maxSharedMemory - 1024;
        CUDA_RT_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(
            execute_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, SharedMemoryUsed));

        size_t executeSM = 0;

        int basic_sm_space = (LOCAL_RTILE_Y + 2 * HALO) * (bdimx + 2 * HALO) + 1;

        size_t sharememory_basic = (basic_sm_space) * sizeof(real);
        executeSM = sharememory_basic;
        {
#define halo HALO
            executeSM += (HALO * 2 * ((REG_FOLDER_Y)*LOCAL_RTILE_Y + isBOX)) * sizeof(real);
#undef halo
        }

        int numBlocksPerSm_current = 1000;

        CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &numBlocksPerSm_current, execute_kernel, bdimx, executeSM));
        CUDA_RT_CALL(hipDeviceSynchronize());
        // printf("");
        // int smbound=SharedMemoryUsed/executeSM;
        // printf("%d,%d,%d\n",numBlocksPerSm_current,blkpsm,smbound);
        if (blkpsm != 0) {
            numBlocksPerSm_current = min(numBlocksPerSm_current, blkpsm);
        }

        dim3 block_dim(bdimx);
        dim3 grid_dim(nx / bdimx, sm_count * numBlocksPerSm_current / (nx / bdimx));

        dim3 executeBlockDim = block_dim;
        dim3 executeGridDim = grid_dim;

#define halo HALO

        size_t max_sm_flder = 0;
        int tmp0 = SharedMemoryUsed / sizeof(real) / numBlocksPerSm_current;
        int tmp1 = 2 * HALO * isBOX;
        int tmp2 = basic_sm_space;
        int tmp3 = 2 * HALO * (REG_FOLDER_Y)*LOCAL_RTILE_Y;
        int tmp4 = 2 * HALO * (bdimx + 2 * HALO);
        tmp0 = tmp0 - tmp1 - tmp2 - tmp3 - tmp4;
        tmp0 = tmp0 > 0 ? tmp0 : 0;
        max_sm_flder = (tmp0) / (bdimx + 4 * HALO) / LOCAL_RTILE_Y;
        // printf("smflder is %d\n",max_sm_flder);
        if (!useSM) max_sm_flder = 0;
        if (useSM && max_sm_flder == 0) {
            std::cout << "Jesse" << std::endl;
        }

        size_t sm_cache_size = max_sm_flder == 0 ? 0
                                                 : (max_sm_flder * LOCAL_RTILE_Y + 2 * HALO) *
                                                       (bdimx + 2 * HALO) * sizeof(real);
        size_t y_axle_halo =
            (HALO * 2 * ((max_sm_flder + REG_FOLDER_Y) * LOCAL_RTILE_Y + isBOX)) * sizeof(real);
        executeSM = sharememory_basic + y_axle_halo;
        executeSM += sm_cache_size;

#undef halo
        void *ExecuteKernelArgs[] = {(void *)&input,     (void **)&ny,         (void *)&nx,
                                     (void *)&__var_2__, (void *)&L2_cache3,   (void *)&L2_cache4,
                                     (void *)&iter_max,  (void *)&max_sm_flder};

        CUDA_RT_CALL(hipLaunchCooperativeKernel((void *)execute_kernel, executeGridDim,
                                                 executeBlockDim, ExecuteKernelArgs, executeSM,
                                                 inner_domain_stream));

        //        CUDA_RT_CALL(hipLaunchCooperativeKernel((void
        //        *)MultiStreamPERKS::boundary_sync_kernel, 2,
        //                                                 dim_block, kernelArgsBoundary, 0,
        //                                                 boundary_sync_stream));
        CUDA_RT_CALL(hipDeviceSynchronize());

        // Need to swap pointers on CPU if iteration count is odd
        // Technically, we don't know the iteration number (since we'll be doing l2-norm)
        // Could write iter to CPU when kernel is done
        // if (iter_max % 2 == 1) {
        // std::swap(a_new[dev_id], a[dev_id]);
        // }

#pragma omp barrier
        double stop = omp_get_wtime();

        if (compare_to_single_gpu) {
            // CUDA_RT_CALL(
            //     hipMemcpy(a_h + iy_start_global * nx, a[dev_id] + nx,
            //                std::min((ny - iy_start_global) * nx, chunk_size * nx) * sizeof(real),
            //                hipMemcpyDeviceToHost));

            // CUDA_RT_CALL(
            // hipMemcpy(a_h + iy_start_global * nx, output,
            // std::min((ny - iy_start_global) * nx, chunk_size * nx) * sizeof(real),
            // hipMemcpyDeviceToHost));
        }

#pragma omp barrier

#pragma omp master
        {
            report_results(ny, nx, a_ref_h, a_h, num_devices, runtime_serial_non_persistent, start,
                           stop, /* compare_to_single_gpu */ false);

            // report_results(ny, nx, a_ref_h, output, num_devices, runtime_serial_non_persistent,
            // start, stop, compare_to_single_gpu);
        }

        // CUDA_RT_CALL(hipFree(a_new[dev_id]));
        // CUDA_RT_CALL(hipFree(a[dev_id]));

        if (compare_to_single_gpu && 0 == dev_id) {
            if (iter_max % 2 == 1) {
                CUDA_RT_CALL(hipMemcpy(output, __var_2__, sizeof(real) * ((ny - 0) * (nx - 0)),
                                        hipMemcpyDeviceToHost));
            } else {
                CUDA_RT_CALL(hipMemcpy(output, input, sizeof(real) * ((ny - 0) * (nx - 0)),
                                        hipMemcpyDeviceToHost));
            }

            int halo = iter_max;

            double error = checkError2D(nx, (real *)output, (real *)output_gold, halo, ny - halo,
                                        halo, nx - halo);

            printf("[Test] RMS Error : %e\n", error);
        }
    }

    return 0;
}
#pragma clang diagnostic pop
