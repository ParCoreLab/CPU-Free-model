#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <cmath>
#include <cstdio>
#include <iostream>

#include <omp.h>

#include <hip/hip_cooperative_groups.h>

#include "../../include/common.h"
#include "../../include/multi-stream/multi-gpu-peer-tiling.cuh"

namespace cg = cooperative_groups;

namespace MultiGPUPeerTiling
{
    __global__ void __launch_bounds__(1024, 1)
        jacobi_kernel(real *a_new, real *a,
                      const int iy_start, const int iy_end,
                      const int nx, const int iter_max,
                      volatile int *iteration_done)
    {
        cg::thread_block cta = cg::this_thread_block();
        cg::grid_group grid = cg::this_grid();

        int iter = 0;

        int cur_iter_mod = 0;
        int next_iter_mod = 1;

        const int comp_size_iy = gridDim.y * blockDim.y * nx;
        const int comp_size_ix = gridDim.x * blockDim.x;

        const int comp_start_iy = (blockIdx.y * blockDim.y + threadIdx.y + iy_start + 1) * nx;
        const int comp_start_ix = (blockIdx.x * blockDim.x + threadIdx.x + 1);

        const int end_iy = (iy_end - 1) * nx;
        const int end_ix = (nx - 1);

        while (iter < iter_max)
        {
            for (int iy = comp_start_iy; iy < end_iy; iy += comp_size_iy)
            {
                for (int ix = comp_start_ix; ix < end_ix; ix += comp_size_ix)
                {
                    a_new[iy + ix] = 0.25 *
                                     (a[iy + ix + 1] + a[iy + ix - 1] +
                                      a[iy + nx + ix] + a[iy - nx + ix]);
                }
            }

            real *temp = a_new;
            a_new = a;
            a = temp;

            iter++;

            cur_iter_mod = next_iter_mod;
            next_iter_mod = 1 - cur_iter_mod;

            if (!grid.thread_rank())
            {
                while (iteration_done[0] != iter)
                {
                }
                iteration_done[1] = iter;
            }

            cg::sync(grid);
        }
    }

    __global__ void __launch_bounds__(1024, 1)
        boundary_sync_kernel(real *a_new, real *a,
                             const int iy_start, const int iy_end,
                             const int nx, const int iter_max,
                             volatile real *local_halo_buffer_for_top_neighbor,
                             volatile real *local_halo_buffer_for_bottom_neighbor,
                             volatile real *remote_my_halo_buffer_on_top_neighbor,
                             volatile real *remote_my_halo_buffer_on_bottom_neighbor,
                             volatile int *local_is_top_neighbor_done_writing_to_me,
                             volatile int *local_is_bottom_neighbor_done_writing_to_me,
                             volatile int *remote_am_done_writing_to_top_neighbor,
                             volatile int *remote_am_done_writing_to_bottom_neighbor,
                             volatile int *iteration_done)
    {
        cg::thread_block cta = cg::this_thread_block();
        cg::grid_group grid = cg::this_grid();

        int iter = 0;

        int cur_iter_mod = 0;
        int next_iter_mod = 1;

        const int end_iy = (iy_end - 1) * nx;
        const int end_ix = (nx - 1);

        const int comm_size_ix = blockDim.x;

        const int comm_start_ix = threadIdx.x + 1;
        const int comm_start_iy = iy_start * nx;

        while (iter < iter_max)
        {
            while (iteration_done[1] != iter)
            {
            }
            if (blockIdx.x == gridDim.x - 1)
            {
                if (!cta.thread_rank())
                {
                    while (local_is_top_neighbor_done_writing_to_me[cur_iter_mod * 2] !=
                           iter)
                    {
                    }
                }
                cg::sync(cta);

                for (int ix = comm_start_ix; ix < end_ix; ix += comm_size_ix)
                {
                    const real first_row_val = 0.25 * (a[comm_start_iy + ix + 1] +
                                                       a[comm_start_iy + ix - 1] +
                                                       a[comm_start_iy + nx + ix] +
                                                       remote_my_halo_buffer_on_top_neighbor[cur_iter_mod * nx + ix]);
                    a_new[comm_start_iy + ix] = first_row_val;
                    local_halo_buffer_for_top_neighbor[nx * next_iter_mod + ix] = first_row_val;
                }

                cg::sync(cta);

                if (!cta.thread_rank())
                {
                    remote_am_done_writing_to_top_neighbor[next_iter_mod * 2 + 1] = iter + 1;
                }
            }
            else if (blockIdx.x == gridDim.x - 2)
            {
                if (!cta.thread_rank())
                {
                    while (
                        local_is_bottom_neighbor_done_writing_to_me[cur_iter_mod * 2 + 1] !=
                        iter)
                    {
                    }
                }
                cg::sync(cta);

                for (int ix = comm_start_ix; ix < end_ix; ix += comm_size_ix)
                {
                    const real last_row_val = 0.25 * (a[end_iy + ix + 1] +
                                                      a[end_iy + ix - 1] +
                                                      remote_my_halo_buffer_on_bottom_neighbor[cur_iter_mod * nx + ix] +
                                                      a[end_iy - nx + ix]);
                    a_new[end_iy + ix] = last_row_val;
                    local_halo_buffer_for_bottom_neighbor[nx * next_iter_mod + ix] = last_row_val;
                }

                cg::sync(cta);

                if (!cta.thread_rank())
                {
                    remote_am_done_writing_to_bottom_neighbor[next_iter_mod * 2] =
                        iter + 1;
                }
            }
            real *temp_pointer_first = a_new;
            a_new = a;
            a = temp_pointer_first;

            iter++;

            cur_iter_mod = next_iter_mod;
            next_iter_mod = 1 - cur_iter_mod;

            if (!grid.thread_rank())
            {
                iteration_done[0] = iter;
            }

            cg::sync(grid);
        }
    }
} // namespace MultiGPUPeerTiling

int MultiGPUPeerTiling::init(int argc, char *argv[])
{
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare");

    real *a[MAX_NUM_DEVICES];
    real *a_new[MAX_NUM_DEVICES];
    int iy_end[MAX_NUM_DEVICES];

    real *halo_buffer_for_top_neighbor[MAX_NUM_DEVICES];
    real *halo_buffer_for_bottom_neighbor[MAX_NUM_DEVICES];

    int *is_top_done_computing_flags[MAX_NUM_DEVICES];
    int *is_bottom_done_computing_flags[MAX_NUM_DEVICES];

    int *iteration_done_flags[MAX_NUM_DEVICES];

    real *a_ref_h;
    real *a_h;

    double runtime_serial_non_persistent = 0.0;

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

#pragma omp parallel num_threads(num_devices)
    {
        int dev_id = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(nullptr));

        if (compare_to_single_gpu && 0 == dev_id)
        {
            CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * sizeof(real)));
            CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * sizeof(real)));

            runtime_serial_non_persistent = single_gpu(nx, ny, iter_max, a_ref_h, 0, true);
        }

#pragma omp barrier

        int chunk_size;
        int chunk_size_low = (ny - 2) / num_devices;
        int chunk_size_high = chunk_size_low + 1;

        int height_per_gpu = ny / num_devices;

        hipDeviceProp_t deviceProp{};
        CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, dev_id));
        int numSms = deviceProp.multiProcessorCount;

        constexpr int dim_block_x = 32;
        constexpr int dim_block_y = 32;

        constexpr int grid_dim_x = 8;
        const int grid_dim_y = (numSms - 2) / grid_dim_x;
        constexpr int num_flags = 4;

        int num_ranks_low = num_devices * chunk_size_low + num_devices - (ny - 2);
        if (dev_id < num_ranks_low)
            chunk_size = chunk_size_low;
        else
            chunk_size = chunk_size_high;

        const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
        const int bottom = (dev_id + 1) % num_devices;

        if (top != dev_id)
        {
            int canAccessPeer = 0;
            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));
            if (canAccessPeer)
            {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
            }
            else
            {
                std::cerr << "P2P access required from " << dev_id << " to " << top << std::endl;
            }
            if (top != bottom)
            {
                canAccessPeer = 0;
                CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, bottom));
                if (canAccessPeer)
                {
                    CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom, 0));
                }
                else
                {
                    std::cerr << "P2P access required from " << dev_id << " to " << bottom
                              << std::endl;
                }
            }
        }

#pragma omp barrier

        CUDA_RT_CALL(hipMalloc(iteration_done_flags + dev_id, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMemset(iteration_done_flags[dev_id], 0, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMalloc(a + dev_id, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(a_new + dev_id, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMemset(a[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMemset(a_new[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(halo_buffer_for_top_neighbor + dev_id, 2 * nx * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(halo_buffer_for_bottom_neighbor + dev_id, 2 * nx * sizeof(real)));

        CUDA_RT_CALL(hipMemset(halo_buffer_for_top_neighbor[dev_id], 0, 2 * nx * sizeof(real)));
        CUDA_RT_CALL(hipMemset(halo_buffer_for_bottom_neighbor[dev_id], 0, 2 * nx * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(is_top_done_computing_flags + dev_id, num_flags * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(is_bottom_done_computing_flags + dev_id, num_flags * sizeof(int)));

        CUDA_RT_CALL(hipMemset(is_top_done_computing_flags[dev_id], 0, num_flags * sizeof(int)));
        CUDA_RT_CALL(
            hipMemset(is_bottom_done_computing_flags[dev_id], 0, num_flags * sizeof(int)));

        // Calculate local domain boundaries
        int iy_start_global; // My start index in the global array
        if (dev_id < num_ranks_low)
        {
            iy_start_global = dev_id * chunk_size_low + 1;
        }
        else
        {
            iy_start_global =
                num_ranks_low * chunk_size_low + (dev_id - num_ranks_low) * chunk_size_high + 1;
        }
        int iy_end_global = iy_start_global + chunk_size - 1; // My last index in the global array

        int iy_start = 1;
        iy_end[dev_id] = (iy_end_global - iy_start_global + 1) + iy_start;

        // Set diriclet boundary conditions on left and right border
        initialize_boundaries<<<(ny / num_devices) / 128 + 1, 128>>>(
            a[dev_id], a_new[dev_id], PI, iy_start_global - 1, nx, chunk_size + 2, ny);
        CUDA_RT_CALL(hipGetLastError());

        CUDA_RT_CALL(hipDeviceSynchronize());

        dim3 comp_dim_grid(grid_dim_x, grid_dim_y);
        dim3 comp_dim_block(dim_block_x, dim_block_y);

        dim3 comm_dim_grid(2);
        dim3 comm_dim_block(dim_block_x * dim_block_y);

        void *kernelArgsInner[] = {(void *)&a_new[dev_id],
                                   (void *)&a[dev_id],
                                   (void *)&iy_start,
                                   (void *)&iy_end[dev_id],
                                   (void *)&nx,
                                   (void *)&iter_max,
                                   (void *)&iteration_done_flags[dev_id]};

        void *kernelArgsBoundary[] = {(void *)&a_new[dev_id],
                                      (void *)&a[dev_id],
                                      (void *)&iy_start,
                                      (void *)&iy_end[dev_id],
                                      (void *)&nx,
                                      (void *)&iter_max,
                                      (void *)&halo_buffer_for_top_neighbor[dev_id],
                                      (void *)&halo_buffer_for_bottom_neighbor[dev_id],
                                      (void *)&halo_buffer_for_bottom_neighbor[top],
                                      (void *)&halo_buffer_for_top_neighbor[bottom],
                                      (void *)&is_top_done_computing_flags[dev_id],
                                      (void *)&is_bottom_done_computing_flags[dev_id],
                                      (void *)&is_bottom_done_computing_flags[top],
                                      (void *)&is_top_done_computing_flags[bottom],
                                      (void *)&iteration_done_flags[dev_id]};

#pragma omp barrier
        double start = omp_get_wtime();

        hipStream_t inner_domain_stream;
        hipStream_t boundary_sync_stream;

        CUDA_RT_CALL(hipStreamCreate(&inner_domain_stream));
        CUDA_RT_CALL(hipStreamCreate(&boundary_sync_stream));

        // THE KERNELS ARE SERIALIZED!
        // perhaps only on V100
        CUDA_RT_CALL(hipLaunchCooperativeKernel((void *)MultiGPUPeerTiling::jacobi_kernel,
                                                 comp_dim_grid, comp_dim_block, kernelArgsInner, 0,
                                                 inner_domain_stream));

        CUDA_RT_CALL(hipLaunchCooperativeKernel((void *)MultiGPUPeerTiling::boundary_sync_kernel,
                                                 comm_dim_grid, comm_dim_block, kernelArgsBoundary, 0,
                                                 boundary_sync_stream));

        CUDA_RT_CALL(hipDeviceSynchronize());

        // Need to swap pointers on CPU if iteration count is odd
        // Technically, we don't know the iteration number (since we'll be doing l2-norm)
        // Could write iter to CPU when kernel is done
        if (iter_max % 2 == 1)
        {
            std::swap(a_new[dev_id], a[dev_id]);
        }

#pragma omp barrier
        double stop = omp_get_wtime();

        if (compare_to_single_gpu)
        {
            CUDA_RT_CALL(
                hipMemcpy(a_h + iy_start_global * nx, a[dev_id] + nx,
                           std::min((ny - iy_start_global) * nx, chunk_size * nx) * sizeof(real),
                           hipMemcpyDeviceToHost));
        }

#pragma omp barrier

#pragma omp master
        {
            report_results(ny, nx, a_ref_h, a_h, num_devices, runtime_serial_non_persistent, start,
                           stop, compare_to_single_gpu);
        }

        CUDA_RT_CALL(hipFree(a_new[dev_id]));
        CUDA_RT_CALL(hipFree(a[dev_id]));
        CUDA_RT_CALL(hipFree(halo_buffer_for_top_neighbor[dev_id]));
        CUDA_RT_CALL(hipFree(halo_buffer_for_bottom_neighbor[dev_id]));
        CUDA_RT_CALL(hipFree(is_top_done_computing_flags[dev_id]));
        CUDA_RT_CALL(hipFree(is_bottom_done_computing_flags[dev_id]));
        CUDA_RT_CALL(hipFree(iteration_done_flags[dev_id]));

        if (compare_to_single_gpu && 0 == dev_id)
        {
            CUDA_RT_CALL(hipHostFree(a_h));
            CUDA_RT_CALL(hipHostFree(a_ref_h));
        }
    }

    return 0;
}
