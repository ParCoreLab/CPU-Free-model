#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */

#include "../../include/single-stream/multi-threaded-one-block-comm.cuh"

namespace cg = cooperative_groups;

namespace SSMultiThreadedOneBlockComm
{
    __global__ void __launch_bounds__(1024, 1)
        jacobi_kernel(real *a_new, real *a, const int iy_start, const int iy_end, const int nx,
                      const int grid_dim_x, const int iter_max,
                      volatile real *local_halo_buffer_for_top_neighbor,
                      volatile real *local_halo_buffer_for_bottom_neighbor,
                      volatile real *remote_my_halo_buffer_on_top_neighbor,
                      volatile real *remote_my_halo_buffer_on_bottom_neighbor,
                      volatile int *local_is_top_neighbor_done_writing_to_me,
                      volatile int *local_is_bottom_neighbor_done_writing_to_me,
                      volatile int *remote_am_done_writing_to_top_neighbor,
                      volatile int *remote_am_done_writing_to_bottom_neighbor)
    {
        cg::thread_block cta = cg::this_thread_block();
        cg::grid_group grid = cg::this_grid();

        int iter = 0;
        int cur_iter_mod = 0;
        int next_iter_mod = 1;

        const int comp_size_iy = ((gridDim.x - 1) / grid_dim_x) * blockDim.y * nx;
        const int comp_size_ix = grid_dim_x * blockDim.x;

        const int comp_start_iy = ((blockIdx.x / grid_dim_x) * blockDim.y + threadIdx.y + iy_start + 1) * nx;
        const int comp_start_ix = ((blockIdx.x % grid_dim_x) * blockDim.x + threadIdx.x + 1);

        const int end_iy = (iy_end - 1) * nx;
        const int end_ix = (nx - 1);

        const int comm_size_ix = blockDim.y * blockDim.x;

        const int comm_start_ix = threadIdx.y * blockDim.x + threadIdx.x + 1;
        const int comm_start_iy = iy_start * nx;

        while (iter < iter_max)
        {
            if (blockIdx.x == gridDim.x - 1)
            {
                if (!cta.thread_rank())
                {
                    while (local_is_top_neighbor_done_writing_to_me[cur_iter_mod * 2] !=
                           iter)
                    {
                    }
                }
                cg::sync(cta);

                for (int ix = comm_start_ix; ix < end_ix; ix += comm_size_ix)
                {
                    const real first_row_val = 0.25 * (a[comm_start_iy + ix + 1] +
                                                       a[comm_start_iy + ix - 1] +
                                                       a[comm_start_iy + nx + ix] +
                                                       remote_my_halo_buffer_on_top_neighbor[cur_iter_mod * nx + ix]);
                    a_new[comm_start_iy + ix] = first_row_val;
                    local_halo_buffer_for_top_neighbor[nx * next_iter_mod + ix] = first_row_val;
                }

                cg::sync(cta);

                if (!cta.thread_rank())
                {
                    remote_am_done_writing_to_top_neighbor[next_iter_mod * 2 + 1] = iter + 1;

                    while (
                        local_is_bottom_neighbor_done_writing_to_me[cur_iter_mod * 2 + 1] !=
                        iter)
                    {
                    }
                }
                cg::sync(cta);

                for (int ix = comm_start_ix; ix < end_ix; ix += comm_size_ix)
                {
                    const real last_row_val = 0.25 * (a[end_iy + ix + 1] +
                                                      a[end_iy + ix - 1] +
                                                      remote_my_halo_buffer_on_bottom_neighbor[cur_iter_mod * nx + ix] +
                                                      a[end_iy - nx + ix]);
                    a_new[end_iy + ix] = last_row_val;
                    local_halo_buffer_for_bottom_neighbor[nx * next_iter_mod + ix] = last_row_val;
                }

                cg::sync(cta);

                if (!cta.thread_rank())
                {
                    remote_am_done_writing_to_bottom_neighbor[next_iter_mod * 2] =
                        iter + 1;
                }
            }
            else
            {
                for (int iy = comp_start_iy; iy < end_iy; iy += comp_size_iy)
                {
                    for (int ix = comp_start_ix; ix < end_ix; ix += comp_size_ix)
                    {
                        a_new[iy + ix] = 0.25 *
                                         (a[iy + ix + 1] + a[iy + ix - 1] +
                                          a[iy + nx + ix] + a[iy - nx + ix]);
                    }
                }
            }

            real *temp_pointer = a_new;
            a_new = a;
            a = temp_pointer;

            iter++;

            next_iter_mod = cur_iter_mod;
            cur_iter_mod = 1 - cur_iter_mod;

            cg::sync(grid);
        }
    }
} // namespace SSMultiThreadedOneBlockComm

int SSMultiThreadedOneBlockComm::init(int argc, char *argv[])
{
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare");

    real *a[MAX_NUM_DEVICES];
    real *a_new[MAX_NUM_DEVICES];
    int iy_end[MAX_NUM_DEVICES];

    real *halo_buffer_for_top_neighbor[MAX_NUM_DEVICES];
    real *halo_buffer_for_bottom_neighbor[MAX_NUM_DEVICES];

    int *is_top_done_computing_flags[MAX_NUM_DEVICES];
    int *is_bottom_done_computing_flags[MAX_NUM_DEVICES];

    real *a_ref_h;
    real *a_h;

    double runtime_serial_non_persistent = 0.0;

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

#pragma omp parallel num_threads(num_devices)
    {
        int dev_id = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(nullptr));

        if (compare_to_single_gpu && 0 == dev_id)
        {
            CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * sizeof(real)));
            CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * sizeof(real)));

            runtime_serial_non_persistent = single_gpu(nx, ny, iter_max, a_ref_h, 0, true);
        }

#pragma omp barrier

        int chunk_size;
        int chunk_size_low = (ny - 2) / num_devices;
        int chunk_size_high = chunk_size_low + 1;

        hipDeviceProp_t deviceProp{};
        CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, dev_id));
        int numSms = deviceProp.multiProcessorCount;

        constexpr int dim_block_x = 32;
        constexpr int dim_block_y = 32;

        constexpr int grid_dim_x = 8;
        const int grid_dim_y = (numSms - 1) / grid_dim_x;

        constexpr int num_flags = 4;

        int num_ranks_low = num_devices * chunk_size_low + num_devices - (ny - 2);
        if (dev_id < num_ranks_low)
            chunk_size = chunk_size_low;
        else
            chunk_size = chunk_size_high;

        const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
        const int bottom = (dev_id + 1) % num_devices;

        if (top != dev_id)
        {
            int canAccessPeer = 0;
            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));
            if (canAccessPeer)
            {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
            }
            else
            {
                std::cerr << "P2P access required from " << dev_id << " to " << top << std::endl;
            }
            if (top != bottom)
            {
                canAccessPeer = 0;
                CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, bottom));
                if (canAccessPeer)
                {
                    CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom, 0));
                }
                else
                {
                    std::cerr << "P2P access required from " << dev_id << " to " << bottom
                              << std::endl;
                }
            }
        }

#pragma omp barrier

        CUDA_RT_CALL(hipMalloc(a + dev_id, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(a_new + dev_id, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMemset(a[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMemset(a_new[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(halo_buffer_for_top_neighbor + dev_id, 2 * nx * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(halo_buffer_for_bottom_neighbor + dev_id, 2 * nx * sizeof(real)));

        CUDA_RT_CALL(hipMemset(halo_buffer_for_top_neighbor[dev_id], 0, 2 * nx * sizeof(real)));
        CUDA_RT_CALL(hipMemset(halo_buffer_for_bottom_neighbor[dev_id], 0, 2 * nx * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(is_top_done_computing_flags + dev_id, num_flags * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(is_bottom_done_computing_flags + dev_id, num_flags * sizeof(int)));

        CUDA_RT_CALL(hipMemset(is_top_done_computing_flags[dev_id], 0, num_flags * sizeof(int)));
        CUDA_RT_CALL(
            hipMemset(is_bottom_done_computing_flags[dev_id], 0, num_flags * sizeof(int)));

        // Calculate local domain boundaries
        int iy_start_global; // My start index in the global array
        if (dev_id < num_ranks_low)
        {
            iy_start_global = dev_id * chunk_size_low + 1;
        }
        else
        {
            iy_start_global =
                num_ranks_low * chunk_size_low + (dev_id - num_ranks_low) * chunk_size_high + 1;
        }
        int iy_end_global = iy_start_global + chunk_size - 1; // My last index in the global array

        int iy_start = 1;
        iy_end[dev_id] = (iy_end_global - iy_start_global + 1) + iy_start;

        // Set diriclet boundary conditions on left and right border
        initialize_boundaries<<<(ny / num_devices) / 128 + 1, 128>>>(
            a_new[dev_id], a[dev_id], PI, iy_start_global - 1, nx, chunk_size + 2, ny);
        CUDA_RT_CALL(hipGetLastError());

        CUDA_RT_CALL(hipDeviceSynchronize());

        CUDA_RT_CALL(hipMemcpy((void *)halo_buffer_for_top_neighbor[dev_id], a[dev_id] + iy_end[dev_id] * nx, nx * sizeof(real), hipMemcpyDeviceToDevice));
        CUDA_RT_CALL(hipMemcpy((void *)halo_buffer_for_bottom_neighbor[dev_id], a[dev_id] , nx * sizeof(real), hipMemcpyDeviceToDevice));

        dim3 dim_grid(grid_dim_x * grid_dim_y + 1);
        dim3 dim_block(dim_block_x, dim_block_y);

        void *kernelArgs[] = {(void *)&a_new[dev_id],
                              (void *)&a[dev_id],
                              (void *)&iy_start,
                              (void *)&iy_end[dev_id],
                              (void *)&nx,
                              (void *)&grid_dim_x,
                              (void *)&iter_max,
                              (void *)&halo_buffer_for_top_neighbor[dev_id],
                              (void *)&halo_buffer_for_bottom_neighbor[dev_id],
                              (void *)&halo_buffer_for_bottom_neighbor[top],
                              (void *)&halo_buffer_for_top_neighbor[bottom],
                              (void *)&is_top_done_computing_flags[dev_id],
                              (void *)&is_bottom_done_computing_flags[dev_id],
                              (void *)&is_bottom_done_computing_flags[top],
                              (void *)&is_top_done_computing_flags[bottom]};

#pragma omp barrier
        double start = omp_get_wtime();

        CUDA_RT_CALL(hipLaunchCooperativeKernel((void *)SSMultiThreadedOneBlockComm::jacobi_kernel,
                                                 dim_grid, dim_block, kernelArgs, 0, nullptr));

        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipDeviceSynchronize());

        // Need to swap pointers on CPU if iteration count is odd
        // Technically, we don't know the iteration number (since we'll be doing l2-norm)
        // Could write iter to CPU when kernel is done
        if (iter_max % 2 == 1)
        {
            std::swap(a_new[dev_id], a[dev_id]);
        }

#pragma omp barrier
        double stop = omp_get_wtime();

        if (compare_to_single_gpu)
        {
            CUDA_RT_CALL(
                hipMemcpy(a_h + iy_start_global * nx, a[dev_id] + nx,
                           std::min((ny - iy_start_global) * nx, chunk_size * nx) * sizeof(real),
                           hipMemcpyDeviceToHost));
        }

#pragma omp barrier

#pragma omp master
        {
            report_results(ny, nx, a_ref_h, a_h, num_devices, runtime_serial_non_persistent, start,
                           stop, compare_to_single_gpu);
        }

        CUDA_RT_CALL(hipFree(a_new[dev_id]));
        CUDA_RT_CALL(hipFree(a[dev_id]));
        CUDA_RT_CALL(hipFree(halo_buffer_for_top_neighbor[dev_id]));
        CUDA_RT_CALL(hipFree(halo_buffer_for_bottom_neighbor[dev_id]));
        CUDA_RT_CALL(hipFree(is_top_done_computing_flags[dev_id]));
        CUDA_RT_CALL(hipFree(is_bottom_done_computing_flags[dev_id]));

        if (compare_to_single_gpu && 0 == dev_id)
        {
            CUDA_RT_CALL(hipHostFree(a_h));
            CUDA_RT_CALL(hipHostFree(a_ref_h));
        }
    }

    return 0;
}
