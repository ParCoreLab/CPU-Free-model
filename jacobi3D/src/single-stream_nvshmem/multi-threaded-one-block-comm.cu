#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <cmath>
#include <cstdio>
#include <iostream>

#include <omp.h>

#include <hip/hip_cooperative_groups.h>

#include <nvshmem.h>
#include <nvshmemx.h>

#include "../../include/single-stream_nvshmem/multi-threaded-one-block-comm.cuh"

namespace cg = cooperative_groups;

namespace SSMultiThreadedOneBlockCommNvshmem
{

    __global__ void __launch_bounds__(1024, 1)
        jacobi_kernel(real *a_new, real *a, const int iz_start, const int iz_end, const int ny,
                      const int nx, const int iter_max, real *halo_buffer_of_top_neighbor,
                      real *halo_buffer_of_bottom_neighbor, uint64_t *is_done_computing_flags, const int top,
                      const int bottom)
    {
        cg::thread_block cta = cg::this_thread_block();
        cg::grid_group grid = cg::this_grid();

        int iter = 0;
        int cur_iter_mod = 0;
        int next_iter_mod = 1;

        const int comm_tile_size_y = blockDim.y * blockDim.z;
        const int comm_tile_size_x = blockDim.x;

        const int comm_base_iy = threadIdx.z * blockDim.y + threadIdx.y;
        const int comm_base_ix = threadIdx.x;

        const int comp_tile_size_x = blockDim.x;
        const int comp_tile_size_y = blockDim.y;
        const int comp_tile_size_z = (gridDim.x - 1) * blockDim.z;

        const int comp_base_iz = blockIdx.x * blockDim.z + threadIdx.z;
        const int comp_base_iy = threadIdx.y;
        const int comp_base_ix = threadIdx.x;

        int iz;
        int iy;
        int ix;

        while (iter < iter_max)
        {
            if (blockIdx.x == gridDim.x - 1)
            {
                nvshmem_quiet();
                nvshmem_uint64_wait_until_all(is_done_computing_flags, 2, NULL, NVSHMEM_CMP_EQ, iter);

                iz = iz_start * ny * nx;
                int iz_below = iz + ny * nx;

                for (iy = (comm_base_iy + 1) * nx; iy < (ny - 1) * nx; iy += comm_tile_size_y * nx)
                {
                    int iy_below = iy + nx;
                    int iy_above = iy - nx;
                    for (ix = (comm_base_ix + 1); ix < (nx - 1); ix += comm_tile_size_x)
                    {
                        const real new_val = (a[iz + iy + ix + 1] + a[iz + iy + ix - 1] + a[iz + iy_below + ix] +
                                              a[iz + iy_above + ix] + a[iz_below + iy + ix] +
                                              halo_buffer_of_top_neighbor[cur_iter_mod * ny * nx + iy + ix]) /
                                             real(6.0);
                        a_new[iz + iy + ix] = new_val;
                    }
                }
                cg::sync(cta);
                nvshmemx_float_put_signal_nbi_block(
                    halo_buffer_of_top_neighbor + next_iter_mod * ny * nx, a_new + iz_start * ny * nx,
                    ny * nx, &(is_done_computing_flags[1]), 1, NVSHMEM_SIGNAL_ADD, top);

                iz = (iz_end - 1) * ny * nx;
                int iz_above = iz - ny * nx;

                for (iy = (comm_base_iy + 1) * nx; iy < (ny - 1) * nx; iy += comm_tile_size_y * nx)
                {
                    int iy_below = iy + nx;
                    int iy_above = iy - nx;

                    for (ix = (comm_base_ix + 1); ix < (nx - 1); ix += comm_tile_size_x)
                    {
                        const real new_val = (a[iz + iy + ix + 1] + a[iz + iy + ix - 1] + a[iz + iy_below + ix] +
                                              a[iz + iy_above + ix] + a[iz_above + iy + ix] +
                                              halo_buffer_of_bottom_neighbor[cur_iter_mod * ny * nx + iy + ix]) /
                                             real(6.0);
                        a_new[iz + iy + ix] = new_val;
                    }
                }
                cg::sync(cta);

                nvshmemx_float_put_signal_nbi_block(
                    halo_buffer_of_bottom_neighbor + next_iter_mod * ny * nx,
                    a_new + (iz_end - 1) * ny * nx, ny * nx, &(is_done_computing_flags[0]), 1,
                    NVSHMEM_SIGNAL_ADD, bottom);
            }
            else
            {

                for (iz = (comp_base_iz + iz_start + 1) * ny * nx; iz < (iz_end - 1) * ny * nx;
                     iz += comp_tile_size_z * ny * nx)
                {
                    int iz_below = iz + ny * nx;
                    int iz_above = iz - ny * nx;
                    for (iy = (comp_base_iy + 1) * nx; iy < (ny - 1) * nx; iy += comp_tile_size_y * nx)
                    {
                        int iy_below = iy + nx;
                        int iy_above = iy - nx;
                        for (ix = (comp_base_ix + 1); ix < (nx - 1); ix += comp_tile_size_x)
                        {
                            const real new_val = (a[iz + iy + ix + 1] + a[iz + iy + ix - 1] +
                                                  a[iz + iy_below + ix] + a[iz + iy_above + ix] +
                                                  a[iz_below + iy + ix] + a[iz_above + iy + ix]) /
                                                 real(6.0);

                            a_new[iz + iy + ix] = new_val;
                        }
                    }
                }
            }

            real *temp_pointer = a_new;
            a_new = a;
            a = temp_pointer;

            iter++;

            next_iter_mod = cur_iter_mod;
            cur_iter_mod = 1 - cur_iter_mod;

            cg::sync(grid);
        }
    }
} // namespace SSMultiThreadedOneBlockCommNvshmem

int SSMultiThreadedOneBlockCommNvshmem::init(int argc, char *argv[])
{
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 512);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 512);
    const int nz = get_argval<int>(argv, argv + argc, "-nz", 512);
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare");

    real *a;
    real *a_new;
    int iz_end;

    real *halo_buffer_for_top_neighbor;
    real *halo_buffer_for_bottom_neighbor;

    uint64_t *is_done_computing_flags;

    real *a_ref_h;
    real *a_h;

    double runtime_serial_non_persistent = 0.0;

    int rank = 0, size = 1;
    MPI_CALL(MPI_Init(&argc, &argv));
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &size));

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));
    int local_rank = -1;
    int local_size = 1;
    {
        MPI_Comm local_comm;
        MPI_CALL(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
                                     &local_comm));

        MPI_CALL(MPI_Comm_rank(local_comm, &local_rank));
        MPI_CALL(MPI_Comm_size(local_comm, &local_size));

        MPI_CALL(MPI_Comm_free(&local_comm));
    }
    if (1 < num_devices && num_devices < local_size)
    {
        fprintf(
            stderr,
            "ERROR Number of visible devices (%d) is less than number of ranks on the node (%d)!\n",
            num_devices, local_size);
        MPI_CALL(MPI_Finalize());
        return 1;
    }
    if (1 == num_devices)
    {
        // Only 1 device visible, assuming GPU affinity is handled via CUDA_VISIBLE_DEVICES
        CUDA_RT_CALL(hipSetDevice(0));
    }
    else
    {
        CUDA_RT_CALL(hipSetDevice(local_rank));
    }
    CUDA_RT_CALL(hipFree(0));
    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr;

    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    constexpr int dim_block_z = 1;

    constexpr int comp_tile_size_x = dim_block_x;
    constexpr int comp_tile_size_y = dim_block_y;

    constexpr int comm_tile_size_x = dim_block_x;
    constexpr int comm_tile_size_y = dim_block_z * dim_block_y;

    constexpr int grid_dim_x = (comp_tile_size_x + dim_block_x - 1) / dim_block_x;
    constexpr int grid_dim_y = (comp_tile_size_y + dim_block_y - 1) / dim_block_y;

    int num_comp_tiles_x = nx / comp_tile_size_x + (nx % comp_tile_size_x != 0);
    int num_comp_tiles_y = ny / comp_tile_size_y + (ny % comp_tile_size_y != 0);

    int num_comm_tiles_x = nx / comm_tile_size_x + (nx % comm_tile_size_x != 0);
    int num_comm_tiles_y = ny / comm_tile_size_y + (ny % comm_tile_size_y != 0);

    int total_num_flags = 2;

    // Set symmetric heap size for nvshmem based on problem size
    // Its default value in nvshmem is 1 GB which is not sufficient
    // for large mesh sizes
    long long unsigned int mesh_size_per_rank = nx * ny * 2 + 1;
    long long unsigned int required_symmetric_heap_size =
        2 * mesh_size_per_rank * sizeof(real) *
        1.1; // Factor 2 is because 2 arrays are allocated - a and a_new
             // 1.1 factor is just for alignment or other usage

    char *value = getenv("NVSHMEM_SYMMETRIC_SIZE");
    if (value)
    { /* env variable is set */
        long long unsigned int size_env = parse_nvshmem_symmetric_size(value);
        if (size_env < required_symmetric_heap_size)
        {
            fprintf(stderr,
                    "ERROR: Minimum NVSHMEM_SYMMETRIC_SIZE = %lluB, Current NVSHMEM_SYMMETRIC_SIZE "
                    "= %s\n",
                    required_symmetric_heap_size, value);
            MPI_CALL(MPI_Finalize());
            return -1;
        }
    }
    else
    {
        char symmetric_heap_size_str[100];
        sprintf(symmetric_heap_size_str, "%llu", required_symmetric_heap_size);
        setenv("NVSHMEM_SYMMETRIC_SIZE", symmetric_heap_size_str, 1);
    }
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

    int npes = nvshmem_n_pes();
    int mype = nvshmem_my_pe();

    nvshmem_barrier_all();

    
    if (compare_to_single_gpu && 0 == mype)
    {
        CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * nz * sizeof(real)));
        CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * nz * sizeof(real)));

        runtime_serial_non_persistent = single_gpu(nz, ny, nx, iter_max, a_ref_h, 0, true);
    }

    nvshmem_barrier_all();

    int chunk_size;
    int chunk_size_low = (nz - 2) / num_devices;
    int chunk_size_high = chunk_size_low + 1;

    int num_ranks_low = num_devices * chunk_size_low + num_devices - (nz - 2);
    if (mype < num_ranks_low)
        chunk_size = chunk_size_low;
    else
        chunk_size = chunk_size_high;

    hipDeviceProp_t deviceProp{};
    CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, mype));
    int numSms = deviceProp.multiProcessorCount;

    int max_thread_blocks_z = (numSms - 1) / (grid_dim_x * grid_dim_y);
    int comp_tile_size_z = dim_block_z * max_thread_blocks_z;
    int num_comp_tiles_z =
        (nz / num_devices) / comp_tile_size_z + ((nz / num_devices) % comp_tile_size_z != 0);

    const int top_pe = mype > 0 ? mype - 1 : (npes - 1);
    const int bottom_pe = (mype + 1) % npes;

    int iy_end_top = (top_pe < num_ranks_low) ? chunk_size_low + 1 : chunk_size_high + 1;
    int iy_start_bottom = 0;

    if (top_pe != mype)
    {
        int canAccessPeer = 0;
        CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, mype, top_pe));
        if (canAccessPeer)
        {
            CUDA_RT_CALL(hipDeviceEnablePeerAccess(top_pe, 0));
        }
        else
        {
            std::cerr << "P2P access required from " << mype << " to " << top_pe << std::endl;
        }
        if (top_pe != bottom_pe)
        {
            canAccessPeer = 0;
            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, mype, bottom_pe));
            if (canAccessPeer)
            {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom_pe, 0));
            }
            else
            {
                std::cerr << "P2P access required from " << mype << " to " << bottom_pe << std::endl;
            }
        }
    }

    nvshmem_barrier_all();

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * (chunk_size + 2) * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * (chunk_size + 2) * sizeof(real)));

    // a = (real *)nvshmem_malloc(nx * ny * (chunk_size + 2) * sizeof(real));
    // a_new = (real *)nvshmem_malloc(nx * ny * (chunk_size + 2) * sizeof(real));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * (chunk_size + 2) * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * (chunk_size + 2) * sizeof(real)));

    halo_buffer_for_top_neighbor = (real *)nvshmem_malloc(2 * nx * ny * sizeof(real));
    halo_buffer_for_bottom_neighbor = (real *)nvshmem_malloc(2 * nx * ny * sizeof(real));

    // CUDA_RT_CALL(hipMalloc(halo_buffer_for_top_neighbor + dev_id, 2 * nx * ny * sizeof(real)));
    // CUDA_RT_CALL(hipMalloc(halo_buffer_for_bottom_neighbor + dev_id, 2 * nx * ny *
    // sizeof(real)));

    CUDA_RT_CALL(hipMemset(halo_buffer_for_top_neighbor, 0, 2 * nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset(halo_buffer_for_bottom_neighbor, 0, 2 * nx * ny * sizeof(real)));

    is_done_computing_flags = (uint64_t *)nvshmem_malloc(total_num_flags * sizeof(uint64_t));
    CUDA_RT_CALL(hipMemset(halo_buffer_for_top_neighbor, 0, 2 * nx * ny * sizeof(real)));
    // CUDA_RT_CALL(hipMalloc(is_top_done_computing_flags + dev_id, total_num_flags *
    // sizeof(int))); CUDA_RT_CALL(hipMalloc(is_bottom_done_computing_flags + dev_id,
    // total_num_flags * sizeof(int)));

    // CUDA_RT_CALL(hipMemset(is_top_done_computing_flags[dev_id], 0, total_num_flags *
    // sizeof(int))); CUDA_RT_CALL(hipMemset(is_bottom_done_computing_flags[dev_id], 0,
    // total_num_flags * sizeof(int)));

    // Calculate local domain boundaries
    int iz_start_global; // My start index in the global array
    if (mype < num_ranks_low)
    {
        iz_start_global = mype * chunk_size_low + 1;
    }
    else
    {
        iz_start_global =
            num_ranks_low * chunk_size_low + (mype - num_ranks_low) * chunk_size_high + 1;
    }
    int iz_end_global = iz_start_global + chunk_size - 1; // My last index in the global array

    int iz_start = 1;
    iz_end = (iz_end_global - iz_start_global + 1) + iz_start;

    initialize_boundaries<<<(nz / num_devices) / 128 + 1, 128>>>(
        a_new, a, PI, iz_start_global - 1, nx, ny, chunk_size + 2, nz);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    dim3 dim_grid(numSms, 1, 1);
    dim3 dim_block(dim_block_x, dim_block_y, dim_block_z);

    void *kernelArgs[] = {(void *)&a_new,
                          (void *)&a,
                          (void *)&iz_start,
                          (void *)&iz_end,
                          (void *)&ny,
                          (void *)&nx,
                          (void *)&iter_max,
                          (void *)&halo_buffer_for_top_neighbor,
                          (void *)&halo_buffer_for_bottom_neighbor,
                          (void *)&is_done_computing_flags,
                          (void *)&top_pe,
                          (void *)&bottom_pe};

    nvshmem_barrier_all();
    double start = MPI_Wtime();

    CUDA_RT_CALL((hipError_t)nvshmemx_collective_launch(
        (void *)SSMultiThreadedOneBlockCommNvshmem::jacobi_kernel, dim_grid, dim_block, kernelArgs,
        0, nullptr));
    // Need to swap pointers on CPU if iteration count is odd
        // Technically, we don't know the iteration number (since we'll be doing
        // l2-norm) Could write iter to CPU when kernel is done
    if (iter_max % 2 == 1)
    {
        std::swap(a_new[mype], a[mype]);
    }

    CUDA_RT_CALL(hipDeviceSynchronize());
    CUDA_RT_CALL(hipGetLastError());


  
    nvshmem_barrier_all();
    double stop = MPI_Wtime();
    nvshmem_barrier_all();
    bool result_correct = true;
    if (compare_to_single_gpu)
    {
        CUDA_RT_CALL(hipMemcpy(
            a_h + iz_start_global * ny * nx, a + ny * nx,
            std::min((nz - iz_start_global) * ny * nx, chunk_size * nx * ny) * sizeof(real),
            hipMemcpyDeviceToHost));

        for (int iz = iz_start_global; result_correct && (iz < iz_end_global); ++iz)
        {
            for (int iy = 1; result_correct && (iy < (ny - 1)); ++iy)
            {
                for (int ix = 1; result_correct && (ix < (nx - 1)); ++ix)
                {
                    if (std::fabs(a_h[iz * ny * nx + iy * nx + ix] -
                                  a_ref_h[iz * ny * nx + iy * nx + ix]) > tol)
                    {
                        fprintf(stderr,
                                "ERROR on rank %d: a[%d * %d + %d * %d + %d] = %f does "
                                "not match %f "
                                "(reference)\n",
                                rank, iz, ny * nx, iy, nx, ix, a_h[iz * ny * nx + iy * nx + ix],
                                a_ref_h[iz * ny * nx + iy * nx + ix]);
                        // result_correct = false;
                    }
                }
            }
        }
    }
    int global_result_correct = 1;
    MPI_CALL(MPI_Allreduce(&result_correct, &global_result_correct, 1, MPI_INT, MPI_MIN,
                           MPI_COMM_WORLD));
    result_correct = global_result_correct;
    if (!mype && result_correct)
    {
        // printf("Num GPUs: %d.\n", num_devices);
        printf("Execution time: %8.4f s\n", (stop - start));

        if (compare_to_single_gpu)
        {
            printf(
                "Non-persistent kernel - %dx%dx%d: 1 GPU: %8.4f s, %d GPUs: "
                "%8.4f "
                "s, speedup: "
                "%8.2f, "
                "efficiency: %8.2f \n",
                nz, ny, nx, runtime_serial_non_persistent, num_devices, (stop - start),
                runtime_serial_non_persistent / (stop - start),
                runtime_serial_non_persistent / (num_devices * (stop - start)) * 100);
        }
    }


    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
    nvshmem_free(halo_buffer_for_top_neighbor);
    nvshmem_free(halo_buffer_for_bottom_neighbor);
    nvshmem_free(is_done_computing_flags);

    if (compare_to_single_gpu && 0 == mype)
    {
        CUDA_RT_CALL(hipHostFree(a_h));
        CUDA_RT_CALL(hipHostFree(a_ref_h));
    }

    nvshmem_finalize();
    MPI_CALL(MPI_Finalize());
    return (result_correct == 1) ? 0 : 1;
}
