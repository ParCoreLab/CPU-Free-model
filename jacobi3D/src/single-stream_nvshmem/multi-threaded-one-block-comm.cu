#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <cmath>
#include <cstdio>
#include <iostream>

#include <omp.h>

#include <hip/hip_cooperative_groups.h>

#include <nvshmem.h>
#include <nvshmemx.h>

#include "../../include/single-stream_nvshmem/multi-threaded-one-block-comm.cuh"

namespace cg = cooperative_groups;

namespace SSMultiThreadedOneBlockCommNvshmem
{

    __global__ void __launch_bounds__(1024, 1)
        jacobi_kernel(real *a_new, real *a, const int iz_start, const int iz_end, const int ny,
                      const int nx, const int iter_max, real *halo_buffer_top,
                      real *halo_buffer_bottom, uint64_t *is_done_computing_flags, const int top,
                      const int bottom)
    {
        cg::thread_block cta = cg::this_thread_block();
        cg::grid_group grid = cg::this_grid();

        int iter = 0;
        int cur_iter_mod = 0;
        int next_iter_mod = 1;

        while (iter < iter_max)
        {
            if (blockIdx.x == gridDim.x - 1)
            {
                if (cta.thread_rank() == 0)
                {
                    nvshmem_signal_wait_until(is_done_computing_flags + cur_iter_mod * 2, NVSHMEM_CMP_EQ, iter);
                }
                cg::sync(cta);
                for (int iy = (threadIdx.z * blockDim.y + threadIdx.y + 1); iy < (ny - 1); iy += blockDim.y * blockDim.z)
                {
                    for (int ix = (threadIdx.x + 1); ix < (nx - 1); ix += blockDim.x)
                    {
                        const real first_row_val = (real(1) / real(6)) * (a[iz_start * ny * nx + iy * nx + ix + 1] +
                                                                          a[iz_start * ny * nx + iy * nx + ix - 1] +
                                                                          a[iz_start * ny * nx + (iy + 1) * nx + ix] +
                                                                          a[iz_start * ny * nx + (iy - 1) * nx + ix] +
                                                                          a[(iz_start + 1) * ny * nx + iy * nx + ix] +
                                                                          halo_buffer_top[cur_iter_mod * ny * nx + iy * nx + ix]);
                        a_new[iz_start * ny * nx + iy * nx + ix] = first_row_val;
                    }
                }
                nvshmemx_putmem_signal_nbi_block(
                    (real *)&halo_buffer_bottom[next_iter_mod * ny * nx], (real *)&a_new[iz_start * ny * nx],
                    ny * nx * sizeof(real), is_done_computing_flags + next_iter_mod * 2 + 1, iter + 1, NVSHMEM_SIGNAL_SET, top);
                if (cta.thread_rank() == 0)
                {
                    nvshmem_signal_wait_until(is_done_computing_flags + cur_iter_mod * 2 + 1, NVSHMEM_CMP_EQ, iter);
                }
                cg::sync(cta);
                for (int iy = (threadIdx.z * blockDim.y + threadIdx.y + 1); iy < (ny - 1); iy += blockDim.y * blockDim.z)
                {
                    for (int ix = (threadIdx.x + 1); ix < (nx - 1); ix += blockDim.x)
                    {

                        const real last_row_val = (real(1) / real(6)) * (a[(iz_end - 1) * ny * nx + iy * nx + ix + 1] +
                                                                         a[(iz_end - 1) * ny * nx + iy * nx + ix - 1] +
                                                                         a[(iz_end - 1) * ny * nx + (iy + 1) * nx + ix] +
                                                                         a[(iz_end - 1) * ny * nx + (iy - 1) * nx + ix] +
                                                                         halo_buffer_bottom[cur_iter_mod * ny * nx + iy * nx + ix]);
                        a_new[(iz_end - 1) * ny * nx + iy * nx + ix] = last_row_val;
                    }
                }

                nvshmemx_putmem_signal_nbi_block(
                    (real *)&halo_buffer_top[next_iter_mod * ny * nx], (real *)&a_new[(iz_end - 1) * ny * nx],
                    ny * nx * sizeof(real), is_done_computing_flags + next_iter_mod * 2, iter + 1, NVSHMEM_SIGNAL_SET, bottom);

                nvshmem_quiet();
            }
            else
            {
                for (int iz = (blockIdx.x * blockDim.z + threadIdx.z + iz_start + 1) * ny * nx;
                     iz < (iz_end - 1) * ny * nx; iz += (gridDim.x - 1) * blockDim.z * ny * nx)
                {
                    for (int iy = (threadIdx.y + 1) * nx; iy < (ny - 1) * nx; iy += blockDim.y * nx)
                    {
                        for (int ix = (threadIdx.x + 1); ix < (nx - 1); ix += blockDim.x)
                        {
                            a_new[iz + iy + ix] = (real(1) / real(6)) *
                                                  (a[iz + iy + ix + 1] + a[iz + iy + ix - 1] + a[iz + iy + nx + ix] +
                                                   a[iz + iy - nx + ix] + a[iz + ny * nx + iy + ix] +
                                                   a[iz - ny * nx + iy + ix]);
                        }
                    }
                }
            }

            real *temp_pointer = a_new;
            a_new = a;
            a = temp_pointer;

            iter++;

            next_iter_mod = cur_iter_mod;
            cur_iter_mod = 1 - cur_iter_mod;
            cg::sync(grid);
        }
    }
} // namespace SSMultiThreadedOneBlockCommNvshmem

int SSMultiThreadedOneBlockCommNvshmem::init(int argc, char *argv[])
{
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 32);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 32);
    const int nz = get_argval<int>(argv, argv + argc, "-nz", 16);
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare");

    real *a;
    real *a_new;

    real *halo_buffer_for_top_neighbor;
    real *halo_buffer_for_bottom_neighbor;

    uint64_t *is_done_computing_flags;

    real *a_ref_h;
    real *a_h;

    double runtime_serial_non_persistent = 0.0;

    int rank = 0, size = 1;
    MPI_CALL(MPI_Init(&argc, &argv));
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &size));

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));
    int local_rank = -1;
    int local_size = 1;
    {
        MPI_Comm local_comm;
        MPI_CALL(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
                                     &local_comm));

        MPI_CALL(MPI_Comm_rank(local_comm, &local_rank));
        MPI_CALL(MPI_Comm_size(local_comm, &local_size));

        MPI_CALL(MPI_Comm_free(&local_comm));
    }
    if (1 < num_devices && num_devices < local_size)
    {
        fprintf(
            stderr,
            "ERROR Number of visible devices (%d) is less than number of ranks on the node (%d)!\n",
            num_devices, local_size);
        MPI_CALL(MPI_Finalize());
        return 1;
    }
    if (1 == num_devices)
    {
        // Only 1 device visible, assuming GPU affinity is handled via CUDA_VISIBLE_DEVICES
        CUDA_RT_CALL(hipSetDevice(0));
    }
    else
    {
        CUDA_RT_CALL(hipSetDevice(local_rank));
    }
    CUDA_RT_CALL(hipFree(0));
    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr;

    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    constexpr int dim_block_z = 1;

    constexpr int comp_tile_size_x = dim_block_x;
    constexpr int comp_tile_size_y = dim_block_y;

    constexpr int comm_tile_size_x = dim_block_x;
    constexpr int comm_tile_size_y = dim_block_z * dim_block_y;

    constexpr int grid_dim_x = (comp_tile_size_x + dim_block_x - 1) / dim_block_x;
    constexpr int grid_dim_y = (comp_tile_size_y + dim_block_y - 1) / dim_block_y;

    int num_comp_tiles_x = nx / comp_tile_size_x + (nx % comp_tile_size_x != 0);
    int num_comp_tiles_y = ny / comp_tile_size_y + (ny % comp_tile_size_y != 0);

    int num_comm_tiles_x = nx / comm_tile_size_x + (nx % comm_tile_size_x != 0);
    int num_comm_tiles_y = ny / comm_tile_size_y + (ny % comm_tile_size_y != 0);

    int total_num_flags = 4;

    // Set symmetric heap size for nvshmem based on problem size
    // Its default value in nvshmem is 1 GB which is not sufficient
    // for large mesh sizes
    long long unsigned int mesh_size_per_rank = nx * ny * 2 + 2;
    long long unsigned int required_symmetric_heap_size =
        2 * mesh_size_per_rank * sizeof(real) *
        1.1; // Factor 2 is because 2 arrays are allocated - a and a_new
             // 1.1 factor is just for alignment or other usage

    char *value = getenv("NVSHMEM_SYMMETRIC_SIZE");
    if (value)
    { /* env variable is set */
        long long unsigned int size_env = parse_nvshmem_symmetric_size(value);
        if (size_env < required_symmetric_heap_size)
        {
            fprintf(stderr,
                    "ERROR: Minimum NVSHMEM_SYMMETRIC_SIZE = %lluB, Current NVSHMEM_SYMMETRIC_SIZE "
                    "= %s\n",
                    required_symmetric_heap_size, value);
            MPI_CALL(MPI_Finalize());
            return -1;
        }
    }
    else
    {
        char symmetric_heap_size_str[100];
        sprintf(symmetric_heap_size_str, "%llu", required_symmetric_heap_size);
        setenv("NVSHMEM_SYMMETRIC_SIZE", symmetric_heap_size_str, 1);
    }
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

    int npes = nvshmem_n_pes();
    int mype = nvshmem_my_pe();

    nvshmem_barrier_all();

    if (compare_to_single_gpu)
    {
        CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * nz * sizeof(real)));
        CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * nz * sizeof(real)));

        runtime_serial_non_persistent = single_gpu(nz, ny, nx, iter_max, a_ref_h, 0, true);
    }

    nvshmem_barrier_all();

    int chunk_size;
    int chunk_size_low = (nz - 2) / npes;
    int chunk_size_high = chunk_size_low + 1;

    int num_ranks_low = npes * chunk_size_low + npes - (nz - 2);
    if (mype < num_ranks_low)
        chunk_size = chunk_size_low;
    else
        chunk_size = chunk_size_high;

    hipDeviceProp_t deviceProp{};
    CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, mype));
    int numSms = deviceProp.multiProcessorCount;

    int max_thread_blocks_z = (numSms - 1) / (grid_dim_x * grid_dim_y);
    int comp_tile_size_z = dim_block_z * max_thread_blocks_z;
    int num_comp_tiles_z =
        (nz / num_devices) / comp_tile_size_z + ((nz / num_devices) % comp_tile_size_z != 0);

    const int top_pe = mype > 0 ? mype - 1 : (npes - 1);
    const int bottom_pe = (mype + 1) % npes;

    if (top_pe != mype)
    {
        int canAccessPeer = 0;
        CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, mype, top_pe));
        if (canAccessPeer)
        {
            CUDA_RT_CALL(hipDeviceEnablePeerAccess(top_pe, 0));
        }
        else
        {
            std::cerr << "P2P access required from " << mype << " to " << top_pe << std::endl;
        }
        if (top_pe != bottom_pe)
        {
            canAccessPeer = 0;
            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, mype, bottom_pe));
            if (canAccessPeer)
            {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom_pe, 0));
            }
            else
            {
                std::cerr << "P2P access required from " << mype << " to " << bottom_pe << std::endl;
            }
        }
    }

    nvshmem_barrier_all();

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * (chunk_size + 2) * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * (chunk_size + 2) * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * (chunk_size + 2) * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * (chunk_size + 2) * sizeof(real)));

    halo_buffer_for_top_neighbor = (real *)nvshmem_malloc(2 * nx * ny * sizeof(real));
    halo_buffer_for_bottom_neighbor = (real *)nvshmem_malloc(2 * nx * ny * sizeof(real));

    CUDA_RT_CALL(hipMemset((void *)halo_buffer_for_top_neighbor, 0, 2 * nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset((void *)halo_buffer_for_bottom_neighbor, 0, 2 * nx * ny * sizeof(real)));

    is_done_computing_flags = (uint64_t *)nvshmem_malloc(total_num_flags * sizeof(uint64_t));
    CUDA_RT_CALL(hipMemset(is_done_computing_flags, 0, total_num_flags * sizeof(uint64_t)));

    // Calculate local domain boundaries
    int iz_start_global; // My start index in the global array
    if (mype < num_ranks_low)
    {
        iz_start_global = mype * chunk_size_low + 1;
    }
    else
    {
        iz_start_global =
            num_ranks_low * chunk_size_low + (mype - num_ranks_low) * chunk_size_high + 1;
    }
    int iz_end_global = iz_start_global + chunk_size - 1; // My last index in the global array

    int iz_start = 1;
    int iz_end = (iz_end_global - iz_start_global + 1) + iz_start;

    initialize_boundaries<<<(nz / npes) / 128 + 1, 128>>>(
        a_new, a, PI, iz_start_global - 1, nx, ny, chunk_size + 2, nz);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    dim3 dim_grid(numSms, 1, 1);
    dim3 dim_block(dim_block_x, dim_block_y, dim_block_z);

    void *kernelArgs[] = {(void *)&a_new,
                          (void *)&a,
                          (void *)&iz_start,
                          (void *)&iz_end,
                          (void *)&ny,
                          (void *)&nx,
                          (void *)&iter_max,
                          (void *)&halo_buffer_for_top_neighbor,
                          (void *)&halo_buffer_for_bottom_neighbor,
                          (void *)&is_done_computing_flags,
                          (void *)&top_pe,
                          (void *)&bottom_pe};

    nvshmem_barrier_all();
    double start = MPI_Wtime();

    CUDA_RT_CALL((hipError_t)nvshmemx_collective_launch(
        (void *)SSMultiThreadedOneBlockCommNvshmem::jacobi_kernel, dim_grid, dim_block, kernelArgs,
        0, nullptr));
    // Need to swap pointers on CPU if iteration count is odd
    // Technically, we don't know the iteration number (since we'll be doing
    // l2-norm) Could write iter to CPU when kernel is done
    if (iter_max % 2 == 1)
    {
        std::swap(a_new, a);
    }

    CUDA_RT_CALL(hipDeviceSynchronize());
    CUDA_RT_CALL(hipGetLastError());

    nvshmem_barrier_all();
    double stop = MPI_Wtime();
    nvshmem_barrier_all();
    bool result_correct = 1;
    if (compare_to_single_gpu)
    {

        CUDA_RT_CALL(hipMemcpy(
            a_h + iz_start_global * ny * nx, a + ny * nx,
            std::min(nz - iz_start_global, chunk_size) * nx * ny * sizeof(real),
            hipMemcpyDeviceToHost));

        for (int iz = iz_start_global; result_correct && (iz <= iz_end_global); ++iz)
        {
            for (int iy = 1; result_correct && (iy < (ny - 1)); ++iy)
            {
                for (int ix = 1; result_correct && (ix < (nx - 1)); ++ix)
                {
                    if (std::fabs(a_h[iz * ny * nx + iy * nx + ix] -
                                  a_ref_h[iz * ny * nx + iy * nx + ix]) > tol)
                    {
                        fprintf(stderr,
                                "ERROR on rank %d: a[%d * %d + %d * %d + %d] = %f does "
                                "not match %f "
                                "(reference)\n",
                                rank, iz, ny * nx, iy, nx, ix, a_h[iz * ny * nx + iy * nx + ix],
                                a_ref_h[iz * ny * nx + iy * nx + ix]);
                        result_correct = 0;
                    }
                }
            }
        }
    }
    int global_result_correct = 1;
    MPI_CALL(MPI_Allreduce(&result_correct, &global_result_correct, 1, MPI_INT, MPI_MIN,
                           MPI_COMM_WORLD));

    if (!mype && global_result_correct)
    {
        // printf("Num GPUs: %d.\n", num_devices);
        printf("Execution time: %8.4f s\n", (stop - start));

        if (compare_to_single_gpu)
        {
            printf(
                "Non-persistent kernel - %dx%dx%d: 1 GPU: %8.4f s, %d GPUs: "
                "%8.4f "
                "s, speedup: "
                "%8.2f, "
                "efficiency: %8.2f \n",
                nz, ny, nx, runtime_serial_non_persistent, npes, (stop - start),
                runtime_serial_non_persistent / (stop - start),
                runtime_serial_non_persistent / (npes * (stop - start)) * 100);
        }
    }

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
    nvshmem_free((void *)halo_buffer_for_top_neighbor);
    nvshmem_free((void *)halo_buffer_for_bottom_neighbor);
    nvshmem_free(is_done_computing_flags);

    if (compare_to_single_gpu)
    {
        CUDA_RT_CALL(hipHostFree(a_h));
        CUDA_RT_CALL(hipHostFree(a_ref_h));
    }

    nvshmem_finalize();
    MPI_CALL(MPI_Finalize());
    return (result_correct == 1) ? 0 : 1;
}
