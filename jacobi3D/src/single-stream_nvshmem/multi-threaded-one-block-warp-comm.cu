#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <cmath>
#include <cstdio>
#include <iostream>

#include <omp.h>

#include <hip/hip_cooperative_groups.h>

#include <nvshmem.h>
#include <nvshmemx.h>
#include "../../include/single-stream_nvshmem/multi-threaded-one-block-warp-comm.cuh"

namespace cg = cooperative_groups;

namespace SSMultiThreadedOneBlockWarpCommNvshmem
{

    __global__ void __launch_bounds__(1024, 1)
        jacobi_kernel(real *a_new, real *a, const int iz_start, const int iz_end, const int ny,
                      const int nx, const int comp_tile_size_x, const int comp_tile_size_y,
                      const int comp_tile_size_z, const int comm_tile_size_x,
                      const int comm_tile_size_y, const int num_comp_tiles_x,
                      const int num_comp_tiles_y, const int num_comp_tiles_z,
                      const int num_comm_tiles_x, const int num_comm_tiles_y, const int iter_max,
                      real *halo_buffer_for_top_neighbor, real *halo_buffer_for_bottom_neighbor,
                      int *is_top_done_computing_flags, int *is_bottom_done_computing_flags,
                      const int top, const int bottom)
    {
        cg::thread_block cta = cg::this_thread_block();
        cg::grid_group grid = cg::this_grid();
        auto warp = cg::tiled_partition<32>(cta);

        int iter = 0;
        int cur_iter_mod = 0;
        int next_iter_mod = 1;

        const int num_flags = 2 * num_comm_tiles_x * num_comm_tiles_y;

        while (iter < iter_max)
        {
            if (blockIdx.x == gridDim.x - 1)
            {
                int cur_iter_comm_tile_flag_idx_x;
                int cur_iter_comm_tile_flag_idx_y;
                int next_iter_comm_tile_flag_idx_x;
                int next_iter_comm_tile_flag_idx_y;
                for (int comm_tile_idx_y = 0; comm_tile_idx_y < num_comm_tiles_y; comm_tile_idx_y++)
                {
                    int comm_tile_start_y =
                        (comm_tile_idx_y == 0) ? 1 : comm_tile_idx_y * comm_tile_size_y;

                    int iy = threadIdx.z * blockDim.y + threadIdx.y + comm_tile_start_y;

                    for (int comm_tile_idx_x = 0; comm_tile_idx_x < num_comm_tiles_x;
                         comm_tile_idx_x++)
                    {
                        int comm_tile_start_x =
                            (comm_tile_idx_x == 0) ? 1 : comm_tile_idx_x * comm_tile_size_x;

                        int ix = threadIdx.x + comm_tile_start_x;

                        if (warp.thread_rank() == 0)
                        {
                            cur_iter_comm_tile_flag_idx_x = comm_tile_idx_x;
                            cur_iter_comm_tile_flag_idx_y = comm_tile_idx_y;
                            nvshmem_int_wait_until(
                                is_top_done_computing_flags + cur_iter_mod * num_flags +
                                    cur_iter_comm_tile_flag_idx_y * num_comm_tiles_x +
                                    cur_iter_comm_tile_flag_idx_x * warp.meta_group_size() +
                                    warp.meta_group_rank(),
                                NVSHMEM_CMP_EQ, iter);

                            /* while (local_is_top_neighbor_done_writing_to_me
                                       [cur_iter_comm_tile_flag_idx_y * num_comm_tiles_x +
                                        cur_iter_comm_tile_flag_idx_x + cur_iter_mod * num_flags] !=
                                   iter) {
                            } */
                        }
                        cg::sync(warp);

                        // copy per row wise (since its warp sized in x dim)
                        if (iy < ny - 1 && ix < nx - 1)
                        {
                            real first_row_val =
                                (a[iz_start * ny * nx + iy * nx + ix + 1] +
                                 a[iz_start * ny * nx + iy * nx + ix - 1] +
                                 a[iz_start * ny * nx + (iy + 1) * nx + ix] +
                                 a[iz_start * ny * nx + (iy - 1) * nx + ix] +
                                 a[(iz_start + 1) * ny * nx + iy * nx + ix] +
                                 // remote_my_halo_buffer_on_top_neighbor[cur_iter_mod * ny * nx +iy *
                                 // nx + ix] +
                                 //
                                 halo_buffer_for_top_neighbor[cur_iter_mod * ny * nx + iy * nx + ix]) /
                                real(6.0);

                            a_new[iz_start * ny * nx + iy * nx + ix] = first_row_val;
                            ///????
                            nvshmemx_float_put_nbi_warp(
                                halo_buffer_for_top_neighbor + next_iter_mod * ny * nx + iy,
                                a_new + iz_start * ny * nx + iy,
                                min(warpSize, nx - 1 - comm_tile_start_x), top);
                            // local_halo_buffer_for_top_neighbor[next_iter_mod * ny * nx + iy * nx +
                            // ix] =
                            //     first_row_val;
                        }

                        cg::sync(warp);

                        if (warp.thread_rank() == 0)
                        {
                            next_iter_comm_tile_flag_idx_x = (num_comm_tiles_x + comm_tile_idx_x);
                            next_iter_comm_tile_flag_idx_y = (comm_tile_idx_y);
                            nvshmem_int_atomic_inc(
                                is_top_done_computing_flags + next_iter_mod * num_flags +
                                    next_iter_comm_tile_flag_idx_y * num_comm_tiles_x +
                                    next_iter_comm_tile_flag_idx_x * warp.meta_group_size() +
                                    warp.meta_group_rank(),
                                top);

                            /*remote_am_done_writing_to_top_neighbor[next_iter_comm_tile_flag_idx_y *
                                                                       num_comm_tiles_x +
                                                                   next_iter_comm_tile_flag_idx_x +
                                                                   next_iter_mod * num_flags] =
                                iter + 1;*/

                            nvshmem_int_wait_until(
                                is_bottom_done_computing_flags + cur_iter_mod * num_flags +
                                    cur_iter_comm_tile_flag_idx_y * num_comm_tiles_x +
                                    cur_iter_comm_tile_flag_idx_x * warp.meta_group_size() +
                                    warp.meta_group_rank(),
                                NVSHMEM_CMP_EQ, iter);

                            /*while (local_is_bottom_neighbor_done_writing_to_me
                                       [cur_iter_comm_tile_flag_idx_y * num_comm_tiles_x +
                                        cur_iter_comm_tile_flag_idx_x + cur_iter_mod * num_flags] !=
                                   iter) {
                            }*/
                        }

                        cg::sync(warp);

                        if (iy < ny - 1 && ix < nx - 1)
                        {
                            const real last_row_val =
                                (a[(iz_end - 1) * ny * nx + iy * nx + ix + 1] +
                                 a[(iz_end - 1) * ny * nx + iy * nx + ix - 1] +
                                 a[(iz_end - 1) * ny * nx + (iy + 1) * nx + ix] +
                                 a[(iz_end - 1) * ny * nx + (iy - 1) * nx + ix] +
                                 halo_buffer_for_bottom_neighbor[cur_iter_mod * ny * nx + iy * nx +
                                                                 ix] +
                                 // remote_my_halo_buffer_on_bottom_neighbor[cur_iter_mod * ny * nx +
                                 //                                          iy * nx + ix] +
                                 a[(iz_end - 2) * ny * nx + iy * nx + ix]) /
                                real(6.0);

                            a_new[(iz_end - 1) * ny * nx + iy * nx + ix] = last_row_val;
                            nvshmemx_float_put_nbi_warp(
                                halo_buffer_for_bottom_neighbor + next_iter_mod * ny * nx + iy,
                                a_new + iz_start * ny * nx + iy,
                                min(warpSize, nx - 1 - comm_tile_start_x), bottom);
                            // local_halo_buffer_for_bottom_neighbor[next_iter_mod * ny * nx + iy * nx +
                            //                                       ix] = last_row_val;
                        }

                        cg::sync(warp);

                        if (warp.thread_rank() == 0)
                        {
                            next_iter_comm_tile_flag_idx_x = comm_tile_idx_x;
                            next_iter_comm_tile_flag_idx_y = comm_tile_idx_y;
                            nvshmem_int_atomic_inc(
                                is_bottom_done_computing_flags + next_iter_mod * num_flags +
                                    next_iter_comm_tile_flag_idx_y * num_comm_tiles_x +
                                    next_iter_comm_tile_flag_idx_x * warp.meta_group_size() +
                                    warp.meta_group_rank(),
                                bottom);
                            /*remote_am_done_writing_to_bottom_neighbor[next_iter_comm_tile_flag_idx_y *
                                                                          num_comm_tiles_x +
                                                                      next_iter_comm_tile_flag_idx_x +
                                                                      next_iter_mod * num_flags] =
                                iter + 1;*/
                        }
                    }
                }
            }
            else
            {
                const int grid_dim_x = (comp_tile_size_x + blockDim.x - 1) / blockDim.x;
                const int grid_dim_y = (comp_tile_size_y + blockDim.y - 1) / blockDim.y;

                const int block_idx_z = blockIdx.x / (grid_dim_x * grid_dim_y);
                const int block_idx_y = (blockIdx.x % (grid_dim_x * grid_dim_y)) / grid_dim_x;
                const int block_idx_x = blockIdx.x % grid_dim_x;

                const int base_iz = block_idx_z * blockDim.z + threadIdx.z;
                const int base_iy = block_idx_y * blockDim.y + threadIdx.y;
                const int base_ix = block_idx_x * blockDim.x + threadIdx.x;
                for (int iz = (base_iz + iz_start + 1) * ny * nx; iz < (iz_end - 1) * ny * nx;
                     iz += comp_tile_size_z * ny * nx)
                {
                    int iz_below = iz + ny * nx;
                    int iz_above = iz - ny * nx;
                    for (int iy = (base_iy + 1) * nx; iy < (ny - 1) * nx; iy += comp_tile_size_y * nx)
                    {
                        int iy_below = iy + nx;
                        int iy_above = iy - nx;
                        for (int ix = (base_ix + 1); ix < (nx - 1); ix += comp_tile_size_x)
                        {
                            // big bottleneck here
                            const real new_val = (a[iz + iy + ix + 1] + a[iz + iy + ix - 1] +
                                                  a[iz + iy_below + ix] + a[iz + iy_above + ix] +
                                                  a[iz_below + iy + ix] + a[iz_above + iy + ix]) /
                                                 real(6.0);

                            a_new[iz + iy + ix] = new_val;
                        }
                    }
                }
            }

            real *temp_pointer = a_new;
            a_new = a;
            a = temp_pointer;

            iter++;

            next_iter_mod = cur_iter_mod;
            cur_iter_mod = 1 - cur_iter_mod;

            cg::sync(grid);
        }
    }
} // namespace SSMultiThreadedOneBlockWarpCommNvshmem

int SSMultiThreadedOneBlockWarpCommNvshmem::init(int argc, char *argv[])
{
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 512);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 512);
    const int nz = get_argval<int>(argv, argv + argc, "-nz", 512);
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare");

    real *a[MAX_NUM_DEVICES];
    real *a_new[MAX_NUM_DEVICES];
    int iz_end[MAX_NUM_DEVICES];

    real *halo_buffer_for_top_neighbor[MAX_NUM_DEVICES];
    real *halo_buffer_for_bottom_neighbor[MAX_NUM_DEVICES];

    int *is_top_done_computing_flags[MAX_NUM_DEVICES];
    int *is_bottom_done_computing_flags[MAX_NUM_DEVICES];

    real *a_ref_h;
    real *a_h;

    double runtime_serial_non_persistent = 0.0;

    int rank = 0, size = 1;
    MPI_CALL(MPI_Init(&argc, &argv));
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &size));

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    int local_rank = -1;
    int local_size = 1;
    {
        MPI_Comm local_comm;
        MPI_CALL(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
                                     &local_comm));

        MPI_CALL(MPI_Comm_rank(local_comm, &local_rank));
        MPI_CALL(MPI_Comm_size(local_comm, &local_size));

        MPI_CALL(MPI_Comm_free(&local_comm));
    }
    if (1 < num_devices && num_devices < local_size)
    {
        fprintf(
            stderr,
            "ERROR Number of visible devices (%d) is less than number of ranks on the node (%d)!\n",
            num_devices, local_size);
        MPI_CALL(MPI_Finalize());
        return 1;
    }
    if (1 == num_devices)
    {
        // Only 1 device visible, assuming GPU affinity is handled via CUDA_VISIBLE_DEVICES
        CUDA_RT_CALL(hipSetDevice(0));
    }
    else
    {
        CUDA_RT_CALL(hipSetDevice(local_rank));
    }

    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr;

    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    constexpr int dim_block_z = 1;

    constexpr int comp_tile_size_x = dim_block_x;
    constexpr int comp_tile_size_y = dim_block_y;
    constexpr int comp_tile_size_z = dim_block_z;

    constexpr int comm_tile_size_x = dim_block_x;
    constexpr int comm_tile_size_y = dim_block_z * dim_block_y;

    constexpr int grid_dim_x = (comp_tile_size_x + dim_block_x - 1) / dim_block_x;
    constexpr int grid_dim_y = (comp_tile_size_y + dim_block_y - 1) / dim_block_y;

    
    int num_comp_tiles_x = nx / comp_tile_size_x + (nx % comp_tile_size_x != 0);
    int num_comp_tiles_y = ny / comp_tile_size_y + (ny % comp_tile_size_y != 0);

    int num_comm_tiles_x = nx / comm_tile_size_x + (nx % comm_tile_size_x != 0);
    int num_comm_tiles_y = ny / comm_tile_size_y + (ny % comm_tile_size_y != 0);

    int total_num_flags = 4 * num_comm_tiles_x * dim_block_y * num_comm_tiles_y;

    // Set symmetric heap size for nvshmem based on problem size
    // Its default value in nvshmem is 1 GB which is not sufficient
    // for large mesh sizes
    long long unsigned int mesh_size_per_rank = 2 * nx * ny + total_num_flags;
    long long unsigned int required_symmetric_heap_size =
        2 * mesh_size_per_rank * sizeof(real) *
        1.1; // Factor 2 is because 2 arrays are allocated - a and a_new
             // 1.1 factor is just for alignment or other usage

    char *value = getenv("NVSHMEM_SYMMETRIC_SIZE");
    if (value)
    { /* env variable is set */
        long long unsigned int size_env = parse_nvshmem_symmetric_size(value);
        if (size_env < required_symmetric_heap_size)
        {
            fprintf(stderr,
                    "ERROR: Minimum NVSHMEM_SYMMETRIC_SIZE = %lluB, Current NVSHMEM_SYMMETRIC_SIZE "
                    "= %s\n",
                    required_symmetric_heap_size, value);
            MPI_CALL(MPI_Finalize());
            return -1;
        }
    }
    else
    {
        char symmetric_heap_size_str[100];
        sprintf(symmetric_heap_size_str, "%llu", required_symmetric_heap_size);
        setenv("NVSHMEM_SYMMETRIC_SIZE", symmetric_heap_size_str, 1);
    }
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

    int npes = nvshmem_n_pes();
    int mype = nvshmem_my_pe();

    nvshmem_barrier_all();

    bool result_correct = true;
    if (compare_to_single_gpu && 0 == mype)
    {
        CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * nz * sizeof(real)));
        CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * nz * sizeof(real)));

        runtime_serial_non_persistent = single_gpu(nz, ny, nx, iter_max, a_ref_h, 0, true);
    }

    nvshmem_barrier_all();

    int chunk_size;
    int chunk_size_low = (nz - 2) / num_devices;
    int chunk_size_high = chunk_size_low + 1;

    int num_ranks_low = num_devices * chunk_size_low + num_devices - (nz - 2);
    if (mype < num_ranks_low)
        chunk_size = chunk_size_low;
    else
        chunk_size = chunk_size_high;

    hipDeviceProp_t deviceProp{};
    CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, mype));
    int numSms = deviceProp.multiProcessorCount;

    int num_comp_tiles_z =
        (nz / num_devices) / comp_tile_size_z + ((nz / num_devices) % comp_tile_size_z != 0);
    int max_thread_blocks_z = (numSms - 1) / (grid_dim_x * grid_dim_y);
    int comp_tile_size_z = dim_block_z * max_thread_blocks_z;
    
    
    const int top = mype > 0 ? mype - 1 : (num_devices - 1);
    const int bottom = (mype + 1) % num_devices;

    if (top != mype)
    {
        int canAccessPeer = 0;
        CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, mype, top));
        if (canAccessPeer)
        {
            CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
        }
        else
        {
            std::cerr << "P2P access required from " << mype << " to " << top << std::endl;
        }
        if (top != bottom)
        {
            canAccessPeer = 0;
            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, mype, bottom));
            if (canAccessPeer)
            {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom, 0));
            }
            else
            {
                std::cerr << "P2P access required from " << mype << " to " << bottom << std::endl;
            }
        }
    }

    nvshmem_barrier_all();

    CUDA_RT_CALL(hipMalloc(a + mype, nx * ny * (chunk_size + 2) * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(a_new + mype, nx * ny * (chunk_size + 2) * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a[mype], 0, nx * ny * (chunk_size + 2) * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new[mype], 0, nx * ny * (chunk_size + 2) * sizeof(real)));

    halo_buffer_for_top_neighbor[mype] = (real *)nvshmem_calloc(2 * nx * ny, sizeof(real));
    halo_buffer_for_bottom_neighbor[mype] = (real *)nvshmem_calloc(2 * nx * ny, sizeof(real));

    // CUDA_RT_CALL(hipMalloc(halo_buffer_for_top_neighbor + dev_id, 2 * nx * ny * sizeof(real)));
    // CUDA_RT_CALL(hipMalloc(halo_buffer_for_bottom_neighbor + dev_id, 2 * nx * ny *
    // sizeof(real)));

    // CUDA_RT_CALL(hipMemset(halo_buffer_for_top_neighbor[dev_id], 0, 2 * nx * ny *
    // sizeof(real))); CUDA_RT_CALL(hipMemset(halo_buffer_for_bottom_neighbor[dev_id], 0, 2 * nx *
    // ny * sizeof(real)));

    is_top_done_computing_flags[mype] = (int *)nvshmem_calloc(total_num_flags, sizeof(int));
    is_bottom_done_computing_flags[mype] = (int *)nvshmem_calloc(total_num_flags, sizeof(int));

    // CUDA_RT_CALL(hipMalloc(is_top_done_computing_flags + dev_id, total_num_flags *
    // sizeof(int))); CUDA_RT_CALL(hipMalloc(is_bottom_done_computing_flags + dev_id,
    // total_num_flags * sizeof(int)));

    // CUDA_RT_CALL(hipMemset(is_top_done_computing_flags[dev_id], 0, total_num_flags *
    // sizeof(int))); CUDA_RT_CALL(hipMemset(is_bottom_done_computing_flags[dev_id], 0,
    // total_num_flags * sizeof(int)));

    // Calculate local domain boundaries
    int iz_start_global; // My start index in the global array
    if (mype < num_ranks_low)
    {
        iz_start_global = mype * chunk_size_low + 1;
    }
    else
    {
        iz_start_global =
            num_ranks_low * chunk_size_low + (mype - num_ranks_low) * chunk_size_high + 1;
    }
    int iz_end_global = iz_start_global + chunk_size - 1; // My last index in the global array

    int iz_start = 1;
    iz_end[mype] = (iz_end_global - iz_start_global + 1) + iz_start;

    initialize_boundaries<<<(nz / num_devices) / 128 + 1, 128>>>(
        a_new[mype], a[mype], PI, iz_start_global - 1, nx, ny, chunk_size + 2, nz);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    dim3 dim_grid(numSms, 1, 1);
    dim3 dim_block(dim_block_x, dim_block_y, dim_block_z);

    void *kernelArgs[] = {(void *)&a_new[mype],
                          (void *)&a[mype],
                          (void *)&iz_start,
                          (void *)&iz_end[mype],
                          (void *)&ny,
                          (void *)&nx,
                          (void *)&comp_tile_size_x,
                          (void *)&comp_tile_size_y,
                          (void *)&comp_tile_size_z,
                          (void *)&comm_tile_size_x,
                          (void *)&comm_tile_size_y,
                          (void *)&num_comp_tiles_x,
                          (void *)&num_comp_tiles_y,
                          (void *)&num_comp_tiles_z,
                          (void *)&num_comm_tiles_x,
                          (void *)&num_comm_tiles_y,
                          (void *)&iter_max,
                          (void *)&halo_buffer_for_top_neighbor,
                          (void *)&halo_buffer_for_bottom_neighbor,
                          (void *)&is_top_done_computing_flags,
                          (void *)&is_bottom_done_computing_flags,
                          (void *)&top,
                          (void *)&bottom};

    nvshmem_barrier_all();
    double start = MPI_Wtime();

    CUDA_RT_CALL((hipError_t)nvshmemx_collective_launch(
        (void *)SSMultiThreadedOneBlockWarpCommNvshmem::jacobi_kernel, dim_grid, dim_block,
        kernelArgs, 0, nullptr));

    CUDA_RT_CALL(hipDeviceSynchronize());
    CUDA_RT_CALL(hipGetLastError());

    // Need to swap pointers on CPU if iteration count is odd
    // Technically, we don't know the iteration number (since we'll be doing
    // l2-norm) Could write iter to CPU when kernel is done
    if (iter_max % 2 == 1)
    {
        std::swap(a_new[mype], a[mype]);
    }

    nvshmem_barrier_all();
    double stop = MPI_Wtime();
    nvshmem_barrier_all();
    if (compare_to_single_gpu)
    {
        CUDA_RT_CALL(hipMemcpy(
            a_h + iz_start_global * ny * nx, a[mype] + ny * nx,
            std::min((nz - iz_start_global) * ny * nx, chunk_size * nx * ny) * sizeof(real),
            hipMemcpyDeviceToHost));

        for (int iz = 1; result_correct && (iz < (nz - 1)); ++iz)
        {
            for (int iy = 1; result_correct && (iy < (ny - 1)); ++iy)
            {
                for (int ix = 1; result_correct && (ix < (nx - 1)); ++ix)
                {
                    if (std::fabs(a_h[iz * ny * nx + iy * nx + ix] -
                                  a_ref_h[iz * ny * nx + iy * nx + ix]) > tol)
                    {
                        fprintf(stderr,
                                "ERROR on rank %d: a[%d * %d + %d * %d + %d] = %f does "
                                "not match %f "
                                "(reference)\n",
                                rank, iz, ny * nx, iy, nx, ix, a_h[iz * ny * nx + iy * nx + ix],
                                a_ref_h[iz * ny * nx + iy * nx + ix]);
                        // result_correct = false;
                    }
                }
            }
        }
        if (result_correct)
        {
            // printf("Num GPUs: %d.\n", num_devices);
            printf("Execution time: %8.4f s\n", (stop - start));

            if (compare_to_single_gpu)
            {
                printf(
                    "Non-persistent kernel - %dx%dx%d: 1 GPU: %8.4f s, %d GPUs: "
                    "%8.4f "
                    "s, speedup: "
                    "%8.2f, "
                    "efficiency: %8.2f \n",
                    nz, ny, nx, runtime_serial_non_persistent, num_devices, (stop - start),
                    runtime_serial_non_persistent / (stop - start),
                    runtime_serial_non_persistent / (num_devices * (stop - start)) * 100);
            }
        }
    }

    int global_result_correct = 1;
    MPI_CALL(MPI_Allreduce(&result_correct, &global_result_correct, 1, MPI_INT, MPI_MIN,
                           MPI_COMM_WORLD));
    result_correct = global_result_correct;

    CUDA_RT_CALL(hipFree(a_new[mype]));
    CUDA_RT_CALL(hipFree(a[mype]));
    nvshmem_free(halo_buffer_for_top_neighbor[mype]);
    nvshmem_free(halo_buffer_for_bottom_neighbor[mype]);
    nvshmem_free(is_top_done_computing_flags[mype]);
    nvshmem_free(is_bottom_done_computing_flags[mype]);

    if (compare_to_single_gpu && 0 == mype)
    {
        CUDA_RT_CALL(hipHostFree(a_h));
        CUDA_RT_CALL(hipHostFree(a_ref_h));
    }

    nvshmem_finalize();
    MPI_CALL(MPI_Finalize());
    return 0;
}
