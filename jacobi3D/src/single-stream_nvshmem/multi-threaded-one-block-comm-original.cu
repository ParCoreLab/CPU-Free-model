#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <cmath>
#include <cstdio>
#include <iostream>

#include "../../include/single-stream_nvshmem/multi-threaded-one-block-comm-original.cuh"
#include <hip/hip_cooperative_groups.h>

#include <nvshmem.h>
#include <nvshmemx.h>

namespace cg = cooperative_groups;

namespace SSMultiThreadedOneBlockCommOriginalNvshmem
{
    __global__ void __launch_bounds__(1024, 1)
        jacobi_kernel(real *a_new, real *a, const int iz_start, const int iz_end,
                      const int ny, const int nx, const int iter_max,
                      real *halo_buffer_top, real *halo_buffer_bottom,
                      uint64_t *is_done_computing_flags, const int top,
                      const int bottom)
    {
        cg::thread_block cta = cg::this_thread_block();
        cg::grid_group grid = cg::this_grid();
        auto warp = cg::tiled_partition<32>(cta);

        int iter = 0;
        int cur_iter_mod = 0;
        int next_iter_mod = 1;

        const int num_comm_tiles_x = nx / blockDim.x + (nx % blockDim.x != 0);
        const int num_comm_tiles_y = ny / (blockDim.y * blockDim.z) + (ny % (blockDim.y * blockDim.z) != 0);
        const int num_flags = 2 * num_comm_tiles_x * num_comm_tiles_y * warp.meta_group_size();
        while (iter < iter_max)
        {
            if (blockIdx.x == gridDim.x - 1)
            {
                int iy = threadIdx.z * blockDim.y + threadIdx.y + 1;
                for (int comm_tile_idx_y = 0; comm_tile_idx_y < num_comm_tiles_y;
                     comm_tile_idx_y++, iy += blockDim.y * blockDim.z)
                {
                    int ix = threadIdx.x + 1;
                    for (int comm_tile_idx_x = 0; comm_tile_idx_x < num_comm_tiles_x;
                         comm_tile_idx_x++, ix += blockDim.x)
                    {
                        if (cta.thread_rank() == 0)
                        {
                            nvshmem_signal_wait_until(
                                is_done_computing_flags + cur_iter_mod * num_flags +
                                    comm_tile_idx_y * num_comm_tiles_x + comm_tile_idx_x,
                                NVSHMEM_CMP_EQ, iter);
                        }
                        cg::sync(cta);

                        if (iy < ny - 1 && ix < nx - 1)
                        {
                            const real first_row_val = (real(1) / real(6)) * (a[iz_start * ny * nx + iy * nx + ix + 1] +
                                                                              a[iz_start * ny * nx + iy * nx + ix - 1] +
                                                                              a[iz_start * ny * nx + (iy + 1) * nx + ix] +
                                                                              a[iz_start * ny * nx + (iy - 1) * nx + ix] +
                                                                              a[(iz_start + 1) * ny * nx + iy * nx + ix] +
                                                                              nvshmem_float_g(halo_buffer_bottom + cur_iter_mod * ny * nx + iy * nx + ix, top));
                            a_new[iz_start * ny * nx + iy * nx + ix] = first_row_val;
                            halo_buffer_top[next_iter_mod * ny * nx + iy * nx + ix] = first_row_val;
                        }
                        cg::sync(cta);
                        if (cta.thread_rank() == 0)
                        {
                            nvshmem_fence();
                            nvshmemx_signal_op(
                                is_done_computing_flags + next_iter_mod * num_flags + num_comm_tiles_x * num_comm_tiles_y +
                                    comm_tile_idx_y * num_comm_tiles_x + comm_tile_idx_x,
                                iter + 1, NVSHMEM_SIGNAL_SET, top);

                            nvshmem_signal_wait_until(
                                is_done_computing_flags + cur_iter_mod * num_flags + num_comm_tiles_x * num_comm_tiles_y +
                                    comm_tile_idx_y * num_comm_tiles_x + comm_tile_idx_x,
                                NVSHMEM_CMP_EQ, iter);
                        }
                        cg::sync(cta);

                        if (iy < ny - 1 && ix < nx - 1)
                        {
                            const real last_row_val = (real(1) / real(6)) * (a[(iz_end - 1) * ny * nx + iy * nx + ix + 1] +
                                                                             a[(iz_end - 1) * ny * nx + iy * nx + ix - 1] +
                                                                             a[(iz_end - 1) * ny * nx + (iy + 1) * nx + ix] +
                                                                             a[(iz_end - 1) * ny * nx + (iy - 1) * nx + ix] +
                                                                             nvshmem_float_g(halo_buffer_top + cur_iter_mod * ny * nx + iy * nx + ix, bottom) +
                                                                             a[(iz_end - 2) * ny * nx + iy * nx + ix]);
                            a_new[(iz_end - 1) * ny * nx + iy * nx + ix] = last_row_val;
                            halo_buffer_bottom[next_iter_mod * ny * nx + iy * nx + ix] = last_row_val;
                        }
                        cg::sync(cta);
                        if (cta.thread_rank() == 0)
                        {
                            nvshmem_fence();
                            nvshmemx_signal_op(
                                is_done_computing_flags + next_iter_mod * num_flags +
                                    comm_tile_idx_y * num_comm_tiles_x + comm_tile_idx_x,
                                iter + 1, NVSHMEM_SIGNAL_SET, bottom);
                        }
                    }
                }
            }
            else
            {
                for (int iz = (blockIdx.x * blockDim.z + threadIdx.z + iz_start + 1) * ny * nx;
                     iz < (iz_end - 1) * ny * nx; iz += (gridDim.x - 1) * blockDim.z * ny * nx)
                {
                    int iz_below = iz + ny * nx;
                    int iz_above = iz - ny * nx;
                    for (int iy = (threadIdx.y + 1) * nx; iy < (ny - 1) * nx; iy += blockDim.y * nx)
                    {
                        int iy_below = iy + nx;
                        int iy_above = iy - nx;
                        for (int ix = (threadIdx.x + 1); ix < (nx - 1); ix += blockDim.x)
                        {
                            const real new_val = (real(1) / real(6)) * (a[iz + iy + ix + 1] + a[iz + iy + ix - 1] +
                                                                        a[iz + iy_below + ix] + a[iz + iy_above + ix] +
                                                                        a[iz_below + iy + ix] + a[iz_above + iy + ix]);

                            a_new[iz + iy + ix] = new_val;
                        }
                    }
                }
            }

            real *temp_pointer = a_new;
            a_new = a;
            a = temp_pointer;

            iter++;

            next_iter_mod = cur_iter_mod;
            cur_iter_mod = 1 - cur_iter_mod;
            if (grid.thread_rank() == 0)
            {
                nvshmem_quiet();
            }
            cg::sync(grid);
        }
    }
} // namespace SSMultiThreadedOneBlockCommOriginalNvshmem

int SSMultiThreadedOneBlockCommOriginalNvshmem::init(int argc, char *argv[])
{
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 512);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 512);
    const int nz = get_argval<int>(argv, argv + argc, "-nz", 512);
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare");

    real *a;
    real *a_new;

    real *halo_buffer_top;
    real *halo_buffer_bottom;

    uint64_t *is_done_computing_flags;

    real *a_ref_h;
    real *a_h;

    double runtime_serial_non_persistent = 0.0;

    int rank = 0, size = 1;
    MPI_CALL(MPI_Init(&argc, &argv));
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &size));

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    int local_rank = -1;
    int local_size = 1;
    {
        MPI_Comm local_comm;
        MPI_CALL(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
                                     &local_comm));

        MPI_CALL(MPI_Comm_rank(local_comm, &local_rank));
        MPI_CALL(MPI_Comm_size(local_comm, &local_size));

        MPI_CALL(MPI_Comm_free(&local_comm));
    }
    if (1 < num_devices && num_devices < local_size)
    {
        fprintf(
            stderr,
            "ERROR Number of visible devices (%d) is less than number of ranks on the node (%d)!\n",
            num_devices, local_size);
        MPI_CALL(MPI_Finalize());
        return 1;
    }
    if (1 == num_devices)
    {
        // Only 1 device visible, assuming GPU affinity is handled via CUDA_VISIBLE_DEVICES
        CUDA_RT_CALL(hipSetDevice(0));
    }
    else
    {
        CUDA_RT_CALL(hipSetDevice(local_rank));
    }

    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr;

    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    constexpr int dim_block_z = 1;

    // constexpr int comp_tile_size_x = dim_block_x;
    // constexpr int comp_tile_size_y = dim_block_y;

    constexpr int comm_tile_size_x = dim_block_x;
    constexpr int comm_tile_size_y = dim_block_z * dim_block_y;

    // constexpr int grid_dim_x = (comp_tile_size_x + dim_block_x - 1) / dim_block_x;
    // constexpr int grid_dim_y = (comp_tile_size_y + dim_block_y - 1) / dim_block_y;

    // int num_comp_tiles_x = nx / comp_tile_size_x + (nx % comp_tile_size_x != 0);
    // int num_comp_tiles_y = ny / comp_tile_size_y + (ny % comp_tile_size_y != 0);

    int num_comm_tiles_x = nx / comm_tile_size_x + (nx % comm_tile_size_x != 0);
    int num_comm_tiles_y = ny / comm_tile_size_y + (ny % comm_tile_size_y != 0);

    int total_num_flags = 4 * num_comm_tiles_x * dim_block_y * num_comm_tiles_y;

    // Set symmetric heap size for nvshmem based on problem size
    // Its default value in nvshmem is 1 GB which is not sufficient
    // for large mesh sizes
    long long unsigned int mesh_size_per_rank = 2 * nx * ny + 2 * num_comm_tiles_x * dim_block_y * num_comm_tiles_y;
    long long unsigned int required_symmetric_heap_size =
        2 * mesh_size_per_rank * sizeof(real) *
        1.1; // Factor 2 is because 2 arrays are allocated - a and a_new
             // 1.1 factor is just for alignment or other usage

    char *value = getenv("NVSHMEM_SYMMETRIC_SIZE");
    if (value)
    { /* env variable is set */
        long long unsigned int size_env = parse_nvshmem_symmetric_size(value);
        if (size_env < required_symmetric_heap_size)
        {
            fprintf(stderr,
                    "ERROR: Minimum NVSHMEM_SYMMETRIC_SIZE = %lluB, Current NVSHMEM_SYMMETRIC_SIZE "
                    "= %s\n",
                    required_symmetric_heap_size, value);
            MPI_CALL(MPI_Finalize());
            return -1;
        }
    }
    else
    {
        char symmetric_heap_size_str[100];
        sprintf(symmetric_heap_size_str, "%llu", required_symmetric_heap_size);
        setenv("NVSHMEM_SYMMETRIC_SIZE", symmetric_heap_size_str, 1);
    }
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

    int npes = nvshmem_n_pes();
    int mype = nvshmem_my_pe();

    nvshmem_barrier_all();

    if (compare_to_single_gpu)
    {
        CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * nz * sizeof(real)));
        CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * nz * sizeof(real)));

        runtime_serial_non_persistent = single_gpu(nz, ny, nx, iter_max, a_ref_h, 0, true);
    }

    nvshmem_barrier_all();

    int chunk_size;
    int chunk_size_low = (nz - 2) / npes;
    int chunk_size_high = chunk_size_low + 1;

    int num_ranks_low = npes * chunk_size_low + npes - (nz - 2);
    if (mype < num_ranks_low)
        chunk_size = chunk_size_low;
    else
        chunk_size = chunk_size_high;

    hipDeviceProp_t deviceProp{};
    CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, mype));
    int numSms = deviceProp.multiProcessorCount;

    // int max_thread_blocks_z = (numSms - 1) / (grid_dim_x * grid_dim_y);
    // int comp_tile_size_z = dim_block_z * max_thread_blocks_z;
    // int num_comp_tiles_z = (nz / npes) / comp_tile_size_z + ((nz / npes) % comp_tile_size_z != 0);

    const int top = mype > 0 ? mype - 1 : (npes - 1);
    const int bottom = (mype + 1) % npes;

    if (top != mype)
    {
        int canAccessPeer = 0;
        CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, mype, top));
        if (canAccessPeer)
        {
            CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
        }
        else
        {
            std::cerr << "P2P access required from " << mype << " to " << top << std::endl;
        }
        if (top != bottom)
        {
            canAccessPeer = 0;
            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, mype, bottom));
            if (canAccessPeer)
            {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom, 0));
            }
            else
            {
                std::cerr << "P2P access required from " << mype << " to " << bottom << std::endl;
            }
        }
    }

    nvshmem_barrier_all();

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * (chunk_size + 2) * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * (chunk_size + 2) * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * (chunk_size + 2) * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * (chunk_size + 2) * sizeof(real)));

    halo_buffer_top = (real *)nvshmem_malloc(2 * nx * ny * sizeof(real));
    halo_buffer_bottom = (real *)nvshmem_malloc(2 * nx * ny * sizeof(real));

    CUDA_RT_CALL(hipMemset((void *)halo_buffer_top, 0, 2 * nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset((void *)halo_buffer_bottom, 0, 2 * nx * ny * sizeof(real)));

    is_done_computing_flags = (uint64_t *)nvshmem_malloc(total_num_flags * sizeof(uint64_t));
    CUDA_RT_CALL(hipMemset(is_done_computing_flags, 0, total_num_flags * sizeof(uint64_t)));

    // Calculate local domain boundaries
    int iz_start_global; // My start index in the global array
    if (mype < num_ranks_low)
    {
        iz_start_global = mype * chunk_size_low + 1;
    }
    else
    {
        iz_start_global =
            num_ranks_low * chunk_size_low + (mype - num_ranks_low) * chunk_size_high + 1;
    }
    int iz_end_global = iz_start_global + chunk_size - 1; // My last index in the global array

    int iz_start = 1;
    int iz_end = (iz_end_global - iz_start_global + 1) + iz_start;

    initialize_boundaries<<<(nz / npes) / 128 + 1, 128>>>(
        a_new, a, PI, iz_start_global - 1, nx, ny, chunk_size + 2, nz);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    dim3 dim_grid(numSms, 1, 1);
    dim3 dim_block(dim_block_x, dim_block_y, dim_block_z);

    void *kernelArgs[] = {(void *)&a_new,
                          (void *)&a,
                          (void *)&iz_start,
                          (void *)&iz_end,
                          (void *)&ny,
                          (void *)&nx,
                          (void *)&iter_max,
                          (void *)&halo_buffer_top,
                          (void *)&halo_buffer_bottom,
                          (void *)&is_done_computing_flags,
                          (void *)&top,
                          (void *)&bottom};

    nvshmem_barrier_all();
    double start = MPI_Wtime();

    CUDA_RT_CALL((hipError_t)nvshmemx_collective_launch(
        (void *)SSMultiThreadedOneBlockCommOriginalNvshmem::jacobi_kernel, dim_grid, dim_block,
        kernelArgs, 0, nullptr));

    CUDA_RT_CALL(hipDeviceSynchronize());
    CUDA_RT_CALL(hipGetLastError());

    // Need to swap pointers on CPU if iteration count is odd
    // Technically, we don't know the iteration number (since we'll be doing
    // l2-norm) Could write iter to CPU when kernel is done
    if (iter_max % 2 == 1)
    {
        std::swap(a_new, a);
    }

    nvshmem_barrier_all();

    double stop = MPI_Wtime();

    nvshmem_barrier_all();

    bool result_correct = true;
    if (compare_to_single_gpu)
    {

        CUDA_RT_CALL(hipMemcpy(
            a_h + iz_start_global * ny * nx, a + ny * nx,
            std::min(nz - iz_start_global, chunk_size) * nx * ny * sizeof(real),
            hipMemcpyDeviceToHost));

        for (int iz = iz_start_global; result_correct && (iz <= iz_end_global); ++iz)
        {
            for (int iy = 1; result_correct && (iy < (ny - 1)); ++iy)
            {
                for (int ix = 1; result_correct && (ix < (nx - 1)); ++ix)
                {
                    if (std::fabs(a_h[iz * ny * nx + iy * nx + ix] -
                                  a_ref_h[iz * ny * nx + iy * nx + ix]) > tol)
                    {
                        fprintf(stderr,
                                "ERROR on rank %d: a[%d * %d + %d * %d + %d] = %f does "
                                "not match %f "
                                "(reference)\n",
                                rank, iz, ny * nx, iy, nx, ix, a_h[iz * ny * nx + iy * nx + ix],
                                a_ref_h[iz * ny * nx + iy * nx + ix]);
                        // result_correct = 0;
                    }
                }
            }
        }
    }
    int global_result_correct = 1;
    MPI_CALL(MPI_Allreduce(&result_correct, &global_result_correct, 1, MPI_INT, MPI_MIN,
                           MPI_COMM_WORLD));

    if (!mype && global_result_correct)
    {
        // printf("Num GPUs: %d.\n", num_devices);
        printf("Execution time: %8.4f s\n", (stop - start));

        if (compare_to_single_gpu)
        {
            printf(
                "Non-persistent kernel - %dx%dx%d: 1 GPU: %8.4f s, %d GPUs: "
                "%8.4f "
                "s, speedup: "
                "%8.2f, "
                "efficiency: %8.2f \n",
                nz, ny, nx, runtime_serial_non_persistent, npes, (stop - start),
                runtime_serial_non_persistent / (stop - start),
                runtime_serial_non_persistent / (npes * (stop - start)) * 100);
        }
    }

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
    nvshmem_free(halo_buffer_top);
    nvshmem_free(halo_buffer_bottom);
    nvshmem_free(is_done_computing_flags);

    if (compare_to_single_gpu)
    {
        CUDA_RT_CALL(hipHostFree(a_h));
        CUDA_RT_CALL(hipHostFree(a_ref_h));
    }

    nvshmem_finalize();
    MPI_CALL(MPI_Finalize());
    return 0;
}
