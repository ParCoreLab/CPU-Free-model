#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>

#include "../include/common.h"

namespace cg = cooperative_groups;

bool get_arg(char **begin, char **end, const std::string &arg) {
    char **itr = std::find(begin, end, arg);
    if (itr != end) {
        return true;
    }
    return false;
}

__global__ void initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                      const real pi, const int offset, const int nx, const int ny,
                                      const int my_nz, const int nz) {
    for (unsigned int iz = blockIdx.x * blockDim.x + threadIdx.x; iz < my_nz; iz += blockDim.x * gridDim.x) {
        for (unsigned int iy = 0; iy < ny; iy++) {
            for (unsigned int ix = 0; ix < nx; ix++) {
                const real y0 = real(offset + iz) - real(iy) - real(ix);

                a[iz * nx * ny + iy * nx + 0] = y0;
                a_new[iz * nx * ny + iy * nx + 0] = y0;
            }
        }
    }
}

__global__ void jacobi_kernel_single_gpu(real *__restrict__ const a_new,
                                         const real *__restrict__ const a,
                                         real *__restrict__ const l2_norm, const int iz_start,
                                         const int iz_end, const int ny, const int nx,
                                         const bool calculate_norm) {
    int iz = blockIdx.z * blockDim.z + threadIdx.z + iz_start;
    int iy = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;
    //    real local_l2_norm = 0.0;

    if (iz < iz_end && iy < (ny - 1) && ix < (nx - 1)) {
        const real new_val =
            (real(1) / real(6)) *
            (a[iz * ny * nx + iy * nx + ix + 1] + a[iz * ny * nx + iy * nx + ix - 1] +
             a[iz * ny * nx + (iy + 1) * nx + ix] + a[iz * ny * nx + (iy - 1) * nx + ix] +
             a[(iz + 1) * ny * nx + iy * nx + ix] + a[(iz - 1) * ny * nx + iy * nx + ix]);
        a_new[iz * ny * nx + iy * nx + ix] = new_val;

        //        if (calculate_norm) {
        //            real residue = new_val - a[iz * ny * nx + iy * nx + ix];
        //            local_l2_norm += residue * residue;
        //        }
    }
    //    if (calculate_norm) {
    //        atomicAdd(l2_norm, local_l2_norm);
    //    }
}

__global__ void jacobi_kernel_single_gpu_persistent(real *a_new, real *a, const int iz_start,
                                                    const int iz_end, const int ny, const int nx,
                                                    const bool calculate_norm, const int iter_max) {
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    int iz = blockIdx.z * blockDim.z + threadIdx.z + iz_start;
    int iy = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;

    //    real local_l2_norm = 0.0;

    int iter = 0;

    while (iter < iter_max) {
        if (iz < iz_end && iy < (ny - 1) && ix < (nx - 1)) {
            const real new_val =
                (real(1) / real(6)) *
                (a[iz * ny * nx + iy * nx + ix + 1] + a[iz * ny * nx + iy * nx + ix - 1] +
                 a[iz * ny * nx + (iy + 1) * nx + ix] + a[iz * ny * nx + (iy - 1) * nx + ix] +
                 a[(iz + 1) * ny * nx + iy * nx + ix] + a[(iz - 1) * ny * nx + iy * nx + ix]);
            a_new[iz * ny * nx + iy * nx + ix] = new_val;

            if (iz_start == iz) {
                a_new[iz_end * ny * nx + iy * nx + ix] = new_val;
            }

            if ((iz_end - 1) == iz) {
                a_new[(iz_start - 1) * ny * nx + iy * nx + ix] = new_val;
            }

            //        if (calculate_norm) {
            //            real residue = new_val - a[iy * nx + ix];
            //            local_l2_norm += residue * residue;
            //        }
        }

        iter++;

        real *temp_pointer = a_new;
        a_new = a;
        a = temp_pointer;

        cg::sync(grid);
    }

    //    if (calculate_norm) {
    //        atomicAdd(l2_norm, local_l2_norm);
    //    }
}

double single_gpu(const int nz, const int ny, const int nx, const int iter_max, real *const a_ref_h,
                  const int nccheck, const bool print) {
    real *a;
    real *a_new;

    hipStream_t compute_stream;
    hipStream_t push_top_stream;
    hipStream_t push_bottom_stream;
    hipEvent_t compute_done;
    hipEvent_t push_top_done;
    hipEvent_t push_bottom_done;

    //    real* l2_norm_d;
    //    real* l2_norm_h;

    int iz_start = 1;
    int iz_end = (nz - 1);

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * nz * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * nz * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * nz * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * nz * sizeof(real)));

    // Set diriclet boundary conditions on left and right boarder
    initialize_boundaries<<<nz / 128 + 1, 128>>>(a_new, a, PI, 0, nx, ny, nz, nz);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    CUDA_RT_CALL(hipStreamCreate(&compute_stream));
    CUDA_RT_CALL(hipStreamCreate(&push_top_stream));
    CUDA_RT_CALL(hipStreamCreate(&push_bottom_stream));
    CUDA_RT_CALL(hipEventCreateWithFlags(&compute_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&push_top_done, hipEventDisableTiming));
    CUDA_RT_CALL(hipEventCreateWithFlags(&push_bottom_done, hipEventDisableTiming));

    //    CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(real)));
    //    CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(real)));

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (print)
        printf(
            "Single GPU jacobi relaxation (non-persistent kernel): %d "
            "iterations on %d x %d x %d "
            "mesh "
            "with "
            "norm "
            "check every %d iterations\n",
            iter_max, nx, ny, nz, nccheck);
    fflush(stdout);
    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 8;
    constexpr int dim_block_z = 4;

    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x, (ny + dim_block_y - 1) / dim_block_y,
                  (nz + dim_block_z - 1) / dim_block_z);

    int iter = 0;
    bool calculate_norm = false;
    //    real l2_norm = 1.0;

    double start = omp_get_wtime();
    PUSH_RANGE("Jacobi solve", 0)
    while (iter < iter_max) {
        //        CUDA_RT_CALL(hipMemsetAsync(l2_norm_d, 0, sizeof(real),
        //        compute_stream));

        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_top_done, 0));
        CUDA_RT_CALL(hipStreamWaitEvent(compute_stream, push_bottom_done, 0));

        //        calculate_norm = (iter % nccheck) == 0 || (print && ((iter %
        //        100)
        //        == 0));
        jacobi_kernel_single_gpu<<<dim_grid, {dim_block_x, dim_block_y, dim_block_z}, 0,
                                   compute_stream>>>(a_new, a, nullptr, iz_start, iz_end, ny, nx,
                                                     calculate_norm);
        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipEventRecord(compute_done, compute_stream));

        //        if (calculate_norm) {
        //            CUDA_RT_CALL(hipMemcpyAsync(l2_norm_h, l2_norm_d,
        //            sizeof(real), hipMemcpyDeviceToHost,
        //                                         compute_stream));
        //        }

        // Apply periodic boundary conditions

        CUDA_RT_CALL(hipStreamWaitEvent(push_top_stream, compute_done, 0));
        CUDA_RT_CALL(hipMemcpyAsync(a_new, a_new + (iz_end - 1) * ny * nx, nx * ny * sizeof(real),
                                     hipMemcpyDeviceToDevice, push_top_stream));
        CUDA_RT_CALL(hipEventRecord(push_top_done, push_top_stream));

        CUDA_RT_CALL(hipStreamWaitEvent(push_bottom_stream, compute_done, 0));
        CUDA_RT_CALL(hipMemcpyAsync(a_new + iz_end * ny * nx, a_new + iz_start * ny * nx,
                                     nx * ny * sizeof(real), hipMemcpyDeviceToDevice,
                                     compute_stream));
        CUDA_RT_CALL(hipEventRecord(push_bottom_done, push_bottom_stream));

        //        if (calculate_norm) {
        //            CUDA_RT_CALL(hipStreamSynchronize(compute_stream));
        //            l2_norm = *l2_norm_h;
        //            l2_norm = std::sqrt(l2_norm);
        //            if (print && (iter % 100) == 0) printf("%5d, %0.6f\n",
        //            iter, l2_norm);
        //        }

        std::swap(a_new, a);
        iter++;
    }
    POP_RANGE
    double stop = omp_get_wtime();

    CUDA_RT_CALL(hipMemcpy(a_ref_h, a, nx * ny * nz * sizeof(real), hipMemcpyDeviceToHost));

    CUDA_RT_CALL(hipEventDestroy(push_bottom_done));
    CUDA_RT_CALL(hipEventDestroy(push_top_done));
    CUDA_RT_CALL(hipEventDestroy(compute_done));
    CUDA_RT_CALL(hipStreamDestroy(push_bottom_stream));
    CUDA_RT_CALL(hipStreamDestroy(push_top_stream));
    CUDA_RT_CALL(hipStreamDestroy(compute_stream));

    //    CUDA_RT_CALL(hipHostFree(l2_norm_h));
    //    CUDA_RT_CALL(hipFree(l2_norm_d));

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));

    return (stop - start);
}

double single_gpu_persistent(const int nz, const int ny, const int nx, const int iter_max,
                             real *const a_ref_h, const int nccheck, const bool print) {
    real *a;
    real *a_new;

    // Skipping l2-norm calculation for now
    //    real* l2_norm_d;
    //    real* l2_norm_h;

    int iz_start = 1;
    int iz_end = (nz - 1);

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * nz * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * nz * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * nz * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * nz * sizeof(real)));

    // Set diriclet boundary conditions on left and right boarder
    initialize_boundaries<<<ny / 128 + 1, 128>>>(a_new, a, PI, 0, nx, ny, nz, nz);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    //    CUDA_RT_CALL(hipMalloc(&l2_norm_d, sizeof(real)));
    //    CUDA_RT_CALL(hipHostMalloc(&l2_norm_h, sizeof(real)));

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (print)
        printf(
            "Single GPU jacobi relaxation (persistent kernel): %d iterations on "
            "%d x %d x %d mesh "
            "with "
            "norm "
            "check every %d iterations\n",
            iter_max, nx, ny, nz, nccheck);

    constexpr int dim_block_x = 8;
    constexpr int dim_block_y = 8;
    constexpr int dim_block_z = 16;

    dim3 dim_block(dim_block_x, dim_block_y, dim_block_z);
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x, (ny + dim_block_y - 1) / dim_block_y,
                  (nz + dim_block_z - 1) / dim_block_z);

    bool calculate_norm = false;
    //    real l2_norm = 1.0;

    void *kernelArgs[] = {(void *)&a_new, (void *)&a,  (void *)&iz_start,       (void *)&iz_end,
                          (void *)&ny,    (void *)&nx, (void *)&calculate_norm, (void *)&iter_max};

    double start = omp_get_wtime();

    CUDA_RT_CALL(hipLaunchCooperativeKernel((void *)jacobi_kernel_single_gpu_persistent, dim_grid,
                                             dim_block, kernelArgs, 0, nullptr));

    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    double stop = omp_get_wtime();

    CUDA_RT_CALL(hipMemcpy(a_ref_h, a, nx * ny * nz * sizeof(real), hipMemcpyDeviceToHost));

    //    CUDA_RT_CALL(hipHostFree(l2_norm_h));
    //    CUDA_RT_CALL(hipFree(l2_norm_d));

    CUDA_RT_CALL(hipFree(a_new));
    CUDA_RT_CALL(hipFree(a));
    return (stop - start);
}

void report_results(const int nz, const int ny, const int nx, real *a_ref_h, real *a_h,
                    const int num_devices, const double runtime_serial_non_persistent,
                    const double start, const double stop, const bool compare_to_single_gpu) {
    bool result_correct = true;

    if (compare_to_single_gpu) {
        for (int iz = 1; result_correct && (iz < (nz - 1)); ++iz) {
            for (int iy = 1; result_correct && (iy < (ny - 1)); ++iy) {
                for (int ix = 1; result_correct && (ix < (nx - 1)); ++ix) {
                    if (std::fabs(a_h[iz * ny * nx + iy * nx + ix] -
                                  a_ref_h[iz * ny * nx + iy * nx + ix]) > tol) {
                        fprintf(stderr,
                                "ERROR: a[%d * %d + %d * %d + %d] = %f does "
                                "not match %f "
                                "(reference)\n",
                                iz, ny * nx, iy, nx, ix, a_h[iz * ny * nx + iy * nx + ix],
                                a_ref_h[iz * ny * nx + iy * nx + ix]);
                        // result_correct = false;
                    }
                }
            }
        }
    }

    if (result_correct) {
        // printf("Num GPUs: %d.\n", num_devices);
        printf("Execution time: %8.4f s\n", (stop - start));

        if (compare_to_single_gpu) {
            printf(
                "Non-persistent kernel - %dx%dx%d: 1 GPU: %8.4f s, %d GPUs: "
                "%8.4f "
                "s, speedup: "
                "%8.2f, "
                "efficiency: %8.2f \n",
                nx, ny, nz, runtime_serial_non_persistent, num_devices, (stop - start),
                runtime_serial_non_persistent / (stop - start),
                runtime_serial_non_persistent / (num_devices * (stop - start)) * 100);
        }
    }
}