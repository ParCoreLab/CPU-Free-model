#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a conjugate gradient solver on multiple GPU using
 * Unified Memory optimized prefetching and usage hints.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <filesystem>
#include <iostream>
#include <map>
#include <set>
#include <utility>

#include <omp.h>

#include "../../include/common.h"
#include "../../include/profiling/discrete-pipelined.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

namespace ProfilingDiscretePipelined {

// delta => <r, r>
// gamma => <r, w>
__device__ double grid_dot_result_delta = 0.0;
__device__ double grid_dot_result_gamma = 0.0;

// Performs two dot products at the same time
// Used to perform <r, r> and <r, w> at the same time
// Can we combined the two atomicAdds somehow?

__global__ void gpuDotProductsMerged(real *vecA_delta, real *vecB_delta, real *vecA_gamma,
                                     real *vecB_gamma, int num_rows, const int device_rank,
                                     const int num_devices, const int sMemSize) {
    cg::thread_block cta = cg::this_thread_block();

    size_t local_grid_size = gridDim.x * blockDim.x;
    size_t local_grid_rank = blockIdx.x * blockDim.x + threadIdx.x;

    size_t global_grid_size = local_grid_size * num_devices;
    size_t global_grid_rank = device_rank * local_grid_size + local_grid_rank;

    extern __shared__ double tmp[];

    double *tmp_delta = (double *)tmp;
    double *tmp_gamma = (double *)&tmp_delta[sMemSize / (2 * sizeof(double))];

    double temp_sum_delta = 0.0;
    double temp_sum_gamma = 0.0;

    for (size_t i = global_grid_rank; i < num_rows; i += global_grid_size) {
        temp_sum_delta += (double)(vecA_delta[i] * vecB_delta[i]);
        temp_sum_gamma += (double)(vecA_gamma[i] * vecB_gamma[i]);
    }

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    temp_sum_delta = cg::reduce(tile32, temp_sum_delta, cg::plus<double>());
    temp_sum_gamma = cg::reduce(tile32, temp_sum_gamma, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
        tmp_delta[tile32.meta_group_rank()] = temp_sum_delta;
        tmp_gamma[tile32.meta_group_rank()] = temp_sum_gamma;
    }

    cg::sync(cta);

    if (tile32.meta_group_rank() == 0) {
        temp_sum_delta =
            tile32.thread_rank() < tile32.meta_group_size() ? tmp_delta[tile32.thread_rank()] : 0.0;
        temp_sum_delta = cg::reduce(tile32, temp_sum_delta, cg::plus<double>());

        temp_sum_gamma =
            tile32.thread_rank() < tile32.meta_group_size() ? tmp_gamma[tile32.thread_rank()] : 0.0;
        temp_sum_gamma = cg::reduce(tile32, temp_sum_gamma, cg::plus<double>());

        if (tile32.thread_rank() == 0) {
            atomicAdd(&grid_dot_result_delta, temp_sum_delta);
            atomicAdd(&grid_dot_result_gamma, temp_sum_gamma);
        }
    }
}

__global__ void addLocalDotContributions(double *dot_result_delta, double *dot_result_gamma) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid == 0) {
        atomicAdd_system(dot_result_delta, grid_dot_result_delta);
        atomicAdd_system(dot_result_gamma, grid_dot_result_gamma);

        grid_dot_result_delta = 0.0;
        grid_dot_result_gamma = 0.0;
    }
}

__global__ void resetLocalDotProducts(double *dot_result_delta, double *dot_result_gamma,
                                      const int gpu_idx) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gpu_idx == 0 && gid == 0) {
        *dot_result_delta = 0.0;
        *dot_result_gamma = 0.0;
    }
}

}  // namespace ProfilingDiscretePipelined

int ProfilingDiscretePipelined::init(int argc, char *argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 10000);
    std::string matrix_path_str = get_argval<std::string>(argv, argv + argc, "-matrix_path", "");
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare-single-gpu");
    const bool compare_to_cpu = get_arg(argv, argv + argc, "-compare-cpu");

    char *matrix_path_char = const_cast<char *>(matrix_path_str.c_str());
    bool generate_random_tridiag_matrix = matrix_path_str.empty();

    std::string matrix_name = std::filesystem::path(matrix_path_str).stem();

    if (generate_random_tridiag_matrix) {
        matrix_name = "random tridiagonal";
    }

    // std::cout << "Running on matrix: " << matrix_name << "\n" << std::endl;

    int num_devices = 0;
    double single_gpu_runtime;

    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    int num_rows = 0;
    int num_cols = 0;
    int nnz = 0;

    int *host_I = NULL;
    int *host_J = NULL;
    real *host_val = NULL;

    real *x_ref_single_gpu = NULL;

    real *s_cpu = NULL;
    real *r_cpu = NULL;
    real *p_cpu = NULL;
    real *x_ref_cpu = NULL;

    int *um_I = NULL;
    int *um_J = NULL;
    real *um_val = NULL;

    real *um_x;
    real *um_r;
    real *um_p;
    real *um_s;
    real *um_z;
    real *um_w;
    real *um_q;
    real *um_ax0;

    double *um_tmp_dot_delta1;
    double *um_tmp_dot_gamma1;

    real real_positive_one = 1.0;
    real real_negative_one = -1.0;

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (generate_random_tridiag_matrix) {
        num_rows = 10485760 * 2;
        num_cols = num_rows;

        nnz = (num_rows - 2) * 3 + 4;

        CUDA_RT_CALL(hipMallocManaged((void **)&um_I, sizeof(int) * (num_rows + 1)));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_J, sizeof(int) * nnz));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_val, sizeof(real) * nnz));

        /* Generate a random tridiagonal symmetric matrix in CSR format */
        genTridiag(um_I, um_J, um_val, num_rows, nnz);
    } else {
        if (loadMMSparseMatrix<real>(matrix_path_char, 'd', true, &num_rows, &num_cols, &nnz,
                                     &host_val, &host_I, &host_J, true)) {
            exit(EXIT_FAILURE);
        }

        CUDA_RT_CALL(hipMallocManaged((void **)&um_I, sizeof(int) * (num_rows + 1)));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_J, sizeof(int) * nnz));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_val, sizeof(real) * nnz));

        memcpy(um_I, host_I, sizeof(int) * (num_rows + 1));
        memcpy(um_J, host_J, sizeof(int) * nnz);
        memcpy(um_val, host_val, sizeof(real) * nnz);
    }

    if (compare_to_single_gpu) {
        CUDA_RT_CALL(hipHostMalloc(&x_ref_single_gpu, num_rows * sizeof(real)));

        single_gpu_runtime = SingleGPUDiscreteStandard::run_single_gpu(
            iter_max, um_I, um_J, um_val, x_ref_single_gpu, num_rows, nnz);

        // single_gpu_runtime = SingleGPUDiscretePipelined::run_single_gpu(
        //     iter_max, um_I, um_J, um_val, x_ref_single_gpu, num_rows, nnz);
    }

    CUDA_RT_CALL(hipMallocManaged((void **)&um_x, sizeof(real) * num_rows));

    CUDA_RT_CALL(hipMallocManaged((void **)&um_tmp_dot_delta1, sizeof(double)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_tmp_dot_gamma1, sizeof(double)));

    // temp memory for ConjugateGradient
    CUDA_RT_CALL(hipMallocManaged((void **)&um_r, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_p, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_s, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_z, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_w, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_q, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_ax0, num_rows * sizeof(real)));

    CUDA_RT_CALL(hipMemset(um_tmp_dot_delta1, 0, sizeof(double)));
    CUDA_RT_CALL(hipMemset(um_tmp_dot_gamma1, 0, sizeof(double)));

    if (compare_to_cpu) {
        s_cpu = (real *)malloc(sizeof(real) * num_rows);
        r_cpu = (real *)malloc(sizeof(real) * num_rows);
        p_cpu = (real *)malloc(sizeof(real) * num_rows);

        CUDA_RT_CALL(hipHostMalloc(&x_ref_cpu, num_rows * sizeof(real)));

        for (int i = 0; i < num_rows; i++) {
            r_cpu[i] = 1.0;
            s_cpu[i] = 0.0;
            x_ref_cpu[i] = 0.0;
        }

        CPU::cpuConjugateGrad(iter_max, um_I, um_J, um_val, x_ref_cpu, s_cpu, p_cpu, r_cpu, nnz,
                              num_rows, tol);
    }

    CUDA_RT_CALL(hipDeviceSynchronize());

    hipEvent_t atomic_add_done[num_devices];
    hipEvent_t iteration_done[num_devices];

#pragma omp parallel num_threads(num_devices)                                                     \
    firstprivate(um_I, um_J, um_val, um_x, um_r, um_p, um_s, um_z, um_w, um_q, um_tmp_dot_delta1, \
                 um_tmp_dot_gamma1)
    {
        int gpu_idx = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(gpu_idx));
        CUDA_RT_CALL(hipFree(0));

        hipStream_t mainStream;

        real tmp_dot_delta0;
        real tmp_dot_gamma0;

        real alpha;
        real negative_alpha;
        real beta;

        CUDA_RT_CALL(hipEventCreateWithFlags(atomic_add_done + gpu_idx, hipEventDisableTiming));
        CUDA_RT_CALL(hipEventCreateWithFlags(iteration_done + gpu_idx, hipEventDisableTiming));

        for (int gpu_idx_j = 0; gpu_idx_j < num_devices; gpu_idx_j++) {
            if (gpu_idx != gpu_idx_j) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(gpu_idx_j, 0));
            }
        }

#pragma omp barrier

        int sMemSize = 2 * (sizeof(double) * ((THREADS_PER_BLOCK / 32) + 1));
        int numBlocks = (num_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

#pragma omp barrier

        CUDA_RT_CALL(hipStreamCreate(&mainStream));

        CUDA_RT_CALL(hipDeviceSynchronize());

#pragma omp barrier

        double start = omp_get_wtime();

        MultiGPU::initVectors<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
            um_r, um_x, num_rows, gpu_idx, num_devices);

        // ax0 = Ax0
        MultiGPU::gpuSpMV<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
            um_I, um_J, um_val, nnz, num_rows, real_positive_one, um_x, um_ax0, gpu_idx,
            num_devices);

        // r0 = b0 - ax0
        // NOTE: b is a unit vector.
        MultiGPU::gpuSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
            um_ax0, um_r, real_negative_one, num_rows, gpu_idx, num_devices);

        // w0 = Ar0
        MultiGPU::gpuSpMV<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
            um_I, um_J, um_val, nnz, num_rows, real_positive_one, um_r, um_w, gpu_idx, num_devices);

        CUDA_RT_CALL(hipEventRecord(iteration_done[gpu_idx], mainStream));

        for (int neighbor_gpu_idx = 0; neighbor_gpu_idx < num_devices; neighbor_gpu_idx++) {
            CUDA_RT_CALL(hipEventSynchronize(iteration_done[neighbor_gpu_idx]))
        }

        int k = 1;

        while (k <= iter_max) {
            PUSH_RANGE("Merged Dots (+Reset)", 0);

            // Two dot products => <r, r> and <r, w>
            resetLocalDotProducts<<<1, 1, 0, mainStream>>>(um_tmp_dot_delta1, um_tmp_dot_gamma1,
                                                           gpu_idx);

            // Dot
            gpuDotProductsMerged<<<numBlocks, THREADS_PER_BLOCK, sMemSize, mainStream>>>(
                um_r, um_r, um_r, um_w, num_rows, gpu_idx, num_devices, sMemSize);

            hipStreamSynchronize(mainStream);

            POP_RANGE

            PUSH_RANGE("SpMV", 1);

            // SpMV
            MultiGPU::gpuSpMV<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
                um_I, um_J, um_val, nnz, num_rows, real_positive_one, um_w, um_q, gpu_idx,
                num_devices);

            hipStreamSynchronize(mainStream);

            POP_RANGE

            PUSH_RANGE("Atomic Adds", 2);

            addLocalDotContributions<<<1, 1, 0, mainStream>>>(um_tmp_dot_delta1, um_tmp_dot_gamma1);

            hipStreamSynchronize(mainStream);

            POP_RANGE

            PUSH_RANGE("Peer Sync 1", 3);

            CUDA_RT_CALL(hipEventRecord(atomic_add_done[gpu_idx], mainStream));

            for (int neighbor_gpu_idx = 0; neighbor_gpu_idx < num_devices; neighbor_gpu_idx++) {
                CUDA_RT_CALL(hipEventSynchronize(atomic_add_done[neighbor_gpu_idx]))
            }

            POP_RANGE

            real real_tmp_dot_delta1 = (real)*um_tmp_dot_delta1;
            real real_tmp_dot_gamma1 = (real)*um_tmp_dot_gamma1;

            if (k > 1) {
                beta = real_tmp_dot_delta1 / tmp_dot_delta0;
                alpha = real_tmp_dot_delta1 /
                        (real_tmp_dot_gamma1 - (beta / alpha) * real_tmp_dot_delta1);
            } else {
                beta = 0.0;
                alpha = real_tmp_dot_delta1 / real_tmp_dot_gamma1;
            }

            CUDA_RT_CALL(hipStreamSynchronize(mainStream));

            PUSH_RANGE("Saxpy 1", 4);

            // z_k = q_k + beta_k * z_(k-1)
            MultiGPU::gpuScaleVectorAndSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
                um_q, um_z, real_positive_one, beta, num_rows, gpu_idx, num_devices);

            CUDA_RT_CALL(hipStreamSynchronize(mainStream));

            POP_RANGE

            PUSH_RANGE("Saxpy 2", 5);

            // s_k = w_k + beta_k * s_(k-1)
            MultiGPU::gpuScaleVectorAndSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
                um_w, um_s, real_positive_one, beta, num_rows, gpu_idx, num_devices);

            CUDA_RT_CALL(hipStreamSynchronize(mainStream));

            POP_RANGE

            PUSH_RANGE("Saxpy 3", 6);

            // p_k = r_k = beta_k * p_(k-1)
            MultiGPU::gpuScaleVectorAndSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
                um_r, um_p, real_positive_one, beta, num_rows, gpu_idx, num_devices);

            CUDA_RT_CALL(hipStreamSynchronize(mainStream));

            POP_RANGE

            PUSH_RANGE("Saxpy 4", 7);

            // x_(k+1) = x_k + alpha_k * p_k
            MultiGPU::gpuSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
                um_p, um_x, alpha, num_rows, gpu_idx, num_devices);

            CUDA_RT_CALL(hipStreamSynchronize(mainStream));

            POP_RANGE

            negative_alpha = -alpha;

            PUSH_RANGE("Saxpy 5", 8);

            // r_(k+1) = r_k - alpha_k * s_k
            MultiGPU::gpuSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
                um_s, um_r, negative_alpha, num_rows, gpu_idx, num_devices);

            CUDA_RT_CALL(hipStreamSynchronize(mainStream));

            POP_RANGE

            PUSH_RANGE("Saxpy 6", 9);

            // w_(k+1) = w_k - alpha_k * z_k
            MultiGPU::gpuSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
                um_z, um_w, negative_alpha, num_rows, gpu_idx, num_devices);

            CUDA_RT_CALL(hipStreamSynchronize(mainStream));

            POP_RANGE

            tmp_dot_delta0 = (real)*um_tmp_dot_delta1;
            tmp_dot_gamma0 = (real)*um_tmp_dot_gamma1;

            PUSH_RANGE("Peer Sync 2", 10);

            CUDA_RT_CALL(hipEventRecord(iteration_done[gpu_idx], 0));

            for (int neighbor_gpu_idx = 0; neighbor_gpu_idx < num_devices; neighbor_gpu_idx++) {
                CUDA_RT_CALL(hipEventSynchronize(iteration_done[neighbor_gpu_idx]))
            }

            POP_RANGE

#pragma omp barrier

            k++;
        }

#pragma omp barrier

        double stop = omp_get_wtime();

#pragma omp master
        {
            report_results(num_rows, x_ref_single_gpu, x_ref_cpu, um_x, num_devices,
                           single_gpu_runtime, start, stop, compare_to_single_gpu, compare_to_cpu);
        }

#pragma omp barrier

        CUDA_RT_CALL(hipStreamDestroy(mainStream));
    }

    CUDA_RT_CALL(hipFree(um_I));
    CUDA_RT_CALL(hipFree(um_J));
    CUDA_RT_CALL(hipFree(um_val));
    CUDA_RT_CALL(hipFree(um_x));
    CUDA_RT_CALL(hipFree(um_r));
    CUDA_RT_CALL(hipFree(um_p));
    CUDA_RT_CALL(hipFree(um_s));
    CUDA_RT_CALL(hipFree(um_tmp_dot_delta1));
    CUDA_RT_CALL(hipFree(um_tmp_dot_gamma1));
    free(host_val);

    return 0;
}
