#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a conjugate gradient solver on multiple GPU using
 * Unified Memory optimized prefetching and usage hints.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <filesystem>
#include <iostream>
#include <map>
#include <set>
#include <utility>

#include <omp.h>

#include "../../include/baseline/persistent-non-pipelined.cuh"
#include "../../include/common.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

namespace BaselinePersistentNonPipelined {

__device__ double grid_dot_result = 0.0;

__device__ void gpuSpMV(int *I, int *J, real *val, int nnz, int num_rows, real alpha,
                        real *inputVecX, real *outputVecY, const PeerGroup &peer_group) {
    for (int i = peer_group.thread_rank(); i < num_rows; i += peer_group.size()) {
        int row_elem = I[i];
        int next_row_elem = I[i + 1];
        int num_elems_this_row = next_row_elem - row_elem;

        real output = 0.0;
        for (int j = 0; j < num_elems_this_row; j++) {
            output += alpha * val[row_elem + j] * inputVecX[J[row_elem + j]];
        }

        outputVecY[i] = output;
    }
}

__device__ void gpuSaxpy(real *x, real *y, real a, int size, const PeerGroup &peer_group) {
    for (int i = peer_group.thread_rank(); i < size; i += peer_group.size()) {
        y[i] = a * x[i] + y[i];
    }
}

__device__ void gpuDotProduct(real *vecA, real *vecB, int size, const cg::thread_block &cta,
                              const PeerGroup &peer_group) {
    extern __shared__ double tmp[];

    double temp_sum = 0.0;

    for (int i = peer_group.thread_rank(); i < size; i += peer_group.size()) {
        temp_sum += (double)(vecA[i] * vecB[i]);
    }

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
        tmp[tile32.meta_group_rank()] = temp_sum;
    }

    cg::sync(cta);

    if (tile32.meta_group_rank() == 0) {
        temp_sum =
            tile32.thread_rank() < tile32.meta_group_size() ? tmp[tile32.thread_rank()] : 0.0;
        temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

        if (tile32.thread_rank() == 0) {
            atomicAdd(&grid_dot_result, temp_sum);
        }
    }
}

__device__ void gpuCopyVector(real *srcA, real *destB, int size, const PeerGroup &peer_group) {
    for (int i = peer_group.thread_rank(); i < size; i += peer_group.size()) {
        destB[i] = srcA[i];
    }
}

__device__ void gpuScaleVectorAndSaxpy(real *x, real *y, real a, real scale, int size,
                                       const PeerGroup &peer_group) {
    for (int i = peer_group.thread_rank(); i < size; i += peer_group.size()) {
        y[i] = a * x[i] + scale * y[i];
    }
}

__global__ void multiGpuConjugateGradient(int *I, int *J, real *val, real *x, real *ax0, real *s,
                                          real *p, real *r, double *dot_result, int nnz, int N,
                                          real tol, MultiDeviceData multi_device_data,
                                          const int iter_max) {
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    PeerGroup peer_group(multi_device_data, grid);

    real real_positive_one = 1.0;
    real real_negative_one = -1.0;

    real r0 = 0.0;
    real r1 = 0.0;
    real b;
    real a;
    real na;

    for (int i = peer_group.thread_rank(); i < N; i += peer_group.size()) {
        r[i] = 1.0;
        x[i] = 0.0;
    }

    cg::sync(grid);

    gpuSpMV(I, J, val, nnz, N, real_positive_one, x, ax0, peer_group);

    cg::sync(grid);

    gpuSaxpy(ax0, r, real_negative_one, N, peer_group);

    cg::sync(grid);

    gpuCopyVector(r, p, N, peer_group);

    cg::sync(grid);

    gpuDotProduct(r, r, N, cta, peer_group);

    cg::sync(grid);

    if (grid.thread_rank() == 0) {
        atomicAdd_system(dot_result, grid_dot_result);
        grid_dot_result = 0.0;
    }

    peer_group.sync();

    r0 = *dot_result;

    int k = 1;

    while (k <= iter_max) {
        gpuSpMV(I, J, val, nnz, N, real_positive_one, p, s, peer_group);

        if (peer_group.thread_rank() == 0) {
            *dot_result = 0.0;
        }

        peer_group.sync();

        gpuDotProduct(p, s, N, cta, peer_group);

        cg::sync(grid);

        if (grid.thread_rank() == 0) {
            atomicAdd_system(dot_result, grid_dot_result);
            grid_dot_result = 0.0;
        }

        peer_group.sync();

        a = r0 / *dot_result;

        gpuSaxpy(p, x, a, N, peer_group);

        na = -a;

        gpuSaxpy(s, r, na, N, peer_group);

        peer_group.sync();

        if (peer_group.thread_rank() == 0) {
            *dot_result = 0.0;
        }

        peer_group.sync();

        gpuDotProduct(r, r, N, cta, peer_group);

        cg::sync(grid);

        if (grid.thread_rank() == 0) {
            atomicAdd_system(dot_result, grid_dot_result);
            grid_dot_result = 0.0;
        }

        peer_group.sync();

        r1 = *dot_result;

        b = r1 / r0;

        gpuScaleVectorAndSaxpy(r, p, real_positive_one, b, N, peer_group);

        r0 = r1;

        peer_group.sync();

        k++;
    }
}
}  // namespace BaselinePersistentNonPipelined

int BaselinePersistentNonPipelined::init(int argc, char *argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 10000);
    std::string matrix_path_str = get_argval<std::string>(argv, argv + argc, "-matrix_path", "");
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare-single-gpu");
    const bool compare_to_cpu = get_arg(argv, argv + argc, "-compare-cpu");

    char *matrix_path_char = const_cast<char *>(matrix_path_str.c_str());
    bool generate_random_tridiag_matrix = matrix_path_str.empty();

    std::string matrix_name = std::filesystem::path(matrix_path_str).stem();

    if (generate_random_tridiag_matrix) {
        matrix_name = "random tridiagonal";
    }

    int num_devices = 0;
    double single_gpu_runtime;

    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    int num_rows = 0;
    int num_cols = 0;
    int nnz = 0;

    // Structure used for cross-grid synchronization.
    MultiDeviceData multi_device_data;

    int *host_I = NULL;
    int *host_J = NULL;
    real *host_val = NULL;

    real *x_ref_single_gpu = NULL;

    real *s_cpu = NULL;
    real *r_cpu = NULL;
    real *p_cpu = NULL;
    real *x_ref_cpu = NULL;

    int *um_I = NULL;
    int *um_J = NULL;
    real *um_val = NULL;

    real r1;

    real *um_x;
    real *um_r;
    real *um_p;
    real *um_s;
    real *um_ax0;

    double *dot_result;

    if (generate_random_tridiag_matrix) {
        num_rows = 10485760 * 2;
        num_cols = num_rows;

        nnz = (num_rows - 2) * 3 + 4;

        CUDA_RT_CALL(hipMallocManaged((void **)&um_I, sizeof(int) * (num_rows + 1)));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_J, sizeof(int) * nnz));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_val, sizeof(real) * nnz));

        /* Generate a random tridiagonal symmetric matrix in CSR format */
        genTridiag(um_I, um_J, um_val, num_rows, nnz);
    } else {
        if (loadMMSparseMatrix<real>(matrix_path_char, 'd', true, &num_rows, &num_cols, &nnz,
                                     &host_val, &host_I, &host_J, true)) {
            exit(EXIT_FAILURE);
        }

        CUDA_RT_CALL(hipMallocManaged((void **)&um_I, sizeof(int) * (num_rows + 1)));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_J, sizeof(int) * nnz));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_val, sizeof(real) * nnz));

        memcpy(um_I, host_I, sizeof(int) * (num_rows + 1));
        memcpy(um_J, host_J, sizeof(int) * nnz);
        memcpy(um_val, host_val, sizeof(real) * nnz);
    }

    CUDA_RT_CALL(hipMallocManaged((void **)&um_x, sizeof(real) * num_rows));

    CUDA_RT_CALL(hipMallocManaged((void **)&dot_result, sizeof(double)));

    // temp memory for ConjugateGradient
    CUDA_RT_CALL(hipMallocManaged((void **)&um_r, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_p, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_s, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_ax0, num_rows * sizeof(real)));

    CUDA_RT_CALL(hipHostAlloc(&multi_device_data.hostMemoryArrivedList,
                               (num_devices - 1) * sizeof(*multi_device_data.hostMemoryArrivedList),
                               hipHostMallocPortable));
    memset(multi_device_data.hostMemoryArrivedList, 0,
           (num_devices - 1) * sizeof(*multi_device_data.hostMemoryArrivedList));
    multi_device_data.numDevices = num_devices;
    multi_device_data.deviceRank = 0;

    CUDA_RT_CALL(hipMemset(dot_result, 0, sizeof(double)));

    CUDA_RT_CALL(hipDeviceSynchronize());

    if (compare_to_single_gpu) {
        CUDA_RT_CALL(hipHostMalloc(&x_ref_single_gpu, num_rows * sizeof(real)));

        single_gpu_runtime = SingleGPUStandardDiscrete::run_single_gpu(
            iter_max, um_I, um_J, um_val, x_ref_single_gpu, num_rows, nnz);

        // single_gpu_runtime = SingleGPUPipelinedDiscrete::run_single_gpu(
        //     iter_max, um_I, um_J, um_val, x_ref_single_gpu, num_rows, nnz);
    }

    if (compare_to_cpu) {
        s_cpu = (real *)malloc(sizeof(real) * num_rows);
        r_cpu = (real *)malloc(sizeof(real) * num_rows);
        p_cpu = (real *)malloc(sizeof(real) * num_rows);

        CUDA_RT_CALL(hipHostMalloc(&x_ref_cpu, num_rows * sizeof(real)));

        for (int i = 0; i < num_rows; i++) {
            r_cpu[i] = 1.0;
            s_cpu[i] = 0.0;
            x_ref_cpu[i] = 0.0;
        }

        CPU::cpuConjugateGrad(iter_max, um_I, um_J, um_val, x_ref_cpu, s_cpu, p_cpu, r_cpu, nnz,
                              num_rows, tol);
    }

    CUDA_RT_CALL(hipDeviceSynchronize());

#pragma omp parallel num_threads(num_devices) \
    firstprivate(um_I, um_J, um_val, um_x, um_r, um_p, um_s, um_ax0, dot_result)
    {
        int gpu_idx = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(gpu_idx));
        CUDA_RT_CALL(hipFree(0));

        hipStream_t mainStream;

#pragma omp barrier

        for (int gpu_idx_j = 0; gpu_idx_j < num_devices; gpu_idx_j++) {
            if (gpu_idx != gpu_idx_j) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(gpu_idx_j, 0));
            }
        }

#pragma omp barrier

        int sMemSize = sizeof(double) * ((THREADS_PER_BLOCK / 32) + 1);
        int numBlocksPerSm = INT_MAX;
        int numThreads = THREADS_PER_BLOCK;

#pragma omp barrier

        CUDA_RT_CALL(hipStreamCreate(&mainStream));

#pragma omp barrier

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        int numSms = deviceProp.multiProcessorCount;

        numBlocksPerSm = INT_MAX;

        CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &numBlocksPerSm, multiGpuConjugateGradient, numThreads, sMemSize));

        if (!numBlocksPerSm) {
            printf("Max active blocks per SM is returned as 0. Exiting!\n");

            exit(EXIT_FAILURE);
        }

#pragma omp barrier

        dim3 dimGrid(numSms * numBlocksPerSm, 1, 1), dimBlock(numThreads, 1, 1);

        void *kernelArgs[] = {
            (void *)&um_I,
            (void *)&um_J,
            (void *)&um_val,
            (void *)&um_x,
            (void *)&um_ax0,
            (void *)&um_s,
            (void *)&um_p,
            (void *)&um_r,
            (void *)&dot_result,
            (void *)&nnz,
            (void *)&num_rows,
            (void *)&tol,
            (void *)&multi_device_data,
            (void *)&iter_max,
        };

        CUDA_RT_CALL(hipDeviceSynchronize());

#pragma omp barrier

        double start = omp_get_wtime();

#pragma omp critical
        {
            multi_device_data.deviceRank = gpu_idx;
            CUDA_RT_CALL(hipLaunchCooperativeKernel((void *)multiGpuConjugateGradient, dimGrid,
                                                     dimBlock, kernelArgs, sMemSize, mainStream));
        }

        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        // #pragma omp barrier

        // #pragma omp master
        //         { r1 = (real)*dot_result; }

#pragma omp barrier

        double stop = omp_get_wtime();

#pragma omp master
        {
            report_results(num_rows, x_ref_single_gpu, x_ref_cpu, um_x, num_devices,
                           single_gpu_runtime, start, stop, compare_to_single_gpu, compare_to_cpu);
        }

#pragma omp barrier

        CUDA_RT_CALL(hipStreamDestroy(mainStream));
    }

    CUDA_RT_CALL(hipHostFree(multi_device_data.hostMemoryArrivedList));
    CUDA_RT_CALL(hipFree(um_I));
    CUDA_RT_CALL(hipFree(um_J));
    CUDA_RT_CALL(hipFree(um_val));
    CUDA_RT_CALL(hipFree(um_x));
    CUDA_RT_CALL(hipFree(um_r));
    CUDA_RT_CALL(hipFree(um_p));
    CUDA_RT_CALL(hipFree(um_s));
    CUDA_RT_CALL(hipFree(um_ax0));
    CUDA_RT_CALL(hipFree(dot_result));
    free(host_val);

    CUDA_RT_CALL(hipHostFree(x_ref_single_gpu));

    return 0;
}
