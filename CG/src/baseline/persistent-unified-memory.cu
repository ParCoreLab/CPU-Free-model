#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a conjugate gradient solver on multiple GPU using
 * Unified Memory optimized prefetching and usage hints.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <map>
#include <set>
#include <utility>

#include <omp.h>

#include "../../include/baseline/persistent-unified-memory.cuh"
#include "../../include/common.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

namespace BaselinePersistentUnifiedMemory {
const char *sSDKname = "conjugateGradientMultiDeviceCG";

#define ENABLE_CPU_DEBUG_CODE 0
#define THREADS_PER_BLOCK 512

__device__ double grid_dot_result = 0.0;

// Data filled on CPU needed for MultiGPU operations.
struct MultiDeviceData {
    unsigned char *hostMemoryArrivedList;
    unsigned int numDevices;
    unsigned int deviceRank;
};

// Class used for coordination of multiple devices.
class PeerGroup {
    const MultiDeviceData &data;
    const cg::grid_group &grid;

    __device__ unsigned char load_arrived(unsigned char *arrived) const {
#if __CUDA_ARCH__ < 700
        return *(volatile unsigned char *)arrived;
#else
        unsigned int result;
        asm volatile("ld.acquire.sys.global.u8 %0, [%1];" : "=r"(result) : "l"(arrived) : "memory");
        return result;
#endif
    }

    __device__ void store_arrived(unsigned char *arrived, unsigned char val) const {
#if __CUDA_ARCH__ < 700
        *(volatile unsigned char *)arrived = val;
#else
        unsigned int reg_val = val;
        asm volatile("st.release.sys.global.u8 [%1], %0;" ::"r"(reg_val) "l"(arrived) : "memory");

        // Avoids compiler warnings from unused variable val.
        (void)(reg_val = reg_val);
#endif
    }

   public:
    __device__ PeerGroup(const MultiDeviceData &data, const cg::grid_group &grid)
        : data(data), grid(grid){};

    __device__ unsigned int size() const { return data.numDevices * grid.size(); }

    __device__ unsigned int thread_rank() const {
        return data.deviceRank * grid.size() + grid.thread_rank();
    }

    __device__ void sync() const {
        grid.sync();

        // One thread from each grid participates in the sync.
        if (grid.thread_rank() == 0) {
            if (data.deviceRank == 0) {
                // Leader grid waits for others to join and then releases them.
                // Other GPUs can arrive in any order, so the leader have to wait for
                // all others.
                for (int i = 0; i < data.numDevices - 1; i++) {
                    while (load_arrived(&data.hostMemoryArrivedList[i]) == 0)
                        ;
                }
                for (int i = 0; i < data.numDevices - 1; i++) {
                    store_arrived(&data.hostMemoryArrivedList[i], 0);
                }
                __threadfence_system();
            } else {
                // Other grids note their arrival and wait to be released.
                store_arrived(&data.hostMemoryArrivedList[data.deviceRank - 1], 1);
                while (load_arrived(&data.hostMemoryArrivedList[data.deviceRank - 1]) == 1)
                    ;
            }
        }

        grid.sync();
    }
};

__device__ void gpuSpMV(int *I, int *J, float *val, int nnz, int num_rows, float alpha,
                        float *inputVecX, float *outputVecY, const PeerGroup &peer_group) {
    for (int i = peer_group.thread_rank(); i < num_rows; i += peer_group.size()) {
        int row_elem = I[i];
        int next_row_elem = I[i + 1];
        int num_elems_this_row = next_row_elem - row_elem;

        float output = 0.0;
        for (int j = 0; j < num_elems_this_row; j++) {
            output += alpha * val[row_elem + j] * inputVecX[J[row_elem + j]];
        }

        outputVecY[i] = output;
    }
}

__device__ void gpuSaxpy(float *x, float *y, float a, int size, const PeerGroup &peer_group) {
    for (int i = peer_group.thread_rank(); i < size; i += peer_group.size()) {
        y[i] = a * x[i] + y[i];
    }
}

__device__ void gpuDotProduct(float *vecA, float *vecB, int size, const cg::thread_block &cta,
                              const PeerGroup &peer_group) {
    extern __shared__ double tmp[];

    double temp_sum = 0.0;

    for (int i = peer_group.thread_rank(); i < size; i += peer_group.size()) {
        temp_sum += (double)(vecA[i] * vecB[i]);
    }

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
        tmp[tile32.meta_group_rank()] = temp_sum;
    }

    cg::sync(cta);

    if (tile32.meta_group_rank() == 0) {
        temp_sum =
            tile32.thread_rank() < tile32.meta_group_size() ? tmp[tile32.thread_rank()] : 0.0;
        temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

        if (tile32.thread_rank() == 0) {
            atomicAdd(&grid_dot_result, temp_sum);
        }
    }
}

__device__ void gpuCopyVector(float *srcA, float *destB, int size, const PeerGroup &peer_group) {
    for (int i = peer_group.thread_rank(); i < size; i += peer_group.size()) {
        destB[i] = srcA[i];
    }
}

__device__ void gpuScaleVectorAndSaxpy(float *x, float *y, float a, float scale, int size,
                                       const PeerGroup &peer_group) {
    for (int i = peer_group.thread_rank(); i < size; i += peer_group.size()) {
        y[i] = a * x[i] + scale * y[i];
    }
}

extern "C" __global__ void multiGpuConjugateGradient(int *I, int *J, float *val, float *x,
                                                     float *Ax, float *p, float *r,
                                                     double *dot_result, int nnz, int N, float tol,
                                                     MultiDeviceData multi_device_data,
                                                     const int iter_max) {
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    PeerGroup peer_group(multi_device_data, grid);

    float alpha = 1.0;
    float alpham1 = -1.0;
    float r0 = 0.0, r1, b, a, na;

    for (int i = peer_group.thread_rank(); i < N; i += peer_group.size()) {
        r[i] = 1.0;
        x[i] = 0.0;
    }

    cg::sync(grid);

    gpuSpMV(I, J, val, nnz, N, alpha, x, Ax, peer_group);

    cg::sync(grid);

    gpuSaxpy(Ax, r, alpham1, N, peer_group);

    cg::sync(grid);

    gpuDotProduct(r, r, N, cta, peer_group);

    cg::sync(grid);

    if (grid.thread_rank() == 0) {
        atomicAdd_system(dot_result, grid_dot_result);
        grid_dot_result = 0.0;
    }
    peer_group.sync();

    r1 = *dot_result;

    int k = 1;

    // while (r1 > tol * tol && k <= iter_max)

    while (k <= iter_max) {
        // Saxpy 1 Start

        if (k > 1) {
            b = r1 / r0;
            gpuScaleVectorAndSaxpy(r, p, alpha, b, N, peer_group);
        } else {
            gpuCopyVector(r, p, N, peer_group);
        }

        peer_group.sync();

        // Saxpy 1 End

        // SpMV Start

        gpuSpMV(I, J, val, nnz, N, alpha, p, Ax, peer_group);

        // SpMV End

        // Dot Product 1 Start

        if (peer_group.thread_rank() == 0) {
            *dot_result = 0.0;
        }
        peer_group.sync();

        gpuDotProduct(p, Ax, N, cta, peer_group);

        cg::sync(grid);

        if (grid.thread_rank() == 0) {
            atomicAdd_system(dot_result, grid_dot_result);
            grid_dot_result = 0.0;
        }

        peer_group.sync();

        // Dot Product 1 End

        // Saxpy 2 Start

        a = r1 / *dot_result;

        gpuSaxpy(p, x, a, N, peer_group);

        na = -a;

        gpuSaxpy(Ax, r, na, N, peer_group);

        r0 = r1;

        peer_group.sync();

        // Saxpy 2 End

        // Dot Product 2 Start

        if (peer_group.thread_rank() == 0) {
            *dot_result = 0.0;
        }

        peer_group.sync();

        gpuDotProduct(r, r, N, cta, peer_group);

        cg::sync(grid);

        if (grid.thread_rank() == 0) {
            atomicAdd_system(dot_result, grid_dot_result);
            grid_dot_result = 0.0;
        }
        peer_group.sync();

        // Dot Product 2 End

        // Saxpy 3 Start

        r1 = *dot_result;

        // Saxpy 3 End

        k++;
    }
}

// Map of device version to device number
std::multimap<std::pair<int, int>, int> getIdenticalGPUs() {
    int numGpus = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&numGpus));

    std::multimap<std::pair<int, int>, int> identicalGpus;

    for (int i = 0; i < numGpus; i++) {
        hipDeviceProp_t deviceProp;
        CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, i));

        // Filter unsupported devices
        if (deviceProp.cooperativeLaunch && deviceProp.concurrentManagedAccess) {
            identicalGpus.emplace(std::make_pair(deviceProp.major, deviceProp.minor), i);
        }
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", i, deviceProp.name,
               deviceProp.major, deviceProp.minor);
    }

    return identicalGpus;
}
}  // namespace BaselinePersistentUnifiedMemory

int BaselinePersistentUnifiedMemory::init(int argc, char *argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 10000);
    std::string matrix_path = get_argval<std::string>(argv, argv + argc, "-matrix_path", "");

    int num_devices = 0;

    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    int N = 0, nz = 0, *I = NULL, *J = NULL;
    float *val = NULL;
    const float tol = 1e-5f;
    float *x;
    float rhs = 1.0;
    float r1;
    float *r, *p, *Ax;

    printf("Starting [%s]...\n", BaselinePersistentUnifiedMemory::sSDKname);
    auto gpusByArch = getIdenticalGPUs();

    auto it = gpusByArch.begin();
    auto end = gpusByArch.end();

    auto bestFit = std::make_pair(it, it);
    // use std::distance to find the largest number of GPUs amongst architectures
    auto distance = [](decltype(bestFit) p) { return std::distance(p.first, p.second); };

    // Read each unique key/pair element in order
    for (; it != end; it = gpusByArch.upper_bound(it->first)) {
        // first and second are iterators bounded within the architecture group
        auto testFit = gpusByArch.equal_range(it->first);
        // Always use devices with highest architecture version or whichever has the
        // most devices available
        if (distance(bestFit) <= distance(testFit)) bestFit = testFit;
    }

    if (distance(bestFit) < num_devices) {
        printf(
            "No two or more GPUs with same architecture capable of "
            "concurrentManagedAccess found. "
            "\nWaiving the sample\n");
    }

    std::set<int> bestFitDeviceIds;

    // Check & select peer-to-peer access capable GPU devices as enabling p2p
    // access between participating GPUs gives better performance.
    for (auto itr = bestFit.first; itr != bestFit.second; itr++) {
        int deviceId = itr->second;
        CUDA_RT_CALL(hipSetDevice(deviceId));

        std::for_each(
            itr, bestFit.second,
            [&deviceId, &bestFitDeviceIds, &num_devices](decltype(*itr) mapPair) {
                if (deviceId != mapPair.second) {
                    int access = 0;
                    CUDA_RT_CALL(hipDeviceCanAccessPeer(&access, deviceId, mapPair.second));
                    printf("Device=%d %s Access Peer Device=%d\n", deviceId,
                           access ? "CAN" : "CANNOT", mapPair.second);
                    if (access && bestFitDeviceIds.size() < num_devices) {
                        bestFitDeviceIds.emplace(deviceId);
                        bestFitDeviceIds.emplace(mapPair.second);
                    } else {
                        printf("Ignoring device %i (max devices exceeded)\n", mapPair.second);
                    }
                }
            });

        if (bestFitDeviceIds.size() >= num_devices) {
            printf("Selected p2p capable devices - ");
            for (auto devicesItr = bestFitDeviceIds.begin(); devicesItr != bestFitDeviceIds.end();
                 devicesItr++) {
                printf("deviceId = %d  ", *devicesItr);
            }
            printf("\n");
            break;
        }
    }

    // if bestFitDeviceIds.size() == 0 it means the GPUs in system are not p2p
    // capable, hence we add it without p2p capability check.
    if (!bestFitDeviceIds.size()) {
        printf("Devices involved are not p2p capable.. selecting %zu of them\n", num_devices);
        std::for_each(bestFit.first, bestFit.second,
                      [&bestFitDeviceIds, &num_devices](decltype(*bestFit.first) mapPair) {
                          if (bestFitDeviceIds.size() < num_devices) {
                              bestFitDeviceIds.emplace(mapPair.second);
                          } else {
                              printf("Ignoring device %i (max devices exceeded)\n", mapPair.second);
                          }
                          // Insert the sequence into the deviceIds set
                      });
    } else {
        // perform hipDeviceEnablePeerAccess in both directions for all
        // participating devices.
        for (auto p1_itr = bestFitDeviceIds.begin(); p1_itr != bestFitDeviceIds.end(); p1_itr++) {
            CUDA_RT_CALL(hipSetDevice(*p1_itr));
            for (auto p2_itr = bestFitDeviceIds.begin(); p2_itr != bestFitDeviceIds.end();
                 p2_itr++) {
                if (*p1_itr != *p2_itr) {
                    CUDA_RT_CALL(hipDeviceEnablePeerAccess(*p2_itr, 0));
                    CUDA_RT_CALL(hipSetDevice(*p1_itr));
                }
            }
        }
    }

    /* Generate a random tridiagonal symmetric matrix in CSR format */
    N = 10485760 * 2;
    nz = (N - 2) * 3 + 4;

    CUDA_RT_CALL(hipMallocManaged((void **)&I, sizeof(int) * (N + 1)));
    CUDA_RT_CALL(hipMallocManaged((void **)&J, sizeof(int) * nz));
    CUDA_RT_CALL(hipMallocManaged((void **)&val, sizeof(float) * nz));

    float *val_cpu = (float *)malloc(sizeof(float) * nz);

    genTridiag(I, J, val_cpu, N, nz);

    memcpy(val, val_cpu, sizeof(float) * nz);
    CUDA_RT_CALL(hipMemAdvise(I, sizeof(int) * (N + 1), hipMemAdviseSetReadMostly, 0));
    CUDA_RT_CALL(hipMemAdvise(J, sizeof(int) * nz, hipMemAdviseSetReadMostly, 0));
    CUDA_RT_CALL(hipMemAdvise(val, sizeof(float) * nz, hipMemAdviseSetReadMostly, 0));

    CUDA_RT_CALL(hipMallocManaged((void **)&x, sizeof(float) * N));

    double *dot_result;
    CUDA_RT_CALL(hipMallocManaged((void **)&dot_result, sizeof(double)));

    CUDA_RT_CALL(hipMemset(dot_result, 0, sizeof(double)));

    // temp memory for ConjugateGradient
    CUDA_RT_CALL(hipMallocManaged((void **)&r, N * sizeof(float)));
    CUDA_RT_CALL(hipMallocManaged((void **)&p, N * sizeof(float)));
    CUDA_RT_CALL(hipMallocManaged((void **)&Ax, N * sizeof(float)));

    std::cout << "\nRunning on GPUs = " << num_devices << std::endl;
    hipStream_t nStreams[num_devices];

    int sMemSize = sizeof(double) * ((THREADS_PER_BLOCK / 32) + 1);
    int numBlocksPerSm = INT_MAX;
    int numThreads = THREADS_PER_BLOCK;
    int numSms = INT_MAX;
    auto deviceId = bestFitDeviceIds.begin();

    // set numSms & numBlocksPerSm to be lowest of 2 devices
    while (deviceId != bestFitDeviceIds.end()) {
        hipDeviceProp_t deviceProp;
        CUDA_RT_CALL(hipSetDevice(*deviceId));
        CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, *deviceId));

        int numBlocksPerSm_current = 0;
        CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &numBlocksPerSm_current, multiGpuConjugateGradient, numThreads, sMemSize));

        if (numBlocksPerSm > numBlocksPerSm_current) {
            numBlocksPerSm = numBlocksPerSm_current;
        }
        if (numSms > deviceProp.multiProcessorCount) {
            numSms = deviceProp.multiProcessorCount;
        }
        deviceId++;
    }

    // Added this line to time the different kernel operations
    numBlocksPerSm = 2;

    if (!numBlocksPerSm) {
        printf(
            "Max active blocks per SM is returned as 0.\n Hence, Waiving the "
            "sample\n");
    }

    int device_count = 0;
    int totalThreadsPerGPU = numSms * numBlocksPerSm * THREADS_PER_BLOCK;
    deviceId = bestFitDeviceIds.begin();
    while (deviceId != bestFitDeviceIds.end()) {
        CUDA_RT_CALL(hipSetDevice(*deviceId));
        CUDA_RT_CALL(hipStreamCreate(&nStreams[device_count]));

        int perGPUIter = N / (totalThreadsPerGPU * num_devices);
        int offset_Ax = device_count * totalThreadsPerGPU;
        int offset_r = device_count * totalThreadsPerGPU;
        int offset_p = device_count * totalThreadsPerGPU;
        int offset_x = device_count * totalThreadsPerGPU;

        CUDA_RT_CALL(hipMemPrefetchAsync(I, sizeof(int) * N, *deviceId, nStreams[device_count]));
        CUDA_RT_CALL(
            hipMemPrefetchAsync(val, sizeof(float) * nz, *deviceId, nStreams[device_count]));
        CUDA_RT_CALL(
            hipMemPrefetchAsync(J, sizeof(float) * nz, *deviceId, nStreams[device_count]));

        if (offset_Ax <= N) {
            for (int i = 0; i < perGPUIter; i++) {
                hipMemAdvise(Ax + offset_Ax, sizeof(float) * totalThreadsPerGPU,
                              hipMemAdviseSetPreferredLocation, *deviceId);
                hipMemAdvise(r + offset_r, sizeof(float) * totalThreadsPerGPU,
                              hipMemAdviseSetPreferredLocation, *deviceId);
                hipMemAdvise(x + offset_x, sizeof(float) * totalThreadsPerGPU,
                              hipMemAdviseSetPreferredLocation, *deviceId);
                hipMemAdvise(p + offset_p, sizeof(float) * totalThreadsPerGPU,
                              hipMemAdviseSetPreferredLocation, *deviceId);

                hipMemAdvise(Ax + offset_Ax, sizeof(float) * totalThreadsPerGPU,
                              hipMemAdviseSetAccessedBy, *deviceId);
                hipMemAdvise(r + offset_r, sizeof(float) * totalThreadsPerGPU,
                              hipMemAdviseSetAccessedBy, *deviceId);
                hipMemAdvise(p + offset_p, sizeof(float) * totalThreadsPerGPU,
                              hipMemAdviseSetAccessedBy, *deviceId);
                hipMemAdvise(x + offset_x, sizeof(float) * totalThreadsPerGPU,
                              hipMemAdviseSetAccessedBy, *deviceId);

                offset_Ax += totalThreadsPerGPU * num_devices;
                offset_r += totalThreadsPerGPU * num_devices;
                offset_p += totalThreadsPerGPU * num_devices;
                offset_x += totalThreadsPerGPU * num_devices;

                if (offset_Ax >= N) {
                    break;
                }
            }
        }

        device_count++;
        deviceId++;
    }

#if ENABLE_CPU_DEBUG_CODE
    float *Ax_cpu = (float *)malloc(sizeof(float) * N);
    float *r_cpu = (float *)malloc(sizeof(float) * N);
    float *p_cpu = (float *)malloc(sizeof(float) * N);
    float *x_cpu = (float *)malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
        r_cpu[i] = 1.0;
        Ax_cpu[i] = x_cpu[i] = 0.0;
    }
#endif

    printf("Total threads per GPU = %d numBlocksPerSm  = %d\n",
           numSms * numBlocksPerSm * THREADS_PER_BLOCK, numBlocksPerSm);
    dim3 dimGrid(numSms * numBlocksPerSm, 1, 1), dimBlock(THREADS_PER_BLOCK, 1, 1);

    // Structure used for cross-grid synchronization.
    BaselinePersistentUnifiedMemory::MultiDeviceData multi_device_data;
    CUDA_RT_CALL(hipHostAlloc(&multi_device_data.hostMemoryArrivedList,
                               (num_devices - 1) * sizeof(*multi_device_data.hostMemoryArrivedList),
                               hipHostMallocPortable));
    memset(multi_device_data.hostMemoryArrivedList, 0,
           (num_devices - 1) * sizeof(*multi_device_data.hostMemoryArrivedList));
    multi_device_data.numDevices = num_devices;
    multi_device_data.deviceRank = 0;

    void *kernelArgs[] = {(void *)&I,       (void *)&J, (void *)&val, (void *)&x,
                          (void *)&Ax,      (void *)&p, (void *)&r,   (void *)&dot_result,
                          (void *)&nz,      (void *)&N, (void *)&tol, (void *)&multi_device_data,
                          (void *)&iter_max};

    printf("Launching kernel\n");

    deviceId = bestFitDeviceIds.begin();
    device_count = 0;

    double start = omp_get_wtime();

    while (deviceId != bestFitDeviceIds.end()) {
        CUDA_RT_CALL(hipSetDevice(*deviceId));
        CUDA_RT_CALL(hipLaunchCooperativeKernel((void *)multiGpuConjugateGradient, dimGrid,
                                                 dimBlock, kernelArgs, sMemSize,
                                                 nStreams[device_count++]));
        multi_device_data.deviceRank++;
        deviceId++;
    }

    CUDA_RT_CALL(hipMemPrefetchAsync(x, sizeof(float) * N, hipCpuDeviceId));
    CUDA_RT_CALL(hipMemPrefetchAsync(dot_result, sizeof(double), hipCpuDeviceId));

    deviceId = bestFitDeviceIds.begin();
    device_count = 0;
    while (deviceId != bestFitDeviceIds.end()) {
        CUDA_RT_CALL(hipSetDevice(*deviceId));
        CUDA_RT_CALL(hipStreamSynchronize(nStreams[device_count++]));
        deviceId++;
    }

    r1 = (float)*dot_result;

    double stop = omp_get_wtime();

    printf("Execution time: %8.4f s\n", (stop - start));

    printf("GPU Final, residual = %e \n  ", sqrt(r1));

#if ENABLE_CPU_DEBUG_CODE
    cpuConjugateGrad(I, J, val, x_cpu, Ax_cpu, p_cpu, r_cpu, nz, N, tol);
#endif

    float rsum, diff, err = 0.0;

    for (int i = 0; i < N; i++) {
        rsum = 0.0;

        for (int j = I[i]; j < I[i + 1]; j++) {
            rsum += val_cpu[j] * x[J[j]];
        }

        diff = fabs(rsum - rhs);

        if (diff > err) {
            err = diff;
        }
    }

    CUDA_RT_CALL(hipHostFree(multi_device_data.hostMemoryArrivedList));
    CUDA_RT_CALL(hipFree(I));
    CUDA_RT_CALL(hipFree(J));
    CUDA_RT_CALL(hipFree(val));
    CUDA_RT_CALL(hipFree(x));
    CUDA_RT_CALL(hipFree(r));
    CUDA_RT_CALL(hipFree(p));
    CUDA_RT_CALL(hipFree(Ax));
    CUDA_RT_CALL(hipFree(dot_result));
    free(val_cpu);

#if ENABLE_CPU_DEBUG_CODE
    free(Ax_cpu);
    free(r_cpu);
    free(p_cpu);
    free(x_cpu);
#endif

    printf("Test Summary:  Error amount = %f \n", err);
    fprintf(stdout, "&&&& conjugateGradientMultiDeviceCG %s\n",
            (sqrt(r1) < tol) ? "PASSED" : "FAILED");
    exit((sqrt(r1) < tol) ? EXIT_SUCCESS : EXIT_FAILURE);
}