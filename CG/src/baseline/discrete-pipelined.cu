#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a conjugate gradient solver on multiple GPU using
 * Unified Memory optimized prefetching and usage hints.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <filesystem>
#include <iostream>
#include <map>
#include <set>
#include <utility>

#include <omp.h>

#include "../../include/baseline/discrete-pipelined.cuh"
#include "../../include/common.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

namespace BaselineDiscretePipelined {

// delta => <r, r>
// gamma => <r, w>
__device__ double grid_dot_result_delta = 0.0;
__device__ double grid_dot_result_gamma = 0.0;

// Performs two dot products at the same time
// Used to perform <r, r> and <r, w> at the same time
// Can we combined the two atomicAdds somehow?

__global__ void gpuDotProductsMerged(real *vecA_delta, real *vecB_delta, real *vecA_gamma,
                                     real *vecB_gamma, int num_rows, const int device_rank,
                                     const int num_devices, const int sMemSize) {
    cg::thread_block cta = cg::this_thread_block();

    size_t local_grid_size = gridDim.x * blockDim.x;
    size_t local_grid_rank = blockIdx.x * blockDim.x + threadIdx.x;

    size_t global_grid_size = local_grid_size * num_devices;
    size_t global_grid_rank = device_rank * local_grid_size + local_grid_rank;

    extern __shared__ double tmp[];

    double *tmp_delta = (double *)tmp;
    double *tmp_gamma = (double *)&tmp_delta[sMemSize / (2 * sizeof(double))];

    double temp_sum_delta = 0.0;
    double temp_sum_gamma = 0.0;

    for (size_t i = global_grid_rank; i < num_rows; i += global_grid_size) {
        temp_sum_delta += (double)(vecA_delta[i] * vecB_delta[i]);
        temp_sum_gamma += (double)(vecA_gamma[i] * vecB_gamma[i]);
    }

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    temp_sum_delta = cg::reduce(tile32, temp_sum_delta, cg::plus<double>());
    temp_sum_gamma = cg::reduce(tile32, temp_sum_gamma, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
        tmp_delta[tile32.meta_group_rank()] = temp_sum_delta;
        tmp_gamma[tile32.meta_group_rank()] = temp_sum_gamma;
    }

    cg::sync(cta);

    if (tile32.meta_group_rank() == 0) {
        temp_sum_delta =
            tile32.thread_rank() < tile32.meta_group_size() ? tmp_delta[tile32.thread_rank()] : 0.0;
        temp_sum_delta = cg::reduce(tile32, temp_sum_delta, cg::plus<double>());

        temp_sum_gamma =
            tile32.thread_rank() < tile32.meta_group_size() ? tmp_gamma[tile32.thread_rank()] : 0.0;
        temp_sum_gamma = cg::reduce(tile32, temp_sum_gamma, cg::plus<double>());

        if (tile32.thread_rank() == 0) {
            atomicAdd(&grid_dot_result_delta, temp_sum_delta);
            atomicAdd(&grid_dot_result_gamma, temp_sum_gamma);
        }
    }
}

__global__ void addLocalDotContributions(double *dot_result_delta, double *dot_result_gamma) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid == 0) {
        atomicAdd_system(dot_result_delta, grid_dot_result_delta);
        atomicAdd_system(dot_result_gamma, grid_dot_result_gamma);

        grid_dot_result_delta = 0.0;
        grid_dot_result_gamma = 0.0;
    }
}

__global__ void resetLocalDotProducts(double *dot_result_delta, double *dot_result_gamma) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid == 0) {
        *dot_result_delta = 0.0;
        *dot_result_gamma = 0.0;
    }
}
}  // namespace BaselineDiscretePipelined

int BaselineDiscretePipelined::init(int argc, char *argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 10000);
    std::string matrix_path_str = get_argval<std::string>(argv, argv + argc, "-matrix_path", "");
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare-single-gpu");

    char *matrix_path_char = const_cast<char *>(matrix_path_str.c_str());
    bool generate_random_tridiag_matrix = matrix_path_str.empty();

    std::string matrix_name = std::filesystem::path(matrix_path_str).stem();

    if (generate_random_tridiag_matrix) {
        matrix_name = "random tridiagonal";
    }

    // std::cout << "Running on matrix: " << matrix_name << "\n" << std::endl;

    int num_devices = 0;
    double single_gpu_runtime;

    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    int num_rows = 0;
    int num_cols = 0;
    int nnz = 0;

    int *host_I = NULL;
    int *host_J = NULL;
    real *host_val = NULL;
    real *x_host = NULL;
    real *x_ref_host = NULL;

    int *um_I = NULL;
    int *um_J = NULL;
    real *um_val = NULL;

    real *um_x;
    real *um_r;
    real *um_p;
    real *um_s;
    real *um_z;
    real *um_w;
    real *um_q;
    real *um_ax0;

    double *um_tmp_dot_delta1;
    double *um_tmp_dot_gamma1;
    real *um_tmp_dot_delta0;
    real *um_tmp_dot_gamma0;

    real *um_alpha;
    real *um_negative_alpha;
    real *um_beta;

    real real_positive_one = 1.0;
    real real_negative_one = -1.0;

    hipStream_t streamsOtherOps[num_devices];
    hipStream_t streamsSaxpy[num_devices];
    hipStream_t streamsDot[num_devices];
    hipStream_t streamsSpMV[num_devices];

    for (int gpu_idx_i = 0; gpu_idx_i < num_devices; gpu_idx_i++) {
        CUDA_RT_CALL(hipSetDevice(gpu_idx_i));

        for (int gpu_idx_j = 0; gpu_idx_j < num_devices; gpu_idx_j++) {
            if (gpu_idx_i != gpu_idx_j) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(gpu_idx_j, 0));
                CUDA_RT_CALL(hipSetDevice(gpu_idx_i));
            }
        }
    }

    if (generate_random_tridiag_matrix) {
        num_rows = 10485760 * 2;
        num_cols = num_rows;

        nnz = (num_rows - 2) * 3 + 4;

        CUDA_RT_CALL(hipMallocManaged((void **)&um_I, sizeof(int) * (num_rows + 1)));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_J, sizeof(int) * nnz));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_val, sizeof(real) * nnz));

        host_val = (real *)malloc(sizeof(real) * nnz);

        /* Generate a random tridiagonal symmetric matrix in CSR format */
        genTridiag(um_I, um_J, host_val, num_rows, nnz);

        memcpy(um_val, host_val, sizeof(real) * nnz);

    } else {
        if (loadMMSparseMatrix<real>(matrix_path_char, 'd', true, &num_rows, &num_cols, &nnz,
                                     &host_val, &host_I, &host_J, true)) {
            exit(EXIT_FAILURE);
        }

        CUDA_RT_CALL(hipMallocManaged((void **)&um_I, sizeof(int) * (num_rows + 1)));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_J, sizeof(int) * nnz));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_val, sizeof(real) * nnz));

        memcpy(um_I, host_I, sizeof(int) * (num_rows + 1));
        memcpy(um_J, host_J, sizeof(int) * nnz);
        memcpy(um_val, host_val, sizeof(real) * nnz);
    }

    CUDA_RT_CALL(hipMallocManaged((void **)&um_x, sizeof(real) * num_rows));

    // Comparing to Single GPU Non-Persistent Non-Pipelined implementation
#pragma omp parallel num_threads(num_devices)
    {
        int gpu_idx = omp_get_thread_num();

        if (compare_to_single_gpu && gpu_idx == 0) {
            CUDA_RT_CALL(hipSetDevice(gpu_idx));

            CUDA_RT_CALL(hipHostMalloc(&x_ref_host, num_rows * sizeof(real)));
            CUDA_RT_CALL(hipHostMalloc(&x_host, num_rows * sizeof(real)));

            single_gpu_runtime = SingleGPUStandardDiscrete::run_single_gpu(
                iter_max, um_I, um_J, um_val, x_ref_host, num_rows, nnz);

            // single_gpu_runtime = SingleGPUPipelinedDiscrete::run_single_gpu(
            //     iter_max, um_I, um_J, um_val, x_ref_host, num_rows, nnz);
        }
    }

    CUDA_RT_CALL(hipMallocManaged((void **)&um_x, sizeof(real) * num_rows));

    CUDA_RT_CALL(hipMallocManaged((void **)&um_tmp_dot_delta1, sizeof(double)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_tmp_dot_gamma1, sizeof(double)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_tmp_dot_delta0, sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_tmp_dot_gamma0, sizeof(real)));

    CUDA_RT_CALL(hipMemset(um_tmp_dot_delta1, 0, sizeof(double)));
    CUDA_RT_CALL(hipMemset(um_tmp_dot_gamma1, 0, sizeof(double)));
    CUDA_RT_CALL(hipMemset(um_tmp_dot_delta0, 0, sizeof(real)));
    CUDA_RT_CALL(hipMemset(um_tmp_dot_gamma0, 0, sizeof(real)));

    // temp memory for ConjugateGradient
    CUDA_RT_CALL(hipMallocManaged((void **)&um_r, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_p, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_s, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_z, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_w, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_q, num_rows * sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_ax0, num_rows * sizeof(real)));

    CUDA_RT_CALL(hipMallocManaged((void **)&um_alpha, sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_negative_alpha, sizeof(real)));
    CUDA_RT_CALL(hipMallocManaged((void **)&um_beta, sizeof(real)));

    // ASSUMPTION: All GPUs are the same and P2P callable

    // Multiplying by 2 because the two dot products are merged
    int sMemSize = 2 * (sizeof(double) * ((THREADS_PER_BLOCK / 32) + 1));

    CUDA_RT_CALL(hipSetDevice(0));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int numSms = deviceProp.multiProcessorCount;

    // Structure used for cross-grid synchronization.
    unsigned char *hostMemoryArrivedList;

    CUDA_RT_CALL(hipHostAlloc((void **)&hostMemoryArrivedList,
                               (num_devices - 1) * sizeof(*hostMemoryArrivedList),
                               hipHostMallocPortable));
    memset(hostMemoryArrivedList, 0, (num_devices - 1) * sizeof(*hostMemoryArrivedList));

    int numBlocksInitVectorsPerSM = 0;
    int numBlocksSpmvPerSM = 0;
    int numBlocksSaxpyPerSM = 0;
    int numBlocksDotProductPerSM = 0;
    int numBlocksScaleVectorAndSaxpyPerSM = 0;

    CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksInitVectorsPerSM, MultiGPU::gpuSpMV, THREADS_PER_BLOCK, 0));
    CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksSpmvPerSM, MultiGPU::gpuSpMV, THREADS_PER_BLOCK, 0));
    CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksSaxpyPerSM, MultiGPU::gpuSaxpy, THREADS_PER_BLOCK, 0));
    CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksDotProductPerSM, gpuDotProductsMerged, THREADS_PER_BLOCK, 0));
    CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksScaleVectorAndSaxpyPerSM,
                                                               MultiGPU::gpuScaleVectorAndSaxpy,
                                                               THREADS_PER_BLOCK, 0));

    int initVectorsGridSize = numBlocksInitVectorsPerSM * numSms;
    int spmvGridSize = numBlocksSpmvPerSM * numSms;
    int saxpyGridSize = numBlocksSaxpyPerSM * numSms;
    int dotProductGridSize = numBlocksDotProductPerSM * numSms;
    int scaleVectorAndSaxpyGridSize = numBlocksScaleVectorAndSaxpyPerSM * numSms;

    double start = omp_get_wtime();

#pragma omp parallel num_threads(num_devices)
    {
        int gpu_idx = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(gpu_idx));

        CUDA_RT_CALL(hipStreamCreate(&streamsOtherOps[gpu_idx]));
        CUDA_RT_CALL(hipStreamCreate(&streamsDot[gpu_idx]));
        CUDA_RT_CALL(hipStreamCreate(&streamsSaxpy[gpu_idx]));
        CUDA_RT_CALL(hipStreamCreate(&streamsSpMV[gpu_idx]));

        MultiGPU::
            initVectors<<<initVectorsGridSize, THREADS_PER_BLOCK, 0, streamsOtherOps[gpu_idx]>>>(
                um_r, um_x, num_rows, gpu_idx, num_devices);

        CUDA_RT_CALL(hipDeviceSynchronize());

        // ax0 = Ax0
        MultiGPU::gpuSpMV<<<spmvGridSize, THREADS_PER_BLOCK, 0, streamsOtherOps[gpu_idx]>>>(
            um_I, um_J, um_val, nnz, num_rows, real_positive_one, um_x, um_ax0, gpu_idx,
            num_devices);

        CUDA_RT_CALL(hipDeviceSynchronize());

        // r0 = b0 - ax0
        // NOTE: b is a unit vector.
        MultiGPU::gpuSaxpy<<<saxpyGridSize, THREADS_PER_BLOCK, 0, streamsOtherOps[gpu_idx]>>>(
            um_ax0, um_r, real_negative_one, num_rows, gpu_idx, num_devices);

        CUDA_RT_CALL(hipDeviceSynchronize());

        // w0 = Ar0
        MultiGPU::gpuSpMV<<<spmvGridSize, THREADS_PER_BLOCK, 0, streamsOtherOps[gpu_idx]>>>(
            um_I, um_J, um_val, nnz, num_rows, real_positive_one, um_r, um_w, gpu_idx, num_devices);

        CUDA_RT_CALL(hipDeviceSynchronize());

        int k = 1;

        MultiGPU::syncPeers<<<1, 1, 0, 0>>>(gpu_idx, num_devices, hostMemoryArrivedList);

        while (k <= iter_max) {
            // Two dot products => <r, r> and <r, w>
            resetLocalDotProducts<<<1, 1, 0, streamsDot[gpu_idx]>>>(um_tmp_dot_delta1,
                                                                    um_tmp_dot_gamma1);

            CUDA_RT_CALL(hipStreamSynchronize(streamsDot[gpu_idx]));

            gpuDotProductsMerged<<<dotProductGridSize, THREADS_PER_BLOCK, sMemSize,
                                   streamsDot[gpu_idx]>>>(um_r, um_r, um_r, um_w, num_rows, gpu_idx,
                                                          num_devices, sMemSize);

            CUDA_RT_CALL(hipStreamSynchronize(streamsDot[gpu_idx]));

            addLocalDotContributions<<<1, 1, 0, streamsDot[gpu_idx]>>>(um_tmp_dot_delta1,
                                                                       um_tmp_dot_gamma1);
            CUDA_RT_CALL(hipStreamSynchronize(streamsDot[gpu_idx]));

            // SpMV
            MultiGPU::gpuSpMV<<<spmvGridSize, THREADS_PER_BLOCK, sMemSize, streamsSpMV[gpu_idx]>>>(
                um_I, um_J, um_val, nnz, num_rows, real_positive_one, um_w, um_q, gpu_idx,
                num_devices);

            CUDA_RT_CALL(hipDeviceSynchronize());

            MultiGPU::syncPeers<<<1, 1, 0, 0>>>(gpu_idx, num_devices, hostMemoryArrivedList);

            if (k > 1) {
                update_b_k<<<1, 1, 0, streamsOtherOps[gpu_idx]>>>((real)*um_tmp_dot_delta1,
                                                                  *um_tmp_dot_delta0, um_beta);
                update_a_k<<<1, 1, 0, streamsOtherOps[gpu_idx]>>>(
                    (real)*um_tmp_dot_delta1, (real)*um_tmp_dot_gamma1, *um_beta, um_alpha);
            } else {
                init_b_k<<<1, 1, 0, streamsOtherOps[gpu_idx]>>>(um_beta);
                init_a_k<<<1, 1, 0, streamsOtherOps[gpu_idx]>>>((real)*um_tmp_dot_delta1,
                                                                (real)*um_tmp_dot_gamma1, um_alpha);
            }

            CUDA_RT_CALL(hipDeviceSynchronize());

            MultiGPU::syncPeers<<<1, 1, 0, 0>>>(gpu_idx, num_devices, hostMemoryArrivedList);

            // z_i = q_i + beta_i * z_(i-1)
            MultiGPU::gpuScaleVectorAndSaxpy<<<scaleVectorAndSaxpyGridSize, THREADS_PER_BLOCK, 0,
                                               streamsSaxpy[gpu_idx]>>>(
                um_q, um_z, real_positive_one, *um_beta, num_rows, gpu_idx, num_devices);

            // s_i = w_i + beta_i * s_(i-1)
            MultiGPU::gpuScaleVectorAndSaxpy<<<scaleVectorAndSaxpyGridSize, THREADS_PER_BLOCK, 0,
                                               streamsSaxpy[gpu_idx]>>>(
                um_w, um_s, real_positive_one, *um_beta, num_rows, gpu_idx, num_devices);

            // p_i = r_i = beta_i * p_(i-1)
            MultiGPU::gpuScaleVectorAndSaxpy<<<scaleVectorAndSaxpyGridSize, THREADS_PER_BLOCK, 0,
                                               streamsSaxpy[gpu_idx]>>>(
                um_r, um_p, real_positive_one, *um_beta, num_rows, gpu_idx, num_devices);

            // x_(i+1) = x_i + alpha_i * p_i
            MultiGPU::gpuSaxpy<<<saxpyGridSize, THREADS_PER_BLOCK, 0, streamsSaxpy[gpu_idx]>>>(
                um_p, um_x, *um_alpha, num_rows, gpu_idx, num_devices);

            CUDA_RT_CALL(hipStreamSynchronize(streamsSaxpy[gpu_idx]));

            a_minus<<<1, 1, 0, streamsSaxpy[gpu_idx]>>>(*um_alpha, um_negative_alpha);

            CUDA_RT_CALL(hipStreamSynchronize(streamsSaxpy[gpu_idx]));

            // r_(i+1) = r_i - alpha_i * s_i
            MultiGPU::gpuSaxpy<<<saxpyGridSize, THREADS_PER_BLOCK, 0, streamsSaxpy[gpu_idx]>>>(
                um_s, um_r, *um_negative_alpha, num_rows, gpu_idx, num_devices);

            // w_(i+1) = w_i - alpha_i * z_i
            MultiGPU::gpuSaxpy<<<saxpyGridSize, THREADS_PER_BLOCK, 0, streamsSaxpy[gpu_idx]>>>(
                um_z, um_w, *um_negative_alpha, num_rows, gpu_idx, num_devices);

            CUDA_RT_CALL(hipDeviceSynchronize());

            MultiGPU::syncPeers<<<1, 1, 0, 0>>>(gpu_idx, num_devices, hostMemoryArrivedList);

            *um_tmp_dot_delta0 = (real)*um_tmp_dot_delta1;
            *um_tmp_dot_gamma0 = (real)*um_tmp_dot_gamma1;

            CUDA_RT_CALL(hipDeviceSynchronize());

            MultiGPU::syncPeers<<<1, 1, 0, 0>>>(gpu_idx, num_devices, hostMemoryArrivedList);

#pragma omp barrier

            k++;
        }
    }

    double stop = omp_get_wtime();

#pragma omp parallel num_threads(num_devices)
    {
        int gpu_idx = omp_get_thread_num();

        if (gpu_idx == 0) {
            if (compare_to_single_gpu) {
                for (int i = 0; i < num_rows; i++) {
                    x_host[i] = um_x[i];
                }
            }

            report_results(num_rows, x_ref_host, x_host, num_devices, single_gpu_runtime, start,
                           stop, compare_to_single_gpu);

            CUDA_RT_CALL(hipHostFree(x_host));
            CUDA_RT_CALL(hipHostFree(x_ref_host));
        }
    }

    CUDA_RT_CALL(hipHostFree(hostMemoryArrivedList));
    CUDA_RT_CALL(hipFree(um_I));
    CUDA_RT_CALL(hipFree(um_J));
    CUDA_RT_CALL(hipFree(um_val));
    CUDA_RT_CALL(hipFree(um_x));
    CUDA_RT_CALL(hipFree(um_r));
    CUDA_RT_CALL(hipFree(um_p));
    CUDA_RT_CALL(hipFree(um_s));
    CUDA_RT_CALL(hipFree(um_tmp_dot_delta0));
    CUDA_RT_CALL(hipFree(um_tmp_dot_delta1));
    CUDA_RT_CALL(hipFree(um_tmp_dot_gamma0));
    CUDA_RT_CALL(hipFree(um_tmp_dot_gamma1));
    free(host_val);

    return 0;
}
