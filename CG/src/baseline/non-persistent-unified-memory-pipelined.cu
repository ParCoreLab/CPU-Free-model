#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a conjugate gradient solver on multiple GPU using
 * Unified Memory optimized prefetching and usage hints.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <filesystem>
#include <iostream>
#include <map>
#include <set>
#include <utility>

#include <omp.h>

#include "../../include/baseline/non-persistent-unified-memory-pipelined.cuh"
#include "../../include/common.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

namespace BaselineNonPersistentUnifiedMemoryPipelined {

#define ENABLE_CPU_DEBUG_CODE 0
#define THREADS_PER_BLOCK 512

__device__ double grid_dot_result = 0.0;

__global__ void initVectors(float *rhs, float *x, int num_rows, const int device_rank,
                            const int num_devices) {
    size_t local_grid_size = gridDim.x * blockDim.x;
    size_t local_grid_rank = blockIdx.x * blockDim.x + threadIdx.x;

    size_t global_grid_size = local_grid_size * num_devices;
    size_t global_grid_rank = device_rank * local_grid_size + local_grid_rank;

    for (size_t i = global_grid_rank; i < num_rows; i += global_grid_size) {
        rhs[i] = 1.0;
        x[i] = 0.0;
    }
}

__global__ void gpuSpMV(int *I, int *J, float *val, int nnz, int num_rows, float alpha,
                        float *inputVecX, float *outputVecY, const int device_rank,
                        const int num_devices) {
    size_t local_grid_size = gridDim.x * blockDim.x;
    size_t local_grid_rank = blockIdx.x * blockDim.x + threadIdx.x;

    size_t global_grid_size = local_grid_size * num_devices;
    size_t global_grid_rank = device_rank * local_grid_size + local_grid_rank;

    for (size_t i = global_grid_rank; i < num_rows; i += global_grid_size) {
        int row_elem = I[i];
        int next_row_elem = I[i + 1];
        int num_elems_this_row = next_row_elem - row_elem;

        float output = 0.0;
        for (int j = 0; j < num_elems_this_row; j++) {
            output += alpha * val[row_elem + j] * inputVecX[J[row_elem + j]];
        }

        outputVecY[i] = output;
    }
}

__global__ void gpuSaxpy(float *x, float *y, float a, int size, const int device_rank,
                         const int num_devices) {
    size_t local_grid_size = gridDim.x * blockDim.x;
    size_t local_grid_rank = blockIdx.x * blockDim.x + threadIdx.x;

    size_t global_grid_size = local_grid_size * num_devices;
    size_t global_grid_rank = device_rank * local_grid_size + local_grid_rank;

    for (size_t i = global_grid_rank; i < num_rows; i += global_grid_size) {
        y[i] = a * x[i] + y[i];
    }
}

__global__ void gpuDotProduct(float *vecA, float *vecB, int num_rows, const int device_rank,
                              const int num_devices) {
    cg::thread_block cta = cg::this_thread_block();

    size_t local_grid_size = gridDim.x * blockDim.x;
    size_t local_grid_rank = blockIdx.x * blockDim.x + threadIdx.x;

    size_t global_grid_size = local_grid_size * num_devices;
    size_t global_grid_rank = device_rank * local_grid_size + local_grid_rank;

    extern __shared__ double tmp[];

    double temp_sum = 0.0;

    for (size_t i = global_grid_rank; i < num_rows; i += global_grid_size) {
        temp_sum += (double)(vecA[i] * vecB[i]);
    }

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
        tmp[tile32.meta_group_rank()] = temp_sum;
    }

    cg::sync(cta);

    if (tile32.meta_group_rank() == 0) {
        temp_sum =
            tile32.thread_rank() < tile32.meta_group_size() ? tmp[tile32.thread_rank()] : 0.0;
        temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

        if (tile32.thread_rank() == 0) {
            atomicAdd(&grid_dot_result, temp_sum);
        }
    }
}

__global__ void gpuCopyVector(float *srcA, float *destB, int size, const int device_rank,
                              const int num_devices) {
    size_t local_grid_size = gridDim.x * blockDim.x;
    size_t local_grid_rank = blockIdx.x * blockDim.x + threadIdx.x;

    size_t global_grid_size = local_grid_size * num_devices;
    size_t global_grid_rank = device_rank * local_grid_size + local_grid_rank;

    for (size_t i = global_grid_rank; i < num_rows; i += global_grid_size) {
        destB[i] = srcA[i];
    }
}

__global__ void gpuScaleVectorAndSaxpy(float *x, float *y, float a, float scale, int size,
                                       const int device_rank, const int num_devices) {
    size_t local_grid_size = gridDim.x * blockDim.x;
    size_t local_grid_rank = blockIdx.x * blockDim.x + threadIdx.x;

    size_t global_grid_size = local_grid_size * num_devices;
    size_t global_grid_rank = device_rank * local_grid_size + local_grid_rank;

    for (size_t i = global_grid_rank; i < num_rows; i += global_grid_size) {
        y[i] = a * x[i] + scale * y[i];
    }
}
}  // namespace BaselineNonPersistentUnifiedMemoryPipelined

int BaselineNonPersistentUnifiedMemoryPipelined::init(int argc, char *argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 10000);
    std::string matrix_path_str = get_argval<std::string>(argv, argv + argc, "-matrix_path", "");
    const bool compare_to_cpu = get_arg(argv, argv + argc, "-compare");

    char *matrix_path_char = const_cast<char *>(matrix_path_str.c_str());
    bool generate_random_tridiag_matrix = matrix_path_str.empty();

    std::string matrix_name = std::filesystem::path(matrix_path_str).stem();

    if (generate_random_tridiag_matrix) {
        matrix_name = "random tridiagonal";
    }

    // std::cout << "Running on matrix: " << matrix_name << "\n" << std::endl;

    int num_devices = 0;

    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    int num_rows = 0;
    int num_cols = 0;
    int nnz = 0;

    int *host_I = NULL;
    int *host_J = NULL;
    float *host_val = NULL;

    int *um_I = NULL;
    int *um_J = NULL;
    float *um_val = NULL;

    const float tol = 1e-5f;
    float *x;
    float rhs = 1.0;
    float r1;
    float *r, *p, *Ax;

    hipStream_t streamDefault[num_devices];
    hipStream_t streamSaxpy[num_devices];
    hipStream_t streamDot[num_devices];
    hipStream_t streamSpMV[num_devices];

    for (int gpu_idx_i = 0; gpu_idx_i < num_devices; gpu_idx_i++) {
        CUDA_RT_CALL(hipSetDevice(gpu_idx_i));

        for (int gpu_idx_j = 0; gpu_idx_j < num_devices; gpu_idx_j++) {
            if (gpu_idx_i != gpu_idx_j) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(gpu_idx_j, 0));
                CUDA_RT_CALL(hipSetDevice(gpu_idx_i));
            }
        }
    }

    if (generate_random_tridiag_matrix) {
        num_rows = 10485760 * 2;
        num_cols = num_rows;

        nnz = (num_rows - 2) * 3 + 4;

        CUDA_RT_CALL(hipMallocManaged((void **)&um_I, sizeof(int) * (num_rows + 1)));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_J, sizeof(int) * nnz));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_val, sizeof(float) * nnz));

        host_val = (float *)malloc(sizeof(float) * nnz);

        /* Generate a random tridiagonal symmetric matrix in CSR format */
        genTridiag(um_I, um_J, host_val, num_rows, nnz);

        memcpy(um_val, host_val, sizeof(float) * nnz);

    } else {
        if (loadMMSparseMatrix<float>(matrix_path_char, 'd', true, &num_rows, &num_cols, &nnz,
                                      &host_val, &host_I, &host_J, true)) {
            exit(EXIT_FAILURE);
        }

        CUDA_RT_CALL(hipMallocManaged((void **)&um_I, sizeof(int) * (num_rows + 1)));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_J, sizeof(int) * nnz));
        CUDA_RT_CALL(hipMallocManaged((void **)&um_val, sizeof(float) * nnz));

        memcpy(um_I, host_I, sizeof(int) * (num_rows + 1));
        memcpy(um_J, host_J, sizeof(int) * nnz);
        memcpy(um_val, host_val, sizeof(float) * nnz);
    }

    CUDA_RT_CALL(hipMallocManaged((void **)&x, sizeof(float) * num_rows));

    double *dot_result;
    CUDA_RT_CALL(hipMallocManaged((void **)&dot_result, sizeof(double)));

    CUDA_RT_CALL(hipMemset(dot_result, 0, sizeof(double)));

    // temp memory for ConjugateGradient
    CUDA_RT_CALL(hipMallocManaged((void **)&r, num_rows * sizeof(float)));
    CUDA_RT_CALL(hipMallocManaged((void **)&p, num_rows * sizeof(float)));
    CUDA_RT_CALL(hipMallocManaged((void **)&Ax, num_rows * sizeof(float)));

    float *d_r1, *d_r0, *d_dot, *d_a, *d_na, *d_b;
    checkCudaErrors(hipMallocManaged((void **)&d_r1, sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&d_r0, sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&d_dot, sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&d_a, sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&d_na, sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&d_b, sizeof(float)));

    // ASSUMPTION: All GPUs are the same and P2P callable

    hipStream_t nStreams[num_devices];

    int sMemSize = sizeof(double) * ((THREADS_PER_BLOCK / 32) + 1);
    int numThreads = THREADS_PER_BLOCK;

#if ENABLE_CPU_DEBUG_CODE
    float *Ax_cpu = (float *)malloc(sizeof(float) * N);
    float *r_cpu = (float *)malloc(sizeof(float) * N);
    float *p_cpu = (float *)malloc(sizeof(float) * N);
    float *x_cpu = (float *)malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
        r_cpu[i] = 1.0;
        Ax_cpu[i] = x_cpu[i] = 0.0;
    }
#endif

    dim3 dimGrid(numSms * numBlocksPerSm, 1, 1), dimBlock(numThreads, 1, 1);

    double start = omp_get_wtime();

    float alpha = 1.0;
    float alpham1 = -1.0;
    float beta = 0.0;

    int numBlocksInitVectors = 0;
    int numBlocksSpmv = 0;
    int numBlocksSaxpy = 0;
    int numBlocksDotProduct = 0;

    int blockSizeInitVectors = 0;

    checkCudaErrors(
        hipOccupancyMaxPotentialBlockSize(&numBlocksInitVectors, &blockSize, initVectors));
    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&numBlocksSpmv, THREADS_PER_BLOCK, gpuSpMV));
    checkCudaErrors(
        hipOccupancyMaxPotentialBlockSize(&numBlocksSaxpy, THREADS_PER_BLOCK, gpuSaxpy));
    checkCudaErrors(
        hipOccupancyMaxPotentialBlockSize(&numBlocksDotProduct, THREADS_PER_BLOCK, gpuDotProduct));

    for (int gpu_idx = 0; gpu_idx < num_devices; gpu_idx++) {
        initVectors<<<numBlocksInitVectors, blockSize, 0, stream1>>>(r, x, N, gpu_idx, num_devices);

        gpuSpMV<<<numBlocks, THREADS_PER_BLOCK, 0, streamDefault>>>(
            um_I, um_J, um_val, nnz, num_rows, alpha, x, Ax, gpu_idx, num_devices);

        gpuSaxpy<<<numBlocksSaxpy, THREADS_PER_BLOCK, 0, streamDefault>>>(Ax, r, alpham1, num_rows,
                                                                          gpu_idx, num_devices);

        gpuDotProduct<<<numBlocksDotProduct, THREADS_PER_BLOCK, 0, streamDefault>>>(
            r, r, num_rows, gpu_idx, num_devices);
    }
}

// for (int gpu_idx = 0; gpu_idx < num_devices; gpu_idx++) {
//     CUDA_RT_CALL(hipSetDevice(gpu_idx));
//     CUDA_RT_CALL(hipStreamSynchronize(nStreams[gpu_idx]));
// }

r1 = (float)*dot_result;

double stop = omp_get_wtime();

printf("Execution time: %8.4f s\n", (stop - start));

#if ENABLE_CPU_DEBUG_CODE
cpuConjugateGrad(I, J, val, x_cpu, Ax_cpu, p_cpu, r_cpu, nz, N, tol);
#endif

float rsum, diff, err = 0.0;

for (int i = 0; i < num_rows; i++) {
    rsum = 0.0;

    for (int j = um_I[i]; j < um_J[i + 1]; j++) {
        rsum += host_val[j] * x[um_J[j]];
    }

    diff = fabs(rsum - rhs);

    if (diff > err) {
        err = diff;
    }
}

CUDA_RT_CALL(hipHostFree(multi_device_data.hostMemoryArrivedList));
CUDA_RT_CALL(hipFree(um_I));
CUDA_RT_CALL(hipFree(um_J));
CUDA_RT_CALL(hipFree(um_val));
CUDA_RT_CALL(hipFree(x));
CUDA_RT_CALL(hipFree(r));
CUDA_RT_CALL(hipFree(p));
CUDA_RT_CALL(hipFree(Ax));
CUDA_RT_CALL(hipFree(dot_result));
free(host_val);

#if ENABLE_CPU_DEBUG_CODE
free(Ax_cpu);
free(r_cpu);
free(p_cpu);
free(x_cpu);
#endif

if (compare_to_cpu) {
    printf("GPU Final, residual = %e \n  ", sqrt(r1));
    printf("Test Summary:  Error amount = %f \n", err);
    fprintf(stdout, "&&&& conjugateGradientMultiDeviceCG %s\n",
            (sqrt(r1) < tol) ? "PASSED" : "FAILED");
}

return 0;
}