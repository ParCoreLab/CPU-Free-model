#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a conjugate gradient solver on multiple GPU using
 * Unified Memory optimized prefetching and usage hints.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <map>
#include <set>
#include <utility>

#include <omp.h>

#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>

#include "../../include_nvshmem/common.h"
#include "../../include_nvshmem/profiling/discrete-pipelined-nvshmem.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

namespace ProfilingDiscretePipelinedNVSHMEM {

__global__ void gpuDotProductsMerged(real *vecA_delta, real *vecB_delta, real *vecA_gamma,
                                     real *vecB_gamma, double *local_dot_result_delta,
                                     double *local_dot_result_gamma, int chunk_size,
                                     const int sMemSize) {
    cg::thread_block cta = cg::this_thread_block();

    size_t grid_rank = blockIdx.x * blockDim.x + threadIdx.x;
    size_t grid_size = gridDim.x * blockDim.x;

    extern __shared__ double tmp[];

    double *tmp_delta = (double *)tmp;
    double *tmp_gamma = (double *)&tmp_delta[sMemSize / (2 * sizeof(double))];

    double temp_sum_delta = 0.0;
    double temp_sum_gamma = 0.0;

    for (size_t i = grid_rank; i < chunk_size; i += grid_size) {
        temp_sum_delta += (double)(vecA_delta[i] * vecB_delta[i]);
        temp_sum_gamma += (double)(vecA_gamma[i] * vecB_gamma[i]);
    }

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    temp_sum_delta = cg::reduce(tile32, temp_sum_delta, cg::plus<double>());
    temp_sum_gamma = cg::reduce(tile32, temp_sum_gamma, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
        tmp_delta[tile32.meta_group_rank()] = temp_sum_delta;
        tmp_gamma[tile32.meta_group_rank()] = temp_sum_gamma;
    }

    cg::sync(cta);

    if (tile32.meta_group_rank() == 0) {
        temp_sum_delta =
            tile32.thread_rank() < tile32.meta_group_size() ? tmp_delta[tile32.thread_rank()] : 0.0;
        temp_sum_delta = cg::reduce(tile32, temp_sum_delta, cg::plus<double>());

        temp_sum_gamma =
            tile32.thread_rank() < tile32.meta_group_size() ? tmp_gamma[tile32.thread_rank()] : 0.0;
        temp_sum_gamma = cg::reduce(tile32, temp_sum_gamma, cg::plus<double>());

        if (tile32.thread_rank() == 0) {
            atomicAdd(local_dot_result_delta, temp_sum_delta);
            atomicAdd(local_dot_result_gamma, temp_sum_gamma);
        }
    }
}

__global__ void resetLocalDotProducts(double *dot_result_delta, double *dot_result_gamma) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid == 0) {
        *dot_result_delta = 0.0;
        *dot_result_gamma = 0.0;
    }
}

}  // namespace ProfilingDiscretePipelinedNVSHMEM

int ProfilingDiscretePipelinedNVSHMEM::init(int argc, char *argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 10000);
    std::string matrix_path_str = get_argval<std::string>(argv, argv + argc, "-matrix_path", "");
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare-single-gpu");
    const bool compare_to_cpu = get_arg(argv, argv + argc, "-compare-cpu");

    char *matrix_path_char = const_cast<char *>(matrix_path_str.c_str());
    bool generate_random_tridiag_matrix = matrix_path_str.empty();

    int num_devices = 0;
    double single_gpu_runtime;

    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    int num_rows = 0;
    int num_cols = 0;
    int nnz = 0;

    int *host_I = NULL;
    int *host_J = NULL;
    real *host_val = NULL;

    real *x_ref_single_gpu = NULL;
    real *x_final_result = NULL;

    real *s_cpu = NULL;
    real *r_cpu = NULL;
    real *p_cpu = NULL;
    real *x_ref_cpu = NULL;

    int *device_I = NULL;
    int *device_J = NULL;
    real *device_val = NULL;

    real *device_x;
    real *device_r;
    real *device_p;
    real *device_s;
    real *device_z;
    real *device_w;
    real *device_q;
    real *device_ax0;

    real alpha;
    real negative_alpha;
    real beta;

    real tmp_dot_delta0;

    double *device_dot_delta1;
    double *device_dot_gamma1;
    double host_dot_gamma1;
    double host_dot_delta1;

    real real_positive_one = 1.0;
    real real_negative_one = -1.0;

    int rank = 0, size = 1;
    MPI_CALL(MPI_Init(&argc, &argv));
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &size));

    int local_rank = -1;
    int local_size = 1;
    {
        MPI_Comm local_comm;
        MPI_CALL(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
                                     &local_comm));

        MPI_CALL(MPI_Comm_rank(local_comm, &local_rank));
        MPI_CALL(MPI_Comm_size(local_comm, &local_size));

        MPI_CALL(MPI_Comm_free(&local_comm));
    }

    if (1 < num_devices && num_devices < local_size) {
        fprintf(stderr,
                "ERROR Number of visible devices (%d) is less than number of ranks on the "
                "node (%d)!\n",
                num_devices, local_size);
        MPI_CALL(MPI_Finalize());
        return 1;
    }

    if (1 == num_devices) {
        // Only 1 device visible, assuming GPU affinity is handled via CUDA_VISIBLE_DEVICES
        CUDA_RT_CALL(hipSetDevice(0));
    } else {
        CUDA_RT_CALL(hipSetDevice(local_rank));
    }

    CUDA_RT_CALL(hipFree(0));

    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr;

    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;

    if (generate_random_tridiag_matrix) {
        num_rows = 10485760 * 2;
        num_cols = num_rows;

        nnz = (num_rows - 2) * 3 + 4;

        host_I = (int *)malloc(sizeof(int) * (num_rows + 1));
        host_J = (int *)malloc(sizeof(int) * nnz);
        host_val = (real *)malloc(sizeof(real) * nnz);

        /* Generate a random tridiagonal symmetric matrix in CSR format */
        genTridiag(host_I, host_J, host_val, num_rows, nnz);
    } else {
        if (loadMMSparseMatrix<real>(matrix_path_char, 'd', true, &num_rows, &num_cols, &nnz,
                                     &host_val, &host_I, &host_J, true)) {
            exit(EXIT_FAILURE);
        }
    }

    CUDA_RT_CALL(hipMalloc((void **)&device_I, sizeof(int) * (num_rows + 1)));
    CUDA_RT_CALL(hipMalloc((void **)&device_J, sizeof(int) * nnz));
    CUDA_RT_CALL(hipMalloc((void **)&device_val, sizeof(real) * nnz));

    CUDA_RT_CALL(
        hipMemcpy(device_I, host_I, sizeof(int) * (num_rows + 1), hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(device_J, host_J, sizeof(int) * nnz, hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(device_val, host_val, sizeof(real) * nnz, hipMemcpyHostToDevice));

    // Set symmetric heap size for nvshmem based on problem size
    // Its default value in nvshmem is 1 GB which is not sufficient
    // for large mesh sizes
    long long unsigned int mesh_size_per_rank = num_rows / size + (num_rows % size != 0);

    long long unsigned int required_symmetric_heap_size =
        8 * mesh_size_per_rank * sizeof(real) * 1.1;

    char *value = getenv("NVSHMEM_SYMMETRIC_SIZE");
    if (value) { /* env variable is set */
        long long unsigned int size_env = parse_nvshmem_symmetric_size(value);
        if (size_env < required_symmetric_heap_size) {
            fprintf(stderr,
                    "ERROR: Minimum NVSHMEM_SYMMETRIC_SIZE = %lluB, Current "
                    "NVSHMEM_SYMMETRIC_SIZE=%s\n",
                    required_symmetric_heap_size, value);
            MPI_CALL(MPI_Finalize());
            return -1;
        }
    } else {
        char symmetric_heap_size_str[100];
        sprintf(symmetric_heap_size_str, "%llu", required_symmetric_heap_size);

        // if (rank == 0) {
        //     printf("Setting environment variable NVSHMEM_SYMMETRIC_SIZE = %llu\n",
        //            required_symmetric_heap_size);
        // }

        setenv("NVSHMEM_SYMMETRIC_SIZE", symmetric_heap_size_str, 1);
    }
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

    int npes = nvshmem_n_pes();
    int mype = nvshmem_my_pe();

    nvshmem_barrier_all();

    hipStream_t mainStream;

    CUDA_RT_CALL(hipStreamCreateWithFlags(&mainStream, hipStreamNonBlocking));

    nvshmem_barrier_all();

    // Load balancing this way isn't ideal
    // On kernel side, we need to calculate PE element belong to
    // Naive load balancing like this makes PE calculation on kernel side easier
    int chunk_size = num_rows / npes + (num_rows % npes != 0);

    device_x = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_r = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_p = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_s = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_z = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_w = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_q = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_ax0 = (real *)nvshmem_malloc(chunk_size * sizeof(real));

    CUDA_RT_CALL(hipMemset(device_x, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_r, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_p, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_s, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_z, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_w, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_q, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_ax0, 0, chunk_size * sizeof(real)));

    device_dot_delta1 = (double *)nvshmem_malloc(sizeof(double));
    device_dot_gamma1 = (double *)nvshmem_malloc(sizeof(double));

    CUDA_RT_CALL(hipMemset(device_dot_delta1, 0, sizeof(double)));
    CUDA_RT_CALL(hipMemset(device_dot_gamma1, 0, sizeof(double)));

    // Calculate local domain boundaries
    int row_start_global_idx = mype * chunk_size;      // My start index in the global array
    int row_end_global_idx = (mype + 1) * chunk_size;  // My end index in the global array

    row_end_global_idx = std::min(row_end_global_idx, num_rows);

    if (compare_to_single_gpu) {
        CUDA_RT_CALL(hipHostMalloc(&x_ref_single_gpu, num_rows * sizeof(real)));

        single_gpu_runtime = SingleGPUDiscreteStandard::run_single_gpu(
            iter_max, device_I, device_J, device_val, x_ref_single_gpu, num_rows, nnz);

        // single_gpu_runtime = SingleGPUDiscretePipelined::run_single_gpu(
        //     iter_max, device_I, device_J, device_val, x_ref_single_gpu, num_rows, nnz);
    }

    if (compare_to_cpu) {
        s_cpu = (real *)malloc(sizeof(real) * num_rows);
        r_cpu = (real *)malloc(sizeof(real) * num_rows);
        p_cpu = (real *)malloc(sizeof(real) * num_rows);

        CUDA_RT_CALL(hipHostMalloc(&x_ref_cpu, num_rows * sizeof(real)));

        for (int i = 0; i < num_rows; i++) {
            r_cpu[i] = 1.0;
            s_cpu[i] = 0.0;
            x_ref_cpu[i] = 0.0;
        }

        CPU::cpuConjugateGrad(iter_max, host_I, host_J, host_val, x_ref_cpu, s_cpu, p_cpu, r_cpu,
                              nnz, num_rows, tol);
    }

    CUDA_RT_CALL(hipDeviceSynchronize());
    nvshmem_barrier_all();

    int sMemSize = 2 * sizeof(double) * ((THREADS_PER_BLOCK / 32) + 1);
    int numBlocks = (chunk_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    nvshmem_barrier_all();

    double start = MPI_Wtime();

    NVSHMEM::initVectors<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
        device_r, device_x, row_start_global_idx, chunk_size, num_rows);

    nvshmemx_barrier_all_on_stream(mainStream);

    // ax0 = Ax0
    NVSHMEM::gpuSpMV<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
        device_I, device_J, device_val, real_positive_one, device_x, device_ax0,
        row_start_global_idx, chunk_size, num_rows);

    nvshmemx_barrier_all_on_stream(mainStream);

    // r0 = b0 - ax0
    // NOTE: b is a unit vector.
    NVSHMEM::gpuSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
        device_ax0, device_r, real_negative_one, chunk_size);

    nvshmemx_barrier_all_on_stream(mainStream);

    // w0 = Ar0
    NVSHMEM::gpuSpMV<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
        device_I, device_J, device_val, real_positive_one, device_r, device_w, row_start_global_idx,
        chunk_size, num_rows);

    nvshmemx_barrier_all_on_stream(mainStream);

    int k = 1;

    while (k <= iter_max) {
        PUSH_RANGE("Merged Dots (+Reset)", 0);

        resetLocalDotProducts<<<1, 1, 0, mainStream>>>(device_dot_delta1, device_dot_gamma1);

        // Dot
        gpuDotProductsMerged<<<numBlocks, THREADS_PER_BLOCK, sMemSize, mainStream>>>(
            device_r, device_r, device_r, device_w, device_dot_delta1, device_dot_gamma1,
            chunk_size, sMemSize);

        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        POP_RANGE

        PUSH_RANGE("SpMV", 1);

        // SpMV
        NVSHMEM::gpuSpMV<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
            device_I, device_J, device_val, real_positive_one, device_w, device_q,
            row_start_global_idx, chunk_size, num_rows);

        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        POP_RANGE

        PUSH_RANGE("NVSHMEM Barrier 1 (After SpMV)", 2);

        nvshmemx_barrier_all_on_stream(mainStream);

        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        POP_RANGE

        // NOTE: Instead of doing this could have the local dots be in contiguous locations
        // And the use the same NVSHMEM call to do both at the same time

        PUSH_RANGE("Global Reductions (+Barrier)", 3);

        nvshmemx_double_sum_reduce_on_stream(NVSHMEM_TEAM_WORLD, device_dot_delta1,
                                             device_dot_delta1, 1, mainStream);

        nvshmemx_double_sum_reduce_on_stream(NVSHMEM_TEAM_WORLD, device_dot_gamma1,
                                             device_dot_gamma1, 1, mainStream);

        // Using nvshmem_barrier_all() here seems to cause a deadlock
        // Wonder why?
        // Because two reductions are enqued to same stream back to back?
        // In any case, should use one contiguous array for reductions
        nvshmemx_barrier_all_on_stream(mainStream);

        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        POP_RANGE

        PUSH_RANGE("Memcpy Dots To Host", 4);

        CUDA_RT_CALL(hipMemcpyAsync(&host_dot_delta1, device_dot_delta1, sizeof(double),
                                     hipMemcpyDeviceToHost, mainStream));

        CUDA_RT_CALL(hipMemcpyAsync(&host_dot_gamma1, device_dot_gamma1, sizeof(double),
                                     hipMemcpyDeviceToHost, mainStream));

        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        POP_RANGE

        real real_tmp_dot_delta1 = (real)host_dot_delta1;
        real real_tmp_dot_gamma1 = (real)host_dot_gamma1;

        if (k > 1) {
            beta = real_tmp_dot_delta1 / tmp_dot_delta0;
            alpha =
                real_tmp_dot_delta1 / (real_tmp_dot_gamma1 - (beta / alpha) * real_tmp_dot_delta1);
        } else {
            beta = 0.0;
            alpha = real_tmp_dot_delta1 / real_tmp_dot_gamma1;
        }

        PUSH_RANGE("Saxpy 1", 5);

        // z_k = q_k + beta_k * z_(k-1)
        NVSHMEM::gpuScaleVectorAndSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
            device_q, device_z, real_positive_one, beta, chunk_size);

        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        POP_RANGE

        PUSH_RANGE("Saxpy 2", 6);

        // s_k = w_k + beta_k * s_(k-1)
        NVSHMEM::gpuScaleVectorAndSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
            device_w, device_s, real_positive_one, beta, chunk_size);

        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        POP_RANGE

        PUSH_RANGE("Saxpy 3", 7);

        // p_k = r_k = beta_k * p_(k-1)
        NVSHMEM::gpuScaleVectorAndSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
            device_r, device_p, real_positive_one, beta, chunk_size);

        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        POP_RANGE

        PUSH_RANGE("Saxpy 4", 8);

        // x_(k+1) = x_k + alpha_k * p_k
        NVSHMEM::gpuSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(device_p, device_x,
                                                                           alpha, chunk_size);

        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        POP_RANGE

        negative_alpha = -alpha;

        PUSH_RANGE("Saxpy 5", 9);

        // r_(k+1) = r_k - alpha_k * s_k
        NVSHMEM::gpuSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
            device_s, device_r, negative_alpha, chunk_size);

        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        POP_RANGE

        PUSH_RANGE("Saxpy 6", 10);

        // w_(k+1) = w_k - alpha_k * z_k
        NVSHMEM::gpuSaxpy<<<numBlocks, THREADS_PER_BLOCK, 0, mainStream>>>(
            device_z, device_w, negative_alpha, chunk_size);

        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        POP_RANGE

        tmp_dot_delta0 = real_tmp_dot_delta1;

        PUSH_RANGE("NVSHMEM Barrier 2 (End of Iteration)", 11);

        nvshmemx_barrier_all_on_stream(mainStream);
        CUDA_RT_CALL(hipStreamSynchronize(mainStream));

        POP_RANGE

        k++;
    }

    nvshmemx_barrier_all_on_stream(mainStream);
    CUDA_RT_CALL(hipDeviceSynchronize());

    double stop = MPI_Wtime();

    if (compare_to_single_gpu || compare_to_cpu) {
        CUDA_RT_CALL(hipHostMalloc(&x_final_result, num_rows * sizeof(real)));

        // Need to do this when when num_rows % npes != 0
        int num_elems_to_copy = row_end_global_idx - row_start_global_idx;

        CUDA_RT_CALL(hipMemcpy(x_final_result + row_start_global_idx, device_x,
                                num_elems_to_copy * sizeof(real), hipMemcpyDeviceToHost));
    }

    bool result_correct_single_gpu = true;
    bool result_correct_cpu = true;

    report_errors(num_rows, x_ref_single_gpu, x_ref_cpu, x_final_result, row_start_global_idx,
                  row_end_global_idx, npes, single_gpu_runtime, start, stop, compare_to_single_gpu,
                  compare_to_cpu, result_correct_single_gpu, result_correct_cpu);

    nvshmem_barrier_all();

    if (mype == 0) {
        report_runtime(npes, single_gpu_runtime, start, stop, result_correct_single_gpu,
                       result_correct_cpu, compare_to_single_gpu);
    }

    nvshmem_free(device_x);
    nvshmem_free(device_r);
    nvshmem_free(device_p);
    nvshmem_free(device_s);
    nvshmem_free(device_z);
    nvshmem_free(device_w);
    nvshmem_free(device_q);
    nvshmem_free(device_ax0);

    nvshmem_free(device_dot_delta1);
    nvshmem_free(device_dot_gamma1);

    CUDA_RT_CALL(hipStreamDestroy(mainStream));

    CUDA_RT_CALL(hipFree(device_I));
    CUDA_RT_CALL(hipFree(device_J));
    CUDA_RT_CALL(hipFree(device_val));

    free(host_I);
    free(host_J);
    free(host_val);

    if (compare_to_single_gpu || compare_to_cpu) {
        hipHostFree(x_final_result);

        if (compare_to_single_gpu) {
            hipHostFree(x_ref_single_gpu);
        }

        if (compare_to_cpu) {
            hipHostFree(x_ref_cpu);
        }
    }

    nvshmem_finalize();
    MPI_CALL(MPI_Finalize());

    return 0;
}