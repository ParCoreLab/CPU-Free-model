#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a conjugate gradient solver on multiple GPU using
 * Unified Memory optimized prefetching and usage hints.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <map>
#include <set>
#include <utility>

#include <omp.h>

#include "../../include_nvshmem/common.h"
#include "../../include_nvshmem/single-stream/pipelined-nvshmem.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>

namespace cg = cooperative_groups;

namespace SingleStreamPipelinedNVSHMEM {

__device__ void gpuSpMV(int *I, int *J, real *val, real alpha, real *inputVecX, real *outputVecY,
                        int row_start_idx, int chunk_size, int num_rows) {
    int grid_rank = blockIdx.x * blockDim.x + threadIdx.x;

    // One thread block spared for communication
    // Need to subtract 1 when calculating total grid size
    int grid_size = (gridDim.x - 1) * blockDim.x;

    int mype = nvshmem_my_pe();

    for (int local_row_idx = grid_rank; local_row_idx < chunk_size; local_row_idx += grid_size) {
        int global_row_idx = row_start_idx + local_row_idx;

        if (global_row_idx < num_rows) {
            int row_elem = I[global_row_idx];
            int next_row_elem = I[global_row_idx + 1];
            int num_elems_this_row = next_row_elem - row_elem;

            real output = 0.0;

            for (int j = 0; j < num_elems_this_row; j++) {
                int input_vec_elem_idx = J[row_elem + j];
                int remote_pe = input_vec_elem_idx / chunk_size;

                int remote_pe_idx_offset = input_vec_elem_idx - remote_pe * chunk_size;

                // NVSHMEM calls require explicitly specifying the type
                // For now this will only work with double

                real elem_val = nvshmem_double_g(inputVecX + remote_pe_idx_offset, remote_pe);

                output += alpha * val[row_elem + j] * elem_val;
            }

            outputVecY[local_row_idx] = output;
        }
    }
}

// Performs two dot products at the same time
// Used to perform <r, r> and <r, w> at the same time
// Can we combined the two atomicAdds somehow?
__device__ void gpuDotProductsMerged(real *vecA_delta, real *vecB_delta, real *vecA_gamma,
                                     real *vecB_gamma, double *local_dot_result_delta,
                                     double *local_dot_result_gamma, const cg::thread_block &cta,
                                     int chunk_size, const int sMemSize) {
    int grid_rank = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = gridDim.x * blockDim.x;

    // First half (up to sMemSize / 2) will be used for delta
    // Second half (from sMemSize / 2) will be used for gamma
    extern __shared__ double tmp[];

    double *tmp_delta = (double *)tmp;
    double *tmp_gamma = (double *)&tmp_delta[sMemSize / (2 * sizeof(double))];

    double temp_sum_delta = 0.0;
    double temp_sum_gamma = 0.0;

    for (int i = grid_rank; i < chunk_size; i += grid_size) {
        temp_sum_delta += (double)(vecA_delta[i] * vecB_delta[i]);
        temp_sum_gamma += (double)(vecA_gamma[i] * vecB_gamma[i]);
    }

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    temp_sum_delta = cg::reduce(tile32, temp_sum_delta, cg::plus<double>());
    temp_sum_gamma = cg::reduce(tile32, temp_sum_gamma, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
        tmp_delta[tile32.meta_group_rank()] = temp_sum_delta;
        tmp_gamma[tile32.meta_group_rank()] = temp_sum_gamma;
    }

    cg::sync(cta);

    if (tile32.meta_group_rank() == 0) {
        temp_sum_delta =
            tile32.thread_rank() < tile32.meta_group_size() ? tmp_delta[tile32.thread_rank()] : 0.0;
        temp_sum_delta = cg::reduce(tile32, temp_sum_delta, cg::plus<double>());

        temp_sum_gamma =
            tile32.thread_rank() < tile32.meta_group_size() ? tmp_gamma[tile32.thread_rank()] : 0.0;
        temp_sum_gamma = cg::reduce(tile32, temp_sum_gamma, cg::plus<double>());

        if (tile32.thread_rank() == 0) {
            atomicAdd(local_dot_result_delta, temp_sum_delta);
            atomicAdd(local_dot_result_gamma, temp_sum_gamma);
        }
    }
}

__device__ void gpuCopyVector(real *srcA, real *destB, int chunk_size) {
    int grid_rank = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = gridDim.x * blockDim.x;

    for (int i = grid_rank; i < chunk_size; i += grid_size) {
        destB[i] = srcA[i];
    }
}

__device__ void gpuSaxpy(real *x, real *y, real a, int chunk_size) {
    int grid_rank = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = gridDim.x * blockDim.x;

    for (int i = grid_rank; i < chunk_size; i += grid_size) {
        y[i] = a * x[i] + y[i];
    }
}

__device__ void gpuScaleVectorAndSaxpy(real *x, real *y, real a, real scale, int chunk_size) {
    int grid_rank = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = gridDim.x * blockDim.x;

    for (int i = grid_rank; i < chunk_size; i += grid_size) {
        y[i] = a * x[i] + scale * y[i];
    }
}

__global__ void __launch_bounds__(1024, 1)
    multiGpuConjugateGradient(int *I, int *J, real *val, real *x, real *r, real *p, real *s,
                              real *z, real *w, real *q, real *ax0, double *device_merged_dots,
                              int nnz, int num_rows, int row_start_idx, int chunk_size, real tol,
                              const int iter_max, const int sMemSize) {
    int grid_rank = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = gridDim.x * blockDim.x;

    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    real real_positive_one = 1.0;
    real real_negative_one = -1.0;

    real tmp_dot_delta0 = 0.0;

    real real_tmp_dot_delta1;
    real real_tmp_dot_gamma1;

    real beta;
    real alpha;
    real negative_alpha;

    int mype = nvshmem_my_pe();

    for (int local_row_idx = grid_rank; local_row_idx < chunk_size; local_row_idx += grid_size) {
        int global_row_idx = row_start_idx + local_row_idx;

        if (global_row_idx < num_rows) {
            r[local_row_idx] = 1.0;
            x[local_row_idx] = 0.0;
        }
    }

    if (grid.thread_rank() == 0) {
        nvshmem_barrier_all();
    }

    cg::sync(grid);

    // ax0 = AX0
    gpuSpMV(I, J, val, real_positive_one, x, ax0, row_start_idx, chunk_size, num_rows);

    if (grid.thread_rank() == 0) {
        nvshmem_barrier_all();
    }

    cg::sync(grid);

    // r0 = b0 - ax0
    // NOTE: b is a unit vector.
    gpuSaxpy(ax0, r, real_negative_one, chunk_size);

    if (grid.thread_rank() == 0) {
        nvshmem_barrier_all();
    }

    cg::sync(grid);

    // w0 = Ar0
    gpuSpMV(I, J, val, real_positive_one, r, w, row_start_idx, chunk_size, num_rows);

    if (grid.thread_rank() == 0) {
        nvshmem_barrier_all();
    }

    cg::sync(grid);

    int k = 1;

    while (k <= iter_max) {
        if (grid.thread_rank() == 0) {
            device_merged_dots[0] = 0.0;
            device_merged_dots[1] = 0.0;
        }

        cg::sync(grid);

        gpuDotProductsMerged(r, r, r, w, &device_merged_dots[0], &device_merged_dots[1], cta,
                             chunk_size, sMemSize);

        cg::sync(grid);

        // Allocate one thread block for dot global reduction (`atomicAdd`s)
        // Rest are for SpMV

        if (blockIdx.x == (gridDim.x - 1)) {
            nvshmemx_double_sum_reduce_block(NVSHMEM_TEAM_WORLD, device_merged_dots,
                                             device_merged_dots, 2);
        } else {
            gpuSpMV(I, J, val, real_positive_one, w, q, row_start_idx, chunk_size, num_rows);
        }

        if (grid.thread_rank() == 0) {
            nvshmem_barrier_all();
        }

        cg::sync(grid);

        real_tmp_dot_delta1 = (real)device_merged_dots[0];
        real_tmp_dot_gamma1 = (real)device_merged_dots[1];

        if (k > 1) {
            beta = real_tmp_dot_delta1 / tmp_dot_delta0;
            alpha =
                real_tmp_dot_delta1 / (real_tmp_dot_gamma1 - (beta / alpha) * real_tmp_dot_delta1);
        } else {
            beta = 0.0;
            alpha = real_tmp_dot_delta1 / real_tmp_dot_gamma1;
        }

        // z_k = q_k + beta_k * z_(k-1)
        gpuScaleVectorAndSaxpy(q, z, real_positive_one, beta, chunk_size);

        // s_k = w_k + beta_k * s_(k-1)
        gpuScaleVectorAndSaxpy(w, s, real_positive_one, beta, chunk_size);

        // p_k = r_k = beta_k * p_(k-1)
        gpuScaleVectorAndSaxpy(r, p, real_positive_one, beta, chunk_size);

        cg::sync(grid);

        // x_(k+1) = x_k + alpha_k * p_k
        gpuSaxpy(p, x, alpha, chunk_size);

        negative_alpha = -alpha;

        // r_(k+1) = r_k - alpha_k * s_k
        gpuSaxpy(s, r, negative_alpha, chunk_size);

        // w_(k+1) = w_k - alpha_k * z_k
        gpuSaxpy(z, w, negative_alpha, chunk_size);

        tmp_dot_delta0 = real_tmp_dot_delta1;

        if (grid.thread_rank() == 0) {
            nvshmem_barrier_all();
        }

        cg::sync(grid);

        k++;
    }
}
}  // namespace SingleStreamPipelinedNVSHMEM

int SingleStreamPipelinedNVSHMEM::init(int argc, char *argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 10000);
    std::string matrix_path_str = get_argval<std::string>(argv, argv + argc, "-matrix_path", "");
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare-single-gpu");
    const bool compare_to_cpu = get_arg(argv, argv + argc, "-compare-cpu");

    char *matrix_path_char = const_cast<char *>(matrix_path_str.c_str());
    bool generate_random_tridiag_matrix = matrix_path_str.empty();

    int num_devices = 0;
    double single_gpu_runtime;

    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    int num_rows = 0;
    int num_cols = 0;
    int nnz = 0;

    int *host_I = NULL;
    int *host_J = NULL;
    real *host_val = NULL;

    real *x_ref_single_gpu = NULL;
    real *x_final_result = NULL;

    real *s_cpu = NULL;
    real *r_cpu = NULL;
    real *p_cpu = NULL;
    real *x_ref_cpu = NULL;

    int *device_I = NULL;
    int *device_J = NULL;
    real *device_val = NULL;

    real *device_x;
    real *device_r;
    real *device_p;
    real *device_s;
    real *device_z;
    real *device_w;
    real *device_q;
    real *device_ax0;

    double *device_merged_dots;

    int rank = 0, size = 1;
    MPI_CALL(MPI_Init(&argc, &argv));
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &size));

    int local_rank = -1;
    int local_size = 1;
    {
        MPI_Comm local_comm;
        MPI_CALL(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
                                     &local_comm));

        MPI_CALL(MPI_Comm_rank(local_comm, &local_rank));
        MPI_CALL(MPI_Comm_size(local_comm, &local_size));

        MPI_CALL(MPI_Comm_free(&local_comm));
    }

    if (1 < num_devices && num_devices < local_size) {
        fprintf(stderr,
                "ERROR Number of visible devices (%d) is less than number of ranks on the "
                "node (%d)!\n",
                num_devices, local_size);
        MPI_CALL(MPI_Finalize());
        return 1;
    }
    if (1 == num_devices) {
        // Only 1 device visible, assuming GPU affinity is handled via CUDA_VISIBLE_DEVICES
        CUDA_RT_CALL(hipSetDevice(0));
    } else {
        CUDA_RT_CALL(hipSetDevice(local_rank));
    }

    CUDA_RT_CALL(hipFree(0));

    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr;

    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;

    if (generate_random_tridiag_matrix) {
        num_rows = 10485760 * 2;
        num_cols = num_rows;

        nnz = (num_rows - 2) * 3 + 4;

        host_I = (int *)malloc(sizeof(int) * (num_rows + 1));
        host_J = (int *)malloc(sizeof(int) * nnz);
        host_val = (real *)malloc(sizeof(real) * nnz);

        /* Generate a random tridiagonal symmetric matrix in CSR format */
        genTridiag(host_I, host_J, host_val, num_rows, nnz);
    } else {
        if (loadMMSparseMatrix<real>(matrix_path_char, 'd', true, &num_rows, &num_cols, &nnz,
                                     &host_val, &host_I, &host_J, true)) {
            exit(EXIT_FAILURE);
        }
    }

    CUDA_RT_CALL(hipMalloc((void **)&device_I, sizeof(int) * (num_rows + 1)));
    CUDA_RT_CALL(hipMalloc((void **)&device_J, sizeof(int) * nnz));
    CUDA_RT_CALL(hipMalloc((void **)&device_val, sizeof(real) * nnz));

    CUDA_RT_CALL(
        hipMemcpy(device_I, host_I, sizeof(int) * (num_rows + 1), hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(device_J, host_J, sizeof(int) * nnz, hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(device_val, host_val, sizeof(real) * nnz, hipMemcpyHostToDevice));

    // Set symmetric heap size for nvshmem based on problem size
    // Its default value in nvshmem is 1 GB which is not sufficient
    // for large mesh sizes
    long long unsigned int mesh_size_per_rank = num_rows / size + (num_rows % size != 0);

    long long unsigned int required_symmetric_heap_size =
        8 * mesh_size_per_rank * sizeof(real) * 1.1;

    char *value = getenv("NVSHMEM_SYMMETRIC_SIZE");
    if (value) { /* env variable is set */
        long long unsigned int size_env = parse_nvshmem_symmetric_size(value);
        if (size_env < required_symmetric_heap_size) {
            fprintf(stderr,
                    "ERROR: Minimum NVSHMEM_SYMMETRIC_SIZE = %lluB, Current "
                    "NVSHMEM_SYMMETRIC_SIZE=%s\n",
                    required_symmetric_heap_size, value);
            MPI_CALL(MPI_Finalize());
            return -1;
        }
    } else {
        char symmetric_heap_size_str[100];
        sprintf(symmetric_heap_size_str, "%llu", required_symmetric_heap_size);

        // if (rank == 0) {
        //     printf("Setting environment variable NVSHMEM_SYMMETRIC_SIZE = %llu\n",
        //            required_symmetric_heap_size);
        // }

        setenv("NVSHMEM_SYMMETRIC_SIZE", symmetric_heap_size_str, 1);
    }
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

    int npes = nvshmem_n_pes();
    int mype = nvshmem_my_pe();

    nvshmem_barrier_all();

    hipStream_t mainStream;
    CUDA_RT_CALL(hipStreamCreateWithFlags(&mainStream, hipStreamNonBlocking));

    nvshmem_barrier_all();

    // Load balancing this way isn't ideal
    // On kernel side, we need to calculate PE element belong to
    // Naive load balancing like this makes PE calculation on kernel side easier
    int chunk_size = num_rows / npes + (num_rows % npes != 0);

    device_x = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_r = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_p = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_s = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_z = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_w = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_q = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_ax0 = (real *)nvshmem_malloc(chunk_size * sizeof(real));

    CUDA_RT_CALL(hipMemset(device_x, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_r, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_p, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_s, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_z, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_w, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_q, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_ax0, 0, chunk_size * sizeof(real)));

    // device_merged_dots[0] is dot delta
    // device_merged_dots[1] is dot gamma
    device_merged_dots = (double *)nvshmem_malloc(2 * sizeof(double));

    CUDA_RT_CALL(hipMemset(device_merged_dots, 0, 2 * sizeof(double)));

    // Calculate local domain boundaries
    int row_start_global_idx = mype * chunk_size;      // My start index in the global array
    int row_end_global_idx = (mype + 1) * chunk_size;  // My end index in the global array

    row_end_global_idx = std::min(row_end_global_idx, num_rows);

    if (compare_to_single_gpu) {
        CUDA_RT_CALL(hipHostMalloc(&x_ref_single_gpu, num_rows * sizeof(real)));

        single_gpu_runtime = SingleGPUDiscreteStandard::run_single_gpu(
            iter_max, device_I, device_J, device_val, x_ref_single_gpu, num_rows, nnz);

        // single_gpu_runtime = SingleGPUDiscretePipelined::run_single_gpu(
        //     iter_max, device_I, device_J, device_val, x_ref_single_gpu, num_rows, nnz);
    }

    if (compare_to_cpu) {
        s_cpu = (real *)malloc(sizeof(real) * num_rows);
        r_cpu = (real *)malloc(sizeof(real) * num_rows);
        p_cpu = (real *)malloc(sizeof(real) * num_rows);

        CUDA_RT_CALL(hipHostMalloc(&x_ref_cpu, num_rows * sizeof(real)));

        for (int i = 0; i < num_rows; i++) {
            r_cpu[i] = 1.0;
            s_cpu[i] = 0.0;
            x_ref_cpu[i] = 0.0;
        }

        CPU::cpuConjugateGrad(iter_max, host_I, host_J, host_val, x_ref_cpu, s_cpu, p_cpu, r_cpu,
                              nnz, num_rows, tol);
    }

    CUDA_RT_CALL(hipDeviceSynchronize());
    nvshmem_barrier_all();

    // WARNING!!!
    // This was causing issues for me
    // Get rid of THREADS_PER_BLOCK
    // Use per version threadsPerBlock variable
    int threadsPerBlock = 1024;
    int sMemSize = 2 * sizeof(double) * ((threadsPerBlock / 32) + 1);

    void *kernelArgs[] = {
        (void *)&device_I,   (void *)&device_J,   (void *)&device_val,
        (void *)&device_x,   (void *)&device_r,   (void *)&device_p,
        (void *)&device_s,   (void *)&device_z,   (void *)&device_w,
        (void *)&device_q,   (void *)&device_ax0, (void *)&device_merged_dots,
        (void *)&nnz,        (void *)&num_rows,   (void *)&row_start_global_idx,
        (void *)&chunk_size, (void *)&tol,        (void *)&iter_max,
        (void *)&sMemSize,
    };

    int numBlocks = 0;

    nvshmemx_collective_launch_query_gridsize((void *)multiGpuConjugateGradient, threadsPerBlock,
                                              kernelArgs, sMemSize, &numBlocks);

    nvshmem_barrier_all();

    double start = MPI_Wtime();

    nvshmemx_collective_launch((void *)multiGpuConjugateGradient, numBlocks, threadsPerBlock,
                               kernelArgs, sMemSize, mainStream);

    nvshmemx_barrier_all_on_stream(mainStream);
    CUDA_RT_CALL(hipDeviceSynchronize());

    double stop = MPI_Wtime();

    if (compare_to_single_gpu || compare_to_cpu) {
        CUDA_RT_CALL(hipHostMalloc(&x_final_result, num_rows * sizeof(real)));

        // Need to do this when when num_rows % npes != 0
        int num_elems_to_copy = row_end_global_idx - row_start_global_idx;

        CUDA_RT_CALL(hipMemcpy(x_final_result + row_start_global_idx, device_x,
                                num_elems_to_copy * sizeof(real), hipMemcpyDeviceToHost));
    }

    bool result_correct_single_gpu = true;
    bool result_correct_cpu = true;

    report_errors(num_rows, x_ref_single_gpu, x_ref_cpu, x_final_result, row_start_global_idx,
                  row_end_global_idx, npes, single_gpu_runtime, start, stop, compare_to_single_gpu,
                  compare_to_cpu, result_correct_single_gpu, result_correct_cpu);

    nvshmem_barrier_all();

    if (mype == 0) {
        report_runtime(npes, single_gpu_runtime, start, stop, result_correct_single_gpu,
                       result_correct_cpu, compare_to_single_gpu);
    }

    nvshmem_free(device_x);
    nvshmem_free(device_r);
    nvshmem_free(device_p);
    nvshmem_free(device_s);
    nvshmem_free(device_z);
    nvshmem_free(device_w);
    nvshmem_free(device_q);
    nvshmem_free(device_ax0);

    nvshmem_free(device_merged_dots);

    CUDA_RT_CALL(hipStreamDestroy(mainStream));

    CUDA_RT_CALL(hipFree(device_I));
    CUDA_RT_CALL(hipFree(device_J));
    CUDA_RT_CALL(hipFree(device_val));

    free(host_I);
    free(host_J);
    free(host_val);

    if (compare_to_single_gpu || compare_to_cpu) {
        hipHostFree(x_final_result);

        if (compare_to_single_gpu) {
            hipHostFree(x_ref_single_gpu);
        }

        if (compare_to_cpu) {
            hipHostFree(x_ref_cpu);
        }
    }

    nvshmem_finalize();
    MPI_CALL(MPI_Finalize());

    return 0;
}
