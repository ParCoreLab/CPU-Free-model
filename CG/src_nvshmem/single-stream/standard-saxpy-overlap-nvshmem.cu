#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a conjugate gradient solver on multiple GPU using
 * Unified Memory optimized prefetching and usage hints.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <map>
#include <set>
#include <utility>

#include <omp.h>

#include "../../include_nvshmem/common.h"
#include "../../include_nvshmem/single-stream/standard-saxpy-overlap-nvshmem.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>

namespace cg = cooperative_groups;

namespace SingleStreamStandardSaxpyOverlapNVSHMEM {

__device__ void gpuSpMV(int *rowInd, int *colInd, real *val, real alpha, real *inputVecX,
                        real *outputVecY, int row_start_idx, int chunk_size, int num_rows,
                        bool matrix_is_zero_indexed, const cg::grid_group &grid) {
    int mype = nvshmem_my_pe();

    for (int local_row_idx = grid.thread_rank(); local_row_idx < chunk_size;
         local_row_idx += grid.size()) {
        int global_row_idx = row_start_idx + local_row_idx;

        if (global_row_idx < num_rows) {
            int row_elem = rowInd[global_row_idx] - int(!matrix_is_zero_indexed);
            int next_row_elem = rowInd[global_row_idx + 1] - int(!matrix_is_zero_indexed);
            int num_elems_this_row = next_row_elem - row_elem;

            real output = 0.0;

            for (int j = 0; j < num_elems_this_row; j++) {
                // If matrix is 1-indexed, need to move indices back by 1
                int input_vec_elem_idx = colInd[row_elem + j] - int(!matrix_is_zero_indexed);
                int remote_pe = input_vec_elem_idx / chunk_size;

                int remote_pe_idx_offset = input_vec_elem_idx - remote_pe * chunk_size;

                // NVSHMEM calls require explicitly specifying the type
                // For now this will only work with double

                real elem_val = nvshmem_double_g(inputVecX + remote_pe_idx_offset, remote_pe);

                output += alpha * val[row_elem + j] * elem_val;
            }

            outputVecY[local_row_idx] = output;
        }
    }
}

__device__ void gpuDotProduct(real *vecA, real *vecB, double *local_dot_result,
                              const cg::thread_block &cta, int chunk_size,
                              const cg::grid_group &grid) {
    extern __shared__ double tmp[];

    double temp_sum = 0.0;

    for (size_t i = grid.thread_rank(); i < chunk_size; i += grid.size()) {
        temp_sum += (double)(vecA[i] * vecB[i]);
    }

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
        tmp[tile32.meta_group_rank()] = temp_sum;
    }

    cg::sync(cta);

    if (tile32.meta_group_rank() == 0) {
        temp_sum =
            tile32.thread_rank() < tile32.meta_group_size() ? tmp[tile32.thread_rank()] : 0.0;
        temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

        if (tile32.thread_rank() == 0) {
            atomicAdd(local_dot_result, temp_sum);
        }
    }
}

__device__ void initVectors(real *r, real *x, int row_start_idx, int chunk_size, int num_rows,
                            const cg::grid_group &grid) {
    for (int local_row_idx = grid.thread_rank(); local_row_idx < chunk_size;
         local_row_idx += grid.size()) {
        int global_row_idx = row_start_idx + local_row_idx;

        if (global_row_idx < num_rows) {
            r[local_row_idx] = 1.0;
            x[local_row_idx] = 0.0;
        }
    }
}

__device__ void gpuCopyVector(real *srcA, real *destB, int chunk_size, const cg::grid_group &grid) {
    for (int i = grid.thread_rank(); i < chunk_size; i += grid.size()) {
        destB[i] = srcA[i];
    }
}

__device__ void gpuSaxpy(real *x, real *y, real a, int chunk_size, bool is_comm_comp_overlap_on,
                         const cg::grid_group &grid) {
    int grid_size = grid.size() - blockDim.x * int(is_comm_comp_overlap_on);

    for (int i = grid.thread_rank(); i < chunk_size; i += grid_size) {
        y[i] = a * x[i] + y[i];
    }
}

__device__ void gpuScaleVectorAndSaxpy(real *x, real *y, real a, real scale, int chunk_size,
                                       const cg::grid_group &grid) {
    for (int i = grid.thread_rank(); i < chunk_size; i += grid.size()) {
        y[i] = a * x[i] + scale * y[i];
    }
}
__global__ void __launch_bounds__(1024, 1)
    multiGpuConjugateGradient(int *device_csrRowIndices, int *device_csrColIndices,
                              real *device_csrVal, real *x, real *r, real *p, real *s, real *ax0,
                              double *dot_delta1, double *dot_gamma1, int nnz, int num_rows,
                              int row_start_idx, int chunk_size, bool matrix_is_zero_indexed,
                              real tol, const int iter_max, const int sMemSize) {
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    int last_thread_idx = grid.size() - 1;

    real real_positive_one = 1.0;
    real real_negative_one = -1.0;

    real tmp_dot_gamma0 = 0.0;

    real real_tmp_dot_gamma1;

    real beta;
    real alpha;
    real negative_alpha;

    initVectors(r, x, row_start_idx, chunk_size, num_rows, grid);

    if (grid.thread_rank() == last_thread_idx) {
        nvshmem_barrier_all();
    }

    cg::sync(grid);

    // ax0 = Ax0
    gpuSpMV(device_csrRowIndices, device_csrColIndices, device_csrVal, real_positive_one, x, ax0,
            row_start_idx, chunk_size, num_rows, matrix_is_zero_indexed, grid);

    if (grid.thread_rank() == last_thread_idx) {
        nvshmem_barrier_all();
    }

    cg::sync(grid);

    // r0 = b0 - ax0
    // NOTE: b is a unit vector.
    // NOTE: We use r here because r is also a unit vector at first (r = b0)
    gpuSaxpy(ax0, r, real_negative_one, chunk_size, false, grid);

    // Don't think we need this barrier here
    if (grid.thread_rank() == last_thread_idx) {
        nvshmem_barrier_all();
    }

    cg::sync(grid);

    // p0 = r0
    gpuCopyVector(r, p, chunk_size, grid);

    cg::sync(grid);

    // First dot - gamma = r * r
    if (grid.thread_rank() == last_thread_idx) {
        *dot_gamma1 = 0.0;
    }

    cg::sync(grid);

    gpuDotProduct(r, r, dot_gamma1, cta, chunk_size, grid);

    cg::sync(grid);

    if (grid.thread_rank() == last_thread_idx) {
        nvshmem_double_sum_reduce(NVSHMEM_TEAM_WORLD, dot_gamma1, dot_gamma1, 1);
        nvshmem_barrier_all();
    }

    cg::sync(grid);

    tmp_dot_gamma0 = (real)*dot_gamma1;

    // s0 = Ar0
    gpuSpMV(device_csrRowIndices, device_csrColIndices, device_csrVal, real_positive_one, r, s,
            row_start_idx, chunk_size, num_rows, matrix_is_zero_indexed, grid);

    if (grid.thread_rank() == last_thread_idx) {
        nvshmem_barrier_all();
    }

    cg::sync(grid);

    // Second dot - delta = p * s
    if (grid.thread_rank() == last_thread_idx) {
        *dot_delta1 = 0.0;
    }

    cg::sync(grid);

    gpuDotProduct(p, s, dot_delta1, cta, chunk_size, grid);

    cg::sync(grid);

    if (grid.thread_rank() == last_thread_idx) {
        nvshmem_double_sum_reduce(NVSHMEM_TEAM_WORLD, dot_delta1, dot_delta1, 1);
        nvshmem_barrier_all();
    }

    cg::sync(grid);

    alpha = tmp_dot_gamma0 / (real)*dot_delta1;

    int k = 1;

    while (k <= iter_max) {
        negative_alpha = -alpha;

        // r_k = r_(k-1) - alpha_(k-1) * s_(k-1)
        gpuSaxpy(s, r, negative_alpha, chunk_size, false, grid);

        if (grid.thread_rank() == last_thread_idx) {
            *dot_gamma1 = 0.0;
        }

        cg::sync(grid);

        gpuDotProduct(r, r, dot_gamma1, cta, chunk_size, grid);

        cg::sync(grid);

        if (cta.group_index().x == (grid.num_blocks() - 1)) {
            nvshmemx_double_sum_reduce_block(NVSHMEM_TEAM_WORLD, dot_gamma1, dot_gamma1, 1);
        } else {
            // x_k = x_(k-1) + alpha_(k-1) * p_(k-1)
            gpuSaxpy(p, x, alpha, chunk_size, true, grid);
        }

        if (grid.thread_rank() == last_thread_idx) {
            nvshmem_barrier_all();
        }

        cg::sync(grid);

        real_tmp_dot_gamma1 = (real)*dot_gamma1;

        beta = real_tmp_dot_gamma1 / tmp_dot_gamma0;

        gpuScaleVectorAndSaxpy(r, p, real_positive_one, beta, chunk_size, grid);

        if (grid.thread_rank() == last_thread_idx) {
            nvshmem_barrier_all();
        }

        cg::sync(grid);

        gpuSpMV(device_csrRowIndices, device_csrColIndices, device_csrVal, real_positive_one, p, s,
                row_start_idx, chunk_size, num_rows, matrix_is_zero_indexed, grid);

        if (grid.thread_rank() == last_thread_idx) {
            nvshmem_barrier_all();
        }

        cg::sync(grid);

        if (grid.thread_rank() == last_thread_idx) {
            *dot_delta1 = 0.0;
        }

        cg::sync(grid);

        gpuDotProduct(p, s, dot_delta1, cta, chunk_size, grid);

        cg::sync(grid);

        if (grid.thread_rank() == last_thread_idx) {
            nvshmem_double_sum_reduce(NVSHMEM_TEAM_WORLD, dot_delta1, dot_delta1, 1);
            nvshmem_barrier_all();
        }

        cg::sync(grid);

        alpha = real_tmp_dot_gamma1 / (real)*dot_delta1;

        if (grid.thread_rank() == last_thread_idx) {
            nvshmem_barrier_all();
        }

        cg::sync(grid);

        k++;
    }
}
}  // namespace SingleStreamStandardSaxpyOverlapNVSHMEM

int SingleStreamStandardSaxpyOverlapNVSHMEM::init(int argc, char *argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 10000);
    std::string matrix_path_str = get_argval<std::string>(argv, argv + argc, "-matrix_path", "");
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare-single-gpu");
    const bool compare_to_cpu = get_arg(argv, argv + argc, "-compare-cpu");

    char *matrix_path_char = const_cast<char *>(matrix_path_str.c_str());
    bool generate_random_tridiag_matrix = matrix_path_str.empty();

    int num_devices = 0;
    double single_gpu_runtime;

    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    int num_rows = 0;
    int num_cols = 0;
    int nnz = 0;
    bool matrix_is_zero_indexed;

    int *host_csrRowIndices = NULL;
    int *host_csrColIndices = NULL;
    real *host_csrVal = NULL;

    real *x_ref_single_gpu = NULL;
    real *x_final_result = NULL;

    real *s_cpu = NULL;
    real *r_cpu = NULL;
    real *p_cpu = NULL;
    real *x_ref_cpu = NULL;

    int *device_csrRowIndices = NULL;
    int *device_csrColIndices = NULL;
    real *device_val = NULL;

    real *device_x;
    real *device_r;
    real *device_p;
    real *device_s;
    real *device_ax0;

    double *device_dot_delta1;
    double *device_dot_gamma1;

    int rank = 0, size = 1;
    MPI_CALL(MPI_Init(&argc, &argv));
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &size));

    int local_rank = -1;
    int local_size = 1;
    {
        MPI_Comm local_comm;
        MPI_CALL(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL,
                                     &local_comm));

        MPI_CALL(MPI_Comm_rank(local_comm, &local_rank));
        MPI_CALL(MPI_Comm_size(local_comm, &local_size));

        MPI_CALL(MPI_Comm_free(&local_comm));
    }

    if (1 < num_devices && num_devices < local_size) {
        fprintf(stderr,
                "ERROR Number of visible devices (%d) is less than number of ranks on the "
                "node (%d)!\n",
                num_devices, local_size);
        MPI_CALL(MPI_Finalize());
        return 1;
    }
    if (1 == num_devices) {
        // Only 1 device visible, assuming GPU affinity is handled via CUDA_VISIBLE_DEVICES
        CUDA_RT_CALL(hipSetDevice(0));
    } else {
        CUDA_RT_CALL(hipSetDevice(local_rank));
    }

    CUDA_RT_CALL(hipFree(0));

    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr;

    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;

    if (generate_random_tridiag_matrix) {
        num_rows = 10485760 * 2;
        num_cols = num_rows;

        nnz = (num_rows - 2) * 3 + 4;

        host_csrRowIndices = (int *)malloc(sizeof(int) * (num_rows + 1));
        host_csrColIndices = (int *)malloc(sizeof(int) * nnz);
        host_csrVal = (real *)malloc(sizeof(real) * nnz);

        /* Generate a random tridiagonal symmetric matrix in CSR format */
        genTridiag(host_csrRowIndices, host_csrColIndices, host_csrVal, num_rows, nnz);
    } else {
        if (loadMMSparseMatrix<real>(matrix_path_char, 'd', true, &num_rows, &num_cols, &nnz,
                                     &host_csrVal, &host_csrRowIndices, &host_csrColIndices,
                                     true)) {
            exit(EXIT_FAILURE);
        }
    }

    // Check if matrix is 0 or 1 indexed
    int index_base = host_csrRowIndices[0];

    if (index_base == 1) {
        matrix_is_zero_indexed = false;
    } else if (index_base == 0) {
        matrix_is_zero_indexed = true;
    }

    CUDA_RT_CALL(hipMalloc((void **)&device_csrRowIndices, sizeof(int) * (num_rows + 1)));
    CUDA_RT_CALL(hipMalloc((void **)&device_csrColIndices, sizeof(int) * nnz));
    CUDA_RT_CALL(hipMalloc((void **)&device_val, sizeof(real) * nnz));

    CUDA_RT_CALL(hipMemcpy(device_csrRowIndices, host_csrRowIndices, sizeof(int) * (num_rows + 1),
                            hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(device_csrColIndices, host_csrColIndices, sizeof(int) * nnz,
                            hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(device_val, host_csrVal, sizeof(real) * nnz, hipMemcpyHostToDevice));

    // Set symmetric heap size for nvshmem based on problem size
    // Its default value in nvshmem is 1 GB which is not sufficient
    // for large mesh sizes
    long long unsigned int mesh_size_per_rank = num_rows / size + (num_rows % size != 0);

    long long unsigned int required_symmetric_heap_size =
        5 * mesh_size_per_rank * sizeof(real) * 1.1;

    char *value = getenv("NVSHMEM_SYMMETRIC_SIZE");
    if (value) { /* env variable is set */
        long long unsigned int size_env = parse_nvshmem_symmetric_size(value);
        if (size_env < required_symmetric_heap_size) {
            fprintf(stderr,
                    "ERROR: Minimum NVSHMEM_SYMMETRIC_SIZE = %lluB, Current "
                    "NVSHMEM_SYMMETRIC_SIZE=%s\n",
                    required_symmetric_heap_size, value);
            MPI_CALL(MPI_Finalize());
            return -1;
        }
    } else {
        char symmetric_heap_size_str[100];
        sprintf(symmetric_heap_size_str, "%llu", required_symmetric_heap_size);

        // if (rank == 0) {
        //     printf("Setting environment variable NVSHMEM_SYMMETRIC_SIZE = %llu\n",
        //            required_symmetric_heap_size);
        // }

        setenv("NVSHMEM_SYMMETRIC_SIZE", symmetric_heap_size_str, 1);
    }
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

    int npes = nvshmem_n_pes();
    int mype = nvshmem_my_pe();

    nvshmem_barrier_all();

    hipStream_t mainStream;
    CUDA_RT_CALL(hipStreamCreateWithFlags(&mainStream, hipStreamNonBlocking));

    nvshmem_barrier_all();

    // Load balancing this way isn't ideal
    // On kernel side, we need to calculate PE element belong to
    // Naive load balancing like this makes PE calculation on kernel side easier
    int chunk_size = num_rows / npes + (num_rows % npes != 0);

    device_x = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_r = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_p = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_s = (real *)nvshmem_malloc(chunk_size * sizeof(real));
    device_ax0 = (real *)nvshmem_malloc(chunk_size * sizeof(real));

    CUDA_RT_CALL(hipMemset(device_x, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_r, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_p, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_s, 0, chunk_size * sizeof(real)));
    CUDA_RT_CALL(hipMemset(device_ax0, 0, chunk_size * sizeof(real)));

    device_dot_delta1 = (double *)nvshmem_malloc(sizeof(double));
    device_dot_gamma1 = (double *)nvshmem_malloc(sizeof(double));

    CUDA_RT_CALL(hipMemset(device_dot_delta1, 0, sizeof(double)));
    CUDA_RT_CALL(hipMemset(device_dot_gamma1, 0, sizeof(double)));

    // Calculate local domain boundaries
    int row_start_global_idx = mype * chunk_size;      // My start index in the global array
    int row_end_global_idx = (mype + 1) * chunk_size;  // My end index in the global array

    row_end_global_idx = std::min(row_end_global_idx, num_rows);

    if (compare_to_single_gpu) {
        CUDA_RT_CALL(hipHostMalloc(&x_ref_single_gpu, num_rows * sizeof(real)));

        single_gpu_runtime = SingleGPUDiscreteStandard::run_single_gpu(
            iter_max, device_csrRowIndices, device_csrColIndices, device_val, x_ref_single_gpu,
            num_rows, nnz, matrix_is_zero_indexed);

        // single_gpu_runtime = SingleGPUDiscretePipelined::run_single_gpu(
        //     iter_max, device_csrRowIndices, device_csrColIndices, device_val, x_ref_single_gpu,
        //     num_rows, nnz, matrix_is_zero_indexed);
    }

    if (compare_to_cpu) {
        s_cpu = (real *)malloc(sizeof(real) * num_rows);
        r_cpu = (real *)malloc(sizeof(real) * num_rows);
        p_cpu = (real *)malloc(sizeof(real) * num_rows);

        CUDA_RT_CALL(hipHostMalloc(&x_ref_cpu, num_rows * sizeof(real)));

        for (int i = 0; i < num_rows; i++) {
            r_cpu[i] = 1.0;
            s_cpu[i] = 0.0;
            x_ref_cpu[i] = 0.0;
        }

        CPU::cpuConjugateGrad(iter_max, host_csrRowIndices, host_csrColIndices, host_csrVal,
                              x_ref_cpu, s_cpu, p_cpu, r_cpu, nnz, num_rows, tol);
    }

    CUDA_RT_CALL(hipDeviceSynchronize());
    nvshmem_barrier_all();

    // WARNING!!!
    // This was causing issues for me
    // Get rid of THREADS_PER_BLOCK
    // Use per version threadsPerBlock variable
    int threadsPerBlock = 1024;
    int sMemSize = sizeof(double) * ((threadsPerBlock / 32) + 1);

    void *kernelArgs[] = {
        (void *)&device_csrRowIndices,
        (void *)&device_csrColIndices,
        (void *)&device_val,
        (void *)&device_x,
        (void *)&device_r,
        (void *)&device_p,
        (void *)&device_s,
        (void *)&device_ax0,
        (void *)&device_dot_delta1,
        (void *)&device_dot_gamma1,
        (void *)&nnz,
        (void *)&num_rows,
        (void *)&row_start_global_idx,
        (void *)&chunk_size,
        (void *)&matrix_is_zero_indexed,
        (void *)&tol,
        (void *)&iter_max,
        (void *)&sMemSize,
    };

    int numBlocks = 0;

    nvshmemx_collective_launch_query_gridsize((void *)multiGpuConjugateGradient, threadsPerBlock,
                                              kernelArgs, sMemSize, &numBlocks);

    nvshmem_barrier_all();

    double start = MPI_Wtime();

    nvshmemx_collective_launch((void *)multiGpuConjugateGradient, numBlocks, threadsPerBlock,
                               kernelArgs, sMemSize, mainStream);

    nvshmemx_barrier_all_on_stream(mainStream);
    CUDA_RT_CALL(hipStreamSynchronize(mainStream));

    double stop = MPI_Wtime();

    if (compare_to_single_gpu || compare_to_cpu) {
        CUDA_RT_CALL(hipHostMalloc(&x_final_result, num_rows * sizeof(real)));

        // Need to do this when when num_rows % npes != 0
        int num_elems_to_copy = row_end_global_idx - row_start_global_idx;

        CUDA_RT_CALL(hipMemcpy(x_final_result + row_start_global_idx, device_x,
                                num_elems_to_copy * sizeof(real), hipMemcpyDeviceToHost));
    }

    bool result_correct_single_gpu = true;
    bool result_correct_cpu = true;

    report_errors(num_rows, x_ref_single_gpu, x_ref_cpu, x_final_result, row_start_global_idx,
                  row_end_global_idx, npes, single_gpu_runtime, start, stop, compare_to_single_gpu,
                  compare_to_cpu, result_correct_single_gpu, result_correct_cpu);

    nvshmem_barrier_all();

    if (mype == 0) {
        report_runtime(npes, single_gpu_runtime, start, stop, result_correct_single_gpu,
                       result_correct_cpu, compare_to_single_gpu);
    }

    nvshmem_free(device_x);
    nvshmem_free(device_r);
    nvshmem_free(device_p);
    nvshmem_free(device_s);
    nvshmem_free(device_ax0);

    nvshmem_free(device_dot_delta1);
    nvshmem_free(device_dot_gamma1);

    CUDA_RT_CALL(hipStreamDestroy(mainStream));

    CUDA_RT_CALL(hipFree(device_csrRowIndices));
    CUDA_RT_CALL(hipFree(device_csrColIndices));
    CUDA_RT_CALL(hipFree(device_val));

    free(host_csrRowIndices);
    free(host_csrColIndices);
    free(host_csrVal);

    if (compare_to_single_gpu || compare_to_cpu) {
        hipHostFree(x_final_result);

        if (compare_to_single_gpu) {
            hipHostFree(x_ref_single_gpu);
        }

        if (compare_to_cpu) {
            hipHostFree(x_ref_cpu);
        }
    }

    nvshmem_finalize();
    MPI_CALL(MPI_Finalize());

    return 0;
}
