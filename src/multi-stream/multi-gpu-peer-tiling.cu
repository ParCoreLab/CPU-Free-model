#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <cmath>
#include <cstdio>
#include <iostream>

#include <omp.h>

#include <hip/hip_cooperative_groups.h>

#include "../../include/common.h"
#include "../../include/multi-stream/multi-gpu-peer-tiling.cuh"

namespace cg = cooperative_groups;

namespace MultiGPUPeerTiling {
__global__ void __launch_bounds__(1024, 1) jacobi_kernel(
        real *a_new, real *a,
        const int iy_start, const int iy_end, const int nx,
        const int comp_tile_size_x, const int comp_tile_size_y,
        const int num_comp_tiles_x, const int num_comp_tiles_y,
        const int top_iy, const int bottom_iy,
        const int iter_max,
        volatile real *local_halo_buffer_for_top_neighbor,
        volatile real *local_halo_buffer_for_bottom_neighbor,
        volatile real *remote_my_halo_buffer_on_top_neighbor,
        volatile real *remote_my_halo_buffer_on_bottom_neighbor,
        volatile int *local_is_top_neighbor_done_writing_to_me,
        volatile int *local_is_bottom_neighbor_done_writing_to_me,
        volatile int *remote_am_done_writing_to_top_neighbor,
        volatile int *remote_am_done_writing_to_bottom_neighbor,
        volatile int *iteration_done) {

    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    int grid_dim_x = (comp_tile_size_x + blockDim.x - 1) / blockDim.x;
    int block_idx_y = blockIdx.x / grid_dim_x;
    int block_idx_x = blockIdx.x % grid_dim_x;

    int base_iy = block_idx_y * blockDim.y + threadIdx.y;
    int base_ix = block_idx_x * blockDim.x + threadIdx.x;

    int iter = 0;

    int cur_iter_mod = 0;
    int next_iter_mod = 1;
    int temp_iter_mod = 0;

    int cur_iter_comm_tile_flag_idx;
    int next_iter_comm_tile_flag_idx;

    int comm_tile_idx;
    int comp_tile_idx_x;
    int comp_tile_idx_y;

    int comm_tile_start;
    int comm_tile_end;
    int comp_tile_start_ny;
    int comp_tile_end_ny;
    int comp_tile_start_nx;
    int comp_tile_end_nx;

    int iy;
    int ix;

    while (iter < iter_max) {
        for (comp_tile_idx_y = 0; comp_tile_idx_y < num_comp_tiles_y; comp_tile_idx_y++) {
            comp_tile_start_ny =
                (comp_tile_idx_y == 0) ? iy_start + 1 : comp_tile_idx_y * comp_tile_size_y;
            comp_tile_end_ny = (comp_tile_idx_y == (num_comp_tiles_y - 1))
                                    ? iy_end - 1
                                    : (comp_tile_idx_y + 1) * comp_tile_size_y;

            for (comp_tile_idx_x = 0; comp_tile_idx_x < num_comp_tiles_x; comp_tile_idx_x++) {
                comp_tile_start_nx =
                    (comp_tile_idx_x == 0) ? 1 : comp_tile_idx_x * comp_tile_size_x;
                comp_tile_end_nx = (comp_tile_idx_x == (num_comp_tiles_x - 1))
                                        ? nx - 1
                                        : (comp_tile_idx_x + 1) * comp_tile_size_x;

                iy = base_iy + comp_tile_start_ny;
                ix = base_ix + comp_tile_start_nx;

                if (iy < comp_tile_end_ny && ix < comp_tile_end_nx) {
                    const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                                    a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
                    a_new[iy * nx + ix] = new_val;
                }
            }
        }

        real *temp_pointer_first = a_new;
        a_new = a;
        a = temp_pointer_first;

        iter++;

        temp_iter_mod = cur_iter_mod;
        cur_iter_mod = next_iter_mod;
        next_iter_mod = temp_iter_mod;

        cg::sync(grid);

        if (threadIdx.x == 0 && threadIdx.y == 0) {
            while (iteration_done[0] != iter) {
            }
            iteration_done[1] = iter;
        }

        cg::sync(grid);
    }
}

__global__ void __launch_bounds__(1024, 1) boundary_sync_kernel(
    real *a_new, real *a,
    const int iy_start, const int iy_end, const int nx,
    const int comm_tile_size, const int num_comm_tiles,
    const int top_iy, const int bottom_iy,
    const int iter_max,
    volatile real *local_halo_buffer_for_top_neighbor,
    volatile real *local_halo_buffer_for_bottom_neighbor,
    volatile real *remote_my_halo_buffer_on_top_neighbor,
    volatile real *remote_my_halo_buffer_on_bottom_neighbor,
    volatile int *local_is_top_neighbor_done_writing_to_me,
    volatile int *local_is_bottom_neighbor_done_writing_to_me,
    volatile int *remote_am_done_writing_to_top_neighbor,
    volatile int *remote_am_done_writing_to_bottom_neighbor,
    volatile int *iteration_done) {

    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    int num_flags = 2 * num_comm_tiles;

    int iter = 0;

    int cur_iter_mod = 0;
    int next_iter_mod = 1;
    int temp_iter_mod = 0;

    int cur_iter_comm_tile_flag_idx;
    int next_iter_comm_tile_flag_idx;

    int comm_tile_idx;
    int comp_tile_idx_x;
    int comp_tile_idx_y;

    int comm_tile_start;
    int comm_tile_end;

    while (iter < iter_max) {
        while (iteration_done[1] != iter) {}

        if (blockIdx.x == gridDim.x - 1) {
            for (comm_tile_idx = 0; comm_tile_idx < num_comm_tiles; comm_tile_idx++) {
                comm_tile_start = (comm_tile_idx == 0) ? 1 : comm_tile_idx * comm_tile_size;
                comm_tile_end = (comm_tile_idx == (num_comm_tiles - 1))
                                    ? nx - 1
                                    : (comm_tile_idx + 1) * comm_tile_size;

                int col = threadIdx.y * blockDim.x + threadIdx.x + comm_tile_start;

                cur_iter_comm_tile_flag_idx = comm_tile_idx + cur_iter_mod * num_flags;
                next_iter_comm_tile_flag_idx =
                    (num_comm_tiles + comm_tile_idx) + next_iter_mod * num_flags;

                if (cta.thread_rank() == 0) {
                    while (local_is_top_neighbor_done_writing_to_me[cur_iter_comm_tile_flag_idx] !=
                           iter) {
                    }
                }

                cg::sync(cta);

                if (col < comm_tile_end) {
                    const real first_row_val =
                        0.25 * (a[iy_start * nx + col + 1] + a[iy_start * nx + col - 1] +
                                a[(iy_start + 1) * nx + col] +
                                remote_my_halo_buffer_on_top_neighbor[nx * cur_iter_mod + col]);

                    a_new[iy_start * nx + col] = first_row_val;
                    local_halo_buffer_for_top_neighbor[nx * next_iter_mod + col] = first_row_val;
                }

                cg::sync(cta);

                if (cta.thread_rank() == 0) {
                    remote_am_done_writing_to_top_neighbor[next_iter_comm_tile_flag_idx] = iter + 1;
                }
            }
        } else if (blockIdx.x == gridDim.x - 2) {
            for (comm_tile_idx = 0; comm_tile_idx < num_comm_tiles; comm_tile_idx++) {
                comm_tile_start = (comm_tile_idx == 0) ? 1 : comm_tile_idx * comm_tile_size;
                comm_tile_end = (comm_tile_idx == (num_comm_tiles - 1))
                                    ? nx - 1
                                    : (comm_tile_idx + 1) * comm_tile_size;

                int col = threadIdx.y * blockDim.x + threadIdx.x + comm_tile_start;

                cur_iter_comm_tile_flag_idx =
                    (num_comm_tiles + comm_tile_idx) + cur_iter_mod * num_flags;
                next_iter_comm_tile_flag_idx = comm_tile_idx + next_iter_mod * num_flags;

                if (cta.thread_rank() == 0) {
                    while (
                        local_is_bottom_neighbor_done_writing_to_me[cur_iter_comm_tile_flag_idx] !=
                        iter) {
                    }
                }

                cg::sync(cta);

                if (col < comm_tile_end) {
                    const real last_row_val =
                        0.25 * (a[(iy_end - 1) * nx + col + 1] + a[(iy_end - 1) * nx + col - 1] +
                                remote_my_halo_buffer_on_bottom_neighbor[nx * cur_iter_mod + col] +
                                a[(iy_end - 2) * nx + col]);

                    a_new[(iy_end - 1) * nx + col] = last_row_val;
                    local_halo_buffer_for_bottom_neighbor[nx * next_iter_mod + col] = last_row_val;
                }

                cg::sync(cta);

                if (cta.thread_rank() == 0) {
                    remote_am_done_writing_to_bottom_neighbor[next_iter_comm_tile_flag_idx] =
                        iter + 1;
                }
            }
        }

        real *temp_pointer_first = a_new;
        a_new = a;
        a = temp_pointer_first;

        iter++;

        temp_iter_mod = cur_iter_mod;
        cur_iter_mod = next_iter_mod;
        next_iter_mod = temp_iter_mod;

        cg::sync(grid);

        if (threadIdx.x == 0 && threadIdx.y == 0) {
            iteration_done[0] = iter;
        }

        cg::sync(grid);
    }
}
}  // namespace MultiGPUPeerTiling

int MultiGPUPeerTiling::init(int argc, char *argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare");

    real *a[MAX_NUM_DEVICES];
    real *a_new[MAX_NUM_DEVICES];
    int iy_end[MAX_NUM_DEVICES];

    real *halo_buffer_for_top_neighbor[MAX_NUM_DEVICES];
    real *halo_buffer_for_bottom_neighbor[MAX_NUM_DEVICES];

    int *is_top_done_computing_flags[MAX_NUM_DEVICES];
    int *is_bottom_done_computing_flags[MAX_NUM_DEVICES];

    real *a_ref_h;
    real *a_h;

    double runtime_serial_non_persistent = 0.0;

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

#pragma omp parallel num_threads(num_devices)
    {
        int dev_id = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(nullptr));

        if (compare_to_single_gpu && 0 == dev_id) {
            CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * sizeof(real)));
            CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * sizeof(real)));

            runtime_serial_non_persistent = single_gpu(nx, ny, iter_max, a_ref_h, 0, true);
        }

#pragma omp barrier

        int chunk_size;
        int chunk_size_low = (ny - 2) / num_devices;
        int chunk_size_high = chunk_size_low + 1;

        int height_per_gpu = ny / num_devices;

        hipDeviceProp_t deviceProp{};
        CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, dev_id));
        int numSms = deviceProp.multiProcessorCount;

        constexpr int dim_block_x = 32;
        constexpr int dim_block_y = 32;

        int comp_tile_size_x = 256;
        int comp_tile_size_y;

        int grid_dim_x = (comp_tile_size_x + dim_block_x - 1) / dim_block_x;
        int max_thread_blocks_y = (numSms - 2) / grid_dim_x;

        comp_tile_size_y = dim_block_y * max_thread_blocks_y;

        // printf("Computation tile dimensions: %dx%d\n", comp_tile_size_x, comp_tile_size_y);

        int num_comp_tiles_x = nx / comp_tile_size_x + (nx % comp_tile_size_x != 0);
        int num_comp_tiles_y =
            height_per_gpu / comp_tile_size_y + (height_per_gpu % comp_tile_size_y != 0);

        int comm_tile_size = dim_block_x * dim_block_y;
        int num_comm_tiles = nx / comm_tile_size + (nx % comm_tile_size != 0);
        int num_flags = 4 * num_comm_tiles;

        // printf("Number of communication tiles: %d\n", num_comm_tiles);

        int num_ranks_low = num_devices * chunk_size_low + num_devices - (ny - 2);
        if (dev_id < num_ranks_low)
            chunk_size = chunk_size_low;
        else
            chunk_size = chunk_size_high;

        const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
        const int bottom = (dev_id + 1) % num_devices;

        if (top != dev_id) {
            int canAccessPeer = 0;
            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));
            if (canAccessPeer) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
            } else {
                std::cerr << "P2P access required from " << dev_id << " to " << top << std::endl;
            }
            if (top != bottom) {
                canAccessPeer = 0;
                CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, bottom));
                if (canAccessPeer) {
                    CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom, 0));
                } else {
                    std::cerr << "P2P access required from " << dev_id << " to " << bottom
                              << std::endl;
                }
            }
        }

#pragma omp barrier

        int *iteration_done_flags[2];

        CUDA_RT_CALL(hipMalloc(iteration_done_flags, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(iteration_done_flags, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMalloc(iteration_done_flags + 1, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(iteration_done_flags + 1, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMemset(iteration_done_flags[0], 0, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMemset(iteration_done_flags[1], 0, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMalloc(a + dev_id, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(a_new + dev_id, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMemset(a[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMemset(a_new[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(halo_buffer_for_top_neighbor + dev_id, 2 * nx * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(halo_buffer_for_bottom_neighbor + dev_id, 2 * nx * sizeof(real)));

        CUDA_RT_CALL(hipMemset(halo_buffer_for_top_neighbor[dev_id], 0, 2 * nx * sizeof(real)));
        CUDA_RT_CALL(hipMemset(halo_buffer_for_bottom_neighbor[dev_id], 0, 2 * nx * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(is_top_done_computing_flags + dev_id, num_flags * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(is_bottom_done_computing_flags + dev_id, num_flags * sizeof(int)));

        CUDA_RT_CALL(hipMemset(is_top_done_computing_flags[dev_id], 0, num_flags * sizeof(int)));
        CUDA_RT_CALL(
            hipMemset(is_bottom_done_computing_flags[dev_id], 0, num_flags * sizeof(int)));

        // Calculate local domain boundaries
        int iy_start_global;  // My start index in the global array
        if (dev_id < num_ranks_low) {
            iy_start_global = dev_id * chunk_size_low + 1;
        } else {
            iy_start_global =
                num_ranks_low * chunk_size_low + (dev_id - num_ranks_low) * chunk_size_high + 1;
        }
        int iy_end_global = iy_start_global + chunk_size - 1;  // My last index in the global array

        int iy_start = 1;
        iy_end[dev_id] = (iy_end_global - iy_start_global + 1) + iy_start;
        int iy_start_bottom = 0;

        // Set diriclet boundary conditions on left and right border
        initialize_boundaries<<<(ny / num_devices) / 128 + 1, 128>>>(
            a[dev_id], a_new[dev_id], PI, iy_start_global - 1, nx, chunk_size + 2, ny);
        CUDA_RT_CALL(hipGetLastError());

        CUDA_RT_CALL(hipDeviceSynchronize());

        dim3 dim_grid(numSms - 2, 1, 1);
        dim3 dim_block(dim_block_x, dim_block_y);

        void *kernelArgsInner[] = {(void *)&a_new[dev_id],
                                    (void *)&a[dev_id],
                                    (void *)&iy_start,
                                    (void *)&iy_end[dev_id],
                                    (void *)&nx,
                                    (void *)&comp_tile_size_x,
                                    (void *)&comp_tile_size_y,
                                    (void *)&num_comp_tiles_x,
                                    (void *)&num_comp_tiles_y,
                                    (void *)&iy_end[top],
                                    (void *)&iy_start_bottom,
                                    (void *)&iter_max,
                                    (void *)&halo_buffer_for_top_neighbor[dev_id],
                                    (void *)&halo_buffer_for_bottom_neighbor[dev_id],
                                    (void *)&halo_buffer_for_bottom_neighbor[top],
                                    (void *)&halo_buffer_for_top_neighbor[bottom],
                                    (void *)&is_top_done_computing_flags[dev_id],
                                    (void *)&is_bottom_done_computing_flags[dev_id],
                                    (void *)&is_bottom_done_computing_flags[top],
                                    (void *)&is_top_done_computing_flags[bottom],
                                    (void *)&iteration_done_flags[0]};

        void *kernelArgsBoundary[] = {(void *)&a_new[dev_id],
                                        (void *)&a[dev_id],
                                        (void *)&iy_start,
                                        (void *)&iy_end[dev_id],
                                        (void *)&nx,
                                        (void *)&comm_tile_size,
                                        (void *)&num_comm_tiles,
                                        (void *)&iy_end[top],
                                        (void *)&iy_start_bottom,
                                        (void *)&iter_max,
                                        (void *)&halo_buffer_for_top_neighbor[dev_id],
                                        (void *)&halo_buffer_for_bottom_neighbor[dev_id],
                                        (void *)&halo_buffer_for_bottom_neighbor[top],
                                        (void *)&halo_buffer_for_top_neighbor[bottom],
                                        (void *)&is_top_done_computing_flags[dev_id],
                                        (void *)&is_bottom_done_computing_flags[dev_id],
                                        (void *)&is_bottom_done_computing_flags[top],
                                        (void *)&is_top_done_computing_flags[bottom],
                                        (void *)&iteration_done_flags[0]};

#pragma omp barrier
        double start = omp_get_wtime();

        hipStream_t inner_domain_stream;
        hipStream_t boundary_sync_stream;

        CUDA_RT_CALL(hipStreamCreate(&inner_domain_stream));
        CUDA_RT_CALL(hipStreamCreate(&boundary_sync_stream));

        // THE KERNELS ARE SERIALIZED!
        // perhaps only on V100
        CUDA_RT_CALL(hipLaunchCooperativeKernel((void *)MultiGPUPeerTiling::jacobi_kernel,
                                                 dim_grid, dim_block, kernelArgsInner, 0,
                                                 inner_domain_stream));

        CUDA_RT_CALL(hipLaunchCooperativeKernel((void *)MultiGPUPeerTiling::boundary_sync_kernel,
                                                 2, dim_block, kernelArgsBoundary, 0,
                                                 boundary_sync_stream));

        CUDA_RT_CALL(hipDeviceSynchronize());

        // Need to swap pointers on CPU if iteration count is odd
        // Technically, we don't know the iteration number (since we'll be doing l2-norm)
        // Could write iter to CPU when kernel is done
        if (iter_max % 2 == 1) {
            std::swap(a_new[dev_id], a[dev_id]);
        }

#pragma omp barrier
        double stop = omp_get_wtime();

        if (compare_to_single_gpu) {
            CUDA_RT_CALL(
                hipMemcpy(a_h + iy_start_global * nx, a[dev_id] + nx,
                           std::min((ny - iy_start_global) * nx, chunk_size * nx) * sizeof(real),
                           hipMemcpyDeviceToHost));
        }

#pragma omp barrier

#pragma omp master
        {
            report_results(ny, nx, a_ref_h, a_h, num_devices, runtime_serial_non_persistent, start,
                           stop, compare_to_single_gpu);
        }

        CUDA_RT_CALL(hipFree(a_new[dev_id]));
        CUDA_RT_CALL(hipFree(a[dev_id]));

        if (compare_to_single_gpu && 0 == dev_id) {
            CUDA_RT_CALL(hipHostFree(a_h));
            CUDA_RT_CALL(hipHostFree(a_ref_h));
        }
    }

    return 0;
}
