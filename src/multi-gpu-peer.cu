#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <iterator>

#include <omp.h>

#include <hip/hip_cooperative_groups.h>

#include "../include/common.h"
#include "../include/single-gpu-naive.cuh"

namespace cg = cooperative_groups;

constexpr real zeroTwentyFive { 0.25 };
const real PI { static_cast<real>(2.0 * std::asin(1.0)) };

__global__ void initialize_boundaries(real* __restrict__ const a_new, real* __restrict__ const a,
                                      const real pi, const int nx, const int ny) {
    for (unsigned int iy = blockIdx.x * blockDim.x + threadIdx.x; iy < ny; iy += blockDim.x * gridDim.x) {
        const real y0 = sin(2.0 * pi * iy / (ny - 1));
        a[iy * nx + 0] = y0;
        a[iy * nx + (nx - 1)] = y0;
        a_new[iy * nx + 0] = y0;
        a_new[iy * nx + (nx - 1)] = y0;
    }
}

__global__ void jacobi_kernel(real* __restrict__ a_new, const real* __restrict__ a,
                              const int iy_start, const int iy_end, const int nx, const int niter,
                              int* flag) {

    cg::grid_group grid = cg::this_grid();

    const unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y + 1;
    const unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;

    int i = 0;

    while (i < niter) {
        if (iy < iy_end) {
            if (ix >= 1 && ix < (nx - 1)) {
                const real new_val = zeroTwentyFive * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                             a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
                a_new[iy * nx + ix] = new_val;

                // apply boundary conditions
                if (iy_start == iy) {
                    a_new[iy_end * nx + ix] = new_val;
                }

                if ((iy_end - 1) == iy) {
                    a_new[(iy_start - 1) * nx + ix] = new_val;
                }
            }
        }

        real* temp_pointer = a_new;
        a = a_new;
        a_new = temp_pointer;

        i++;
        grid.sync();
    }

    if (threadIdx.x == 0) {
        *flag = 1;
    }
}

__global__ void boundary_sync_kernel(real* __restrict__ a_new, int* flag) {
    while (!*flag) {
    }

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        printf("Sync\n");
        *flag = false;
    }
}

bool get_arg(char** begin, char** end, const std::string& arg) {
    char** itr = std::find(begin, end, arg);
    if (itr != end) {
        return true;
    }
    return false;
}

constexpr int THREADS_PER_BLOCK = 1024;

int init(int argc, char* argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nccheck = get_argval<int>(argv, argv + argc, "-nccheck", 1);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const bool csv = get_arg(argv, argv + argc, "-csv");

    if (nccheck != 1) {
        fprintf(stderr, "Only nccheck = 1 is supported\n");
        return -1;
    }

    real* a;
    real* a_new;

    int iy_start = 1;
    int iy_end = (ny - 1);

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    // This will pick the best possible CUDA capable device
    hipDeviceProp_t deviceProp{};
    int devID = 0;  // findCudaDevice(argc, (const char **)argv);
    CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, devID));
    int numSms = deviceProp.multiProcessorCount;

    int numBlocksPerSm = 0;
    int numThreads = THREADS_PER_BLOCK;

    CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, jacobi_kernel,
                                                               numThreads, 0));

    // This is stupid
    int blocks_each = (int)sqrt(numSms * numBlocksPerSm);
    int threads_each = (int)sqrt(THREADS_PER_BLOCK);
    dim3 dimGrid(blocks_each, blocks_each), dimBlock(threads_each, threads_each);

    int leastPriority = 0;
    int greatestPriority = leastPriority;
    CUDA_RT_CALL(hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority));

#pragma omp parallel num_threads(num_devices)
    {
        int dev_id = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(nullptr));

        // For debugging locally
        if (num_devices > 1) {
            int canAccessPeer = 0;
            const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);

            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));
            if (canAccessPeer) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
            } else {
                std::cerr << "P2P access required from " << dev_id << " to " << top << std::endl;
                std::exit(1);
            }
        }

        CUDA_RT_CALL(hipMalloc(&a, nx * ny * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * sizeof(real)));

        CUDA_RT_CALL(hipMemset(a, 0, nx * ny * sizeof(real)));
        CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * sizeof(real)));

        // Set diriclet boundary conditions on left and right boarder
        initialize_boundaries<<<ny / 128 + 1, 128>>>(a, a_new, PI, nx, ny);
        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipDeviceSynchronize());

        if (!csv)
            printf(
                "Jacobi relaxation: %d iterations on %d x %d mesh with norm check "
                "every %d iterations\n",
                iter_max, ny, nx, nccheck);

        PUSH_RANGE("Jacobi solve", 0)

        int* flag;
        CUDA_RT_CALL(hipMalloc(&flag, 1 * sizeof(int)));
        CUDA_RT_CALL(hipMemset(flag, 0, 1 * sizeof(int)));

        void* kernelArgs[] = {(void*)&a_new, (void*)&a,        (void*)&iy_start, (void*)&iy_end,
                              (void*)&nx,    (void*)&iter_max, (void*)&flag};

        hipStream_t inner_domain_stream;
        hipStream_t boundary_sync_stream;

        // Creating streams with priority
        CUDA_RT_CALL(hipStreamCreateWithPriority(&inner_domain_stream, hipStreamNonBlocking,
                                                  greatestPriority));
        CUDA_RT_CALL(hipStreamCreateWithPriority(&boundary_sync_stream, hipStreamNonBlocking,
                                                  leastPriority));

        CUDA_RT_CALL(hipSetDevice(dev_id));

        // Inner domain
        CUDA_RT_CALL(hipLaunchCooperativeKernel((void*)jacobi_kernel, dimGrid, dimBlock,
                                                 kernelArgs, 0, inner_domain_stream));

        // Boundary
        boundary_sync_kernel<<<1, 1, 0, boundary_sync_stream>>>(a, flag);

        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipDeviceSynchronize());
    }
}
