#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <iterator>

#include <omp.h>

#include <hip/hip_cooperative_groups.h>

#include "../include/common.h"
#include "../include/multi-gpu-peer.cuh"
#include "../include/single-gpu-naive.cuh"

namespace cg = cooperative_groups;

constexpr real ZERO_TWENTY_FIVE{0.25};

namespace MultiGPUPeer {
__global__ void jacobi_kernel(real* a_new, real* a, const int iy_start, const int iy_end,
                              const int nx, real* a_new_top, const int top_iy, real* a_new_bottom,
                              const int bottom_iy, const int iter_max, volatile int* iteration_done) {
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;

    int iter = 0;

    while (iter < iter_max * 10000) {
        if (iy > iy_start && iy < iy_end - 1 && ix < (nx - 1)) {
            const real new_val = ZERO_TWENTY_FIVE * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                                     a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
            a_new[iy * nx + ix] = new_val;
        }

        real* temp_pointer_first = a_new;
        a_new = a;
        a = temp_pointer_first;

        iter++;

        // wait until 1
        if (threadIdx.x == 0 && threadIdx.y == 0) {
            while (!*iteration_done) {}

            *iteration_done = 0;
        }

//        cg::sync(cta);

        grid.sync();
    }
}

__global__ void boundary_sync_kernel(
    real* a_new, const real* a, const int iy_start, const int iy_end, const int nx, real* a_new_top,
    const int top_iy, real* a_new_bottom, const int bottom_iy, const int iter,
    const volatile int* local_is_top_neighbor_done_writing_to_me,
    const volatile int* local_is_bottom_neighbor_done_writing_to_me,
    volatile int* remote_am_done_writing_to_top_neighbor,
    volatile int* remote_am_done_writing_to_bottom_neighbor,
    volatile int* iteration_done, const int dev_id) {
    unsigned int iy = threadIdx.y + iy_start;
    unsigned int ix = threadIdx.x + 1;
    unsigned int col = iy * blockDim.x + ix;

    printf("ok\n");

    // wait until 0
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        while (*iteration_done) {}
    }

    __syncthreads();

    if (col < nx) {
        // Wait until top GPU puts its bottom row as my top halo
        while (local_is_top_neighbor_done_writing_to_me[iter % 2] != iter) {
        }

        const real first_row_val =
            ZERO_TWENTY_FIVE * (a[iy_start * nx + col + 1] + a[iy_start * nx + col - 1] +
                                a[(iy_start + 1) * nx + col] + a[(iy_start - 1) * nx + col]);

        a_new[iy_start * nx + col] = first_row_val;

        while (local_is_bottom_neighbor_done_writing_to_me[iter % 2] != iter) {
        }

        const real last_row_val =
            ZERO_TWENTY_FIVE * (a[(iy_end - 1) * nx + col + 1] + a[(iy_end - 1) * nx + col - 1] +
                                a[(iy_end - 2) * nx + col] + a[(iy_end)*nx + col]);

        a_new[(iy_end - 1) * nx + col] = last_row_val;

        // Communication
        a_new_top[top_iy * nx + col] = first_row_val;
        a_new_bottom[bottom_iy * nx + col] = last_row_val;
    }

    __syncthreads();

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        remote_am_done_writing_to_top_neighbor[(iter + 1) % 2] = iter + 1;
        remote_am_done_writing_to_bottom_neighbor[(iter + 1) % 2] = iter + 1;

        *iteration_done = 1;
    }
}

}  // namespace MultiGPUPeer

constexpr int THREADS_PER_BLOCK = 1024;

int MultiGPUPeer::init(int argc, char** argv) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nccheck = get_argval<int>(argv, argv + argc, "-nccheck", 1);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 256);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 256);
    //    const bool csv = get_arg(argv, argv + argc, "-csv");

    if (nccheck != 1) {
        fprintf(stderr, "Only nccheck = 1 is supported\n");
        return -1;
    }

    printf(
        "Jacobi relaxation: %d iterations on %d x %d mesh with norm check "
        "every %d iterations\n",
        iter_max, ny, nx, nccheck);

    real* a[MAX_NUM_DEVICES];
    real* a_new[MAX_NUM_DEVICES];

    int iy_start = 1;
    int iy_end[MAX_NUM_DEVICES];

    int* is_top_done_computing_flags[MAX_NUM_DEVICES];
    int* is_bottom_done_computing_flags[MAX_NUM_DEVICES];

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    // This will pick the best possible CUDA capable device
    hipDeviceProp_t deviceProp{};
    int devID = 0;  // findCudaDevice(argc, (const char **)argv);
    CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, devID));
    int numSms = deviceProp.multiProcessorCount;

    int numBlocksPerSm = 0;
    int numThreads = THREADS_PER_BLOCK;

    CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSm, MultiGPUPeer::jacobi_kernel, numThreads, 0));

    // This is stupid
    int blocks_each = (int)sqrt(numSms * numBlocksPerSm);
    int threads_each = (int)sqrt(THREADS_PER_BLOCK);
    dim3 dimGrid(blocks_each, blocks_each), dimBlock(threads_each, threads_each);

    int leastPriority = 0;
    int greatestPriority = leastPriority;
    CUDA_RT_CALL(hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority));

    // Chunks for each GPU
    int chunk_size;
    int chunk_size_low = (ny - 2) / num_devices;
    int chunk_size_high = chunk_size_low + 1;

    int num_ranks_low = num_devices * chunk_size_low + num_devices - (ny - 2);

#pragma omp parallel num_threads(num_devices)
    {
        const int dev_id = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(nullptr));

        // For debugging locally
        if (num_devices > 1) {
            int canAccessPeer = 0;
            const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);

            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));
            if (canAccessPeer) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
            } else {
                std::cerr << "P2P access required from " << dev_id << " to " << top << std::endl;
                std::exit(1);
            }
        }

#pragma omp barrier
        if (dev_id < num_ranks_low) {
            chunk_size = chunk_size_low;
        } else {
            chunk_size = chunk_size_high;
        }

        const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
        const int bottom = (dev_id + 1) % num_devices;

#pragma omp barrier

        CUDA_RT_CALL(hipMalloc(a + dev_id, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(a_new + dev_id, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMemset(a[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMemset(a_new[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(is_top_done_computing_flags + dev_id, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(is_bottom_done_computing_flags + dev_id, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMemset(is_top_done_computing_flags[dev_id], 0, sizeof(int)));
        CUDA_RT_CALL(hipMemset(is_bottom_done_computing_flags[dev_id], 0, sizeof(int)));

        // Calculate local domain boundaries
        int iy_start_global;  // My start index in the global array
        if (dev_id < num_ranks_low) {
            iy_start_global = dev_id * chunk_size_low + 1;
        } else {
            iy_start_global =
                num_ranks_low * chunk_size_low + (dev_id - num_ranks_low) * chunk_size_high + 1;
        }
        int iy_end_global = iy_start_global + chunk_size - 1;  // My last index in the global array

        iy_end[dev_id] = (iy_end_global - iy_start_global + 1) + iy_start;
        int iy_start_bottom = 0;

        // Set dirichlet boundary conditions on left and right border
        initialize_boundaries<<<(ny / num_devices) / 128 + 1, 128>>>(
            a[dev_id], a_new[dev_id], PI, iy_start_global - 1, nx, (chunk_size + 2), ny);

        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipDeviceSynchronize());

        int* flag;
        CUDA_RT_CALL(hipMalloc(&flag, 1 * sizeof(int)));
        CUDA_RT_CALL(hipMemset(flag, 0, 1 * sizeof(int)));

        void* kernelArgs[] = {(void*)&a_new[dev_id],
                              (void*)&a[dev_id],
                              (void*)&iy_start,
                              (void*)&iy_end[dev_id],
                              (void*)&nx,
                              (void*)&a_new[top],
                              (void*)&iy_end[top],
                              (void*)&a_new[bottom],
                              (void*)&iy_start_bottom,
                              (void*)&iter_max,
                              (void*)&flag};

        hipStream_t inner_domain_stream;
        hipStream_t boundary_sync_stream;

        // Creating streams with priority
//        CUDA_RT_CALL(hipStreamCreateWithPriority(&inner_domain_stream, hipStreamNonBlocking,
//                                                  leastPriority));
        CUDA_RT_CALL(hipStreamCreateWithPriority(&boundary_sync_stream, hipStreamNonBlocking,
                                                  greatestPriority));

        CUDA_RT_CALL(hipSetDevice(dev_id));

#pragma omp barrier

        // Inner domain
//        CUDA_RT_CALL(hipLaunchCooperativeKernel((void*)MultiGPUPeer::jacobi_kernel, dimGrid,
//                                                 dimBlock, kernelArgs, 0, inner_domain_stream));

        for (int iter = 0; iter < iter_max; iter++) {
            std::cout << "Trying to call boundary sync kernel" << std::endl;
            // Boundary
            boundary_sync_kernel<<<1, dimBlock, 0, boundary_sync_stream>>>(
                a_new[dev_id], a[dev_id], iy_start, iy_end[dev_id], nx, a_new[top], iy_end[top],
                a_new[bottom], iy_start_bottom, iter, is_top_done_computing_flags[dev_id],
                is_bottom_done_computing_flags[dev_id], is_bottom_done_computing_flags[top],
                is_top_done_computing_flags[bottom], flag, dev_id);

            //            std::cout << dev_id << ": " << iter << std::endl;

            std::cout << "ok" << std::endl;

            CUDA_RT_CALL(hipGetLastError());
            CUDA_RT_CALL(hipStreamSynchronize(boundary_sync_stream));
        }

        //        std::cout << dev_id << std::endl;

        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipStreamSynchronize(inner_domain_stream));
    }

    return 0;
};
