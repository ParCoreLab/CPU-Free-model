#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <iterator>

#include <omp.h>

#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

#include "../include/common.h"
#include "../include/single-gpu-naive.cuh"

typedef float real;
constexpr real tol = 1.0e-8;

const real PI = 2.0 * std::asin(1.0);

__global__ void initialize_boundaries(real* __restrict__ const a_new, real* __restrict__ const a,
                                      const real pi, const int nx, const int ny) {
    for (int iy = blockIdx.x * blockDim.x + threadIdx.x; iy < ny; iy += blockDim.x * gridDim.x) {
        const real y0 = sin(2.0 * pi * iy / (ny - 1));
        a[iy * nx + 0] = y0;
        a[iy * nx + (nx - 1)] = y0;
        a_new[iy * nx + 0] = y0;
        a_new[iy * nx + (nx - 1)] = y0;
    }
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real* __restrict__ a_new, const real* __restrict__ a,
                              const int iy_start, const int iy_end, const int nx, const int niter,
                              int* flag) {

    cg::grid_group grid = cg::this_grid();

    real local_l2_norm = 0.0;

    int i = 0;

    while (i < niter) {
        if (iy < iy_end) {
            if (ix >= 1 && ix < (nx - 1)) {
                const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                             a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
                a_new[iy * nx + ix] = new_val;

                // apply boundary conditions
                if (iy_start == iy) {
                    a_new[iy_end * nx + ix] = new_val;
                }

                if ((iy_end - 1) == iy) {
                    a_new[(iy_start - 1) * nx + ix] = new_val;
                }
            }
        }

        real* temp_pointer = a_new;
        a = a_new;
        a_new = temp_pointer;

        i++;
        grid.sync();
    }

    if (threadIdx.x == 0) {
        *flag = 1;
    }
}

__global__ void boundary_sync_kernel(real* __restrict__ a_new, int* flag) {
    while (!*flag) {
    }

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        printf("Sync\n");
        *flag = false;
    }
}

bool get_arg(char** begin, char** end, const std::string& arg) {
    char** itr = std::find(begin, end, arg);
    if (itr != end) {
        return true;
    }
    return false;
}

constexpr int THREADS_PER_BLOCK = 1024;

int init(int argc, char* argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nccheck = get_argval<int>(argv, argv + argc, "-nccheck", 1);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const bool csv = get_arg(argv, argv + argc, "-csv");

    if (nccheck != 1) {
        fprintf(stderr, "Only nccheck = 1 is supported\n");
        return -1;
    }

    real* a;
    real* a_new;

//    hipStream_t compute_stream;
//    hipStream_t copy_l2_norm_stream;
//    hipStream_t reset_l2_norm_stream;
//
//    hipEvent_t compute_done;
//    hipEvent_t reset_l2_norm_done[2];

    real l2_norms[2];
    l2_norm_buf l2_norm_bufs[2];

    int iy_start = 1;
    int iy_end = (ny - 1);

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

#pragma omp parallel num_threads(num_devices)
    {
        int dev_id = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(0));

        // For debugging locally
        if (num_devices > 1) {
            int canAccessPeer = 0;
            const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);

            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));
            if (canAccessPeer) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
            } else {
                std::cerr << "P2P access required from " << dev_id << " to " << top << std::endl;
                std::exit(1);
            }
        }
    }

    CUDA_RT_CALL(hipMalloc(&a, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMalloc(&a_new, nx * ny * sizeof(real)));

    CUDA_RT_CALL(hipMemset(a, 0, nx * ny * sizeof(real)));
    CUDA_RT_CALL(hipMemset(a_new, 0, nx * ny * sizeof(real)));

    // Set diriclet boundary conditions on left and right boarder
    initialize_boundaries<<<ny / 128 + 1, 128>>>(a, a_new, PI, nx, ny);
    CUDA_RT_CALL(hipGetLastError());
    CUDA_RT_CALL(hipDeviceSynchronize());

    if (!csv)
        printf(
            "Jacobi relaxation: %d iterations on %d x %d mesh with norm check "
            "every %d iterations\n",
            iter_max, ny, nx, nccheck);

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    dim3 dim_grid((nx + dim_block_x - 1) / dim_block_x, (ny + dim_block_y - 1) / dim_block_y, 1);

    int iter = 0;
    for (int i = 0; i < 2; ++i) {
        l2_norms[i] = 0.0;
    }

    double start = omp_get_wtime();

    PUSH_RANGE("Jacobi solve", 0)

    int *flag;
    CUDA_RT_CALL(hipMalloc(&flag, 1 * sizeof(int)));
    CUDA_RT_CALL(hipMemset(flag, 0, 1 * sizeof(int)));

    bool l2_norm_greater_than_tol = true;
    void* kernelArgs[] = {
        (void*)&a_new,
        (void*)&a,
        //        (void *)&l2_norm_bufs[curr].d,
        (void*)&iy_start,
        (void*)&iy_end,
        (void*)&nx,
        (void*)&iter_max,
        (void*)&flag
    };

    // This will pick the best possible CUDA capable device
    hipDeviceProp_t deviceProp{};
    int devID = 0;  // findCudaDevice(argc, (const char **)argv);
    CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, devID));
    int numSms = deviceProp.multiProcessorCount;

    constexpr int THREADS_PER_BLOCK = 1024;

    int sMemSize = sizeof(double) * ((THREADS_PER_BLOCK / 32) + 1);
    int numBlocksPerSm = 0;
    int numThreads = THREADS_PER_BLOCK;

    CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSm, jacobi_kernel<dim_block_x, dim_block_y>, numThreads, 0));

//    numSms -= 1;

    // This is stupid
    int blocks_each = (int) sqrt(numSms * numBlocksPerSm);
    int threads_each = (int) sqrt(THREADS_PER_BLOCK);
    dim3 dimGrid(blocks_each, blocks_each), dimBlock(threads_each, threads_each);

    //   dim3 threads(2, 2);
    //   dim3 blocks(5, 5);

    int leastPriority = 0;
    int greatestPriority = leastPriority;
    CUDA_RT_CALL(hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority));

#pragma omp parallel num_threads(num_devices)
    {
        // Add stream priority
        hipStream_t inner_domain_stream;
        hipStream_t boundary_sync_stream;

        CUDA_RT_CALL(hipStreamCreateWithPriority(&inner_domain_stream, hipStreamNonBlocking, greatestPriority));
        CUDA_RT_CALL(hipStreamCreateWithPriority(&boundary_sync_stream, hipStreamNonBlocking, leastPriority));

        int dev_id = omp_get_thread_num();
        CUDA_RT_CALL(hipSetDevice(dev_id));

        // Inner domain
        CUDA_RT_CALL(hipLaunchCooperativeKernel((void*)jacobi_kernel<dim_block_x, dim_block_y>,
                                                 dimGrid, dimBlock, kernelArgs, 0, inner_domain_stream));

        // Boundary
        boundary_sync_kernel<<<1, 1, 0, boundary_sync_stream>>>(a, flag);

        CUDA_RT_CALL(hipGetLastError());

        hipDeviceSynchronize();
    }
}
