#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <iostream>
#include <iterator>

#include <omp.h>

#include <hip/hip_cooperative_groups.h>

#include "../include/common.h"
#include "../include/multi-gpu-peer.cuh"

namespace cg = cooperative_groups;

constexpr real ZERO_TWENTY_FIVE{0.25};

namespace MultiGPUPeer {
    __global__ void __launch_bounds__(1024, 1) jacobi_kernel(real *a_new, real *a, const int iy_start, const int iy_end,
                                const int nx, real *a_new_top, real *a_top, const int top_iy,
                                real *a_new_bottom, real *a_bottom, const int bottom_iy,
                                const int iter_max, volatile int *iteration_done) {
        cg::thread_block cta = cg::this_thread_block();
        cg::grid_group grid = cg::this_grid();

        unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
        unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;

        int iter = 0;

        while (iter < iter_max) {
            if (iy > iy_start && iy < iy_end - 1 && ix < (nx - 1)) {
                const real new_val = ZERO_TWENTY_FIVE * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                                         a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
                a_new[iy * nx + ix] = new_val;
            }

            real *temp_pointer_first = a_new;
            a_new = a;
            a = temp_pointer_first;

            real *temp_pointer_second = a_new_top;
            a_new_top = a_top;
            a_top = temp_pointer_second;

            real *temp_pointer_third = a_new_bottom;
            a_new_bottom = a_bottom;
            a_bottom = temp_pointer_third;

            iter++;

            if (threadIdx.x == 0 && threadIdx.y == 0) {
                while (iteration_done[0] != iter) {
                }
                iteration_done[1] = iter;
            }

            cg::sync(grid);
        }
    }
}

__global__ void __launch_bounds__(1024, 1) boundary_sync_kernel(
    real* a_new, real* a, const int iy_start, const int iy_end,
    const int nx, real* a_new_top, real *a_top, const int top_iy,
    real *a_new_bottom, real *a_bottom, const int bottom_iy,
    const int iter,
    const volatile int* local_is_top_neighbor_done_writing_to_me,
    const volatile int* local_is_bottom_neighbor_done_writing_to_me,
    volatile int* remote_am_done_writing_to_top_neighbor,
    volatile int* remote_am_done_writing_to_bottom_neighbor,
    volatile int* iteration_done, const int dev_id) {
    unsigned int col = threadIdx.y * blockDim.x + threadIdx.x + 1;

    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        while (iteration_done[1] != iter) {
        }
    }

    if (col < nx - 1) {
        // Wait until top GPU puts its bottom row as my top halo
        while (local_is_top_neighbor_done_writing_to_me[iter % 2] != iter) {
        }

        const real first_row_val =
            ZERO_TWENTY_FIVE * (a[iy_start * nx + col + 1] + a[iy_start * nx + col - 1] +
                                a[(iy_start + 1) * nx + col] + a[(iy_start - 1) * nx + col]);

        a_new[iy_start * nx + col] = first_row_val;

        while (local_is_bottom_neighbor_done_writing_to_me[iter % 2] != iter) {
        }

        const real last_row_val =
            ZERO_TWENTY_FIVE * (a[(iy_end - 1) * nx + col + 1] + a[(iy_end - 1) * nx + col - 1] +
                                a[(iy_end - 2) * nx + col] + a[(iy_end)*nx + col]);

        a_new[(iy_end - 1) * nx + col] = last_row_val;

        // Communication
        a_new_top[top_iy * nx + col] = first_row_val;
        a_new_bottom[bottom_iy * nx + col] = last_row_val;
    }

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        remote_am_done_writing_to_top_neighbor[(iter + 1) % 2] = iter + 1;
        remote_am_done_writing_to_bottom_neighbor[(iter + 1) % 2] = iter + 1;

        iteration_done[0] = iter + 1;
    }
}

constexpr int THREADS_PER_BLOCK = 1024;

int MultiGPUPeer::init(int argc, char **argv) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nccheck = get_argval<int>(argv, argv + argc, "-nccheck", 1);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 256);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 256);
    //    const bool csv = get_arg(argv, argv + argc, "-csv");

    if (nccheck != 1) {
        fprintf(stderr, "Only nccheck = 1 is supported\n");
        return -1;
    }

    printf(
            "Jacobi relaxation: %d iterations on %d x %d mesh with norm check "
            "every %d iterations\n",
            iter_max, ny, nx, nccheck);

    real *a[MAX_NUM_DEVICES];
    real *a_new[MAX_NUM_DEVICES];

    int iy_start = 1;
    int iy_end[MAX_NUM_DEVICES];

    int *is_top_done_computing_flags[MAX_NUM_DEVICES];
    int *is_bottom_done_computing_flags[MAX_NUM_DEVICES];

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    // This will pick the best possible CUDA capable device
    hipDeviceProp_t deviceProp{};
    int devID = 0;  // findCudaDevice(argc, (const char **)argv);
    CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, devID));
    int numSms = deviceProp.multiProcessorCount;

    int numBlocksPerSm = 0;
    int numThreads = THREADS_PER_BLOCK;

    CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &numBlocksPerSm, MultiGPUPeer::jacobi_kernel, numThreads, 0));

    // This is stupid
    int blocks_each = (int) sqrt(numSms * numBlocksPerSm);
    int threads_each = (int) sqrt(THREADS_PER_BLOCK);
    dim3 dimGrid(blocks_each, blocks_each), dimBlock(threads_each, threads_each);

    int leastPriority = 0;
    int greatestPriority = leastPriority;
    CUDA_RT_CALL(hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority));

    // Chunks for each GPU
    int chunk_size;
    int chunk_size_low = (ny - 2) / num_devices;
    int chunk_size_high = chunk_size_low + 1;

    int num_ranks_low = num_devices * chunk_size_low + num_devices - (ny - 2);

#pragma omp parallel num_threads(num_devices)
    {
        const int dev_id = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(nullptr));

        // For debugging locally
        if (num_devices > 1) {
            int canAccessPeer = 0;
            const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);

            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));
            if (canAccessPeer) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
            } else {
                std::cerr << "P2P access required from " << dev_id << " to " << top << std::endl;
                std::exit(1);
            }
        }

#pragma omp barrier
        if (dev_id < num_ranks_low) {
            chunk_size = chunk_size_low;
        } else {
            chunk_size = chunk_size_high;
        }

        const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
        const int bottom = (dev_id + 1) % num_devices;

#pragma omp barrier

        int *flag[2];
        //    CUDA_RT_CALL(hipMalloc(&flag[0], 1 * sizeof(int)));
        //    CUDA_RT_CALL(hipMalloc(&flag[1], 1 * sizeof(int)))

        //    CUDA_RT_CALL(hipMemset(&flag[0], 0, 1 * sizeof(int)));
        //    CUDA_RT_CALL(hipMemset(&flag[1], 0, 1 * sizeof(int)));

        CUDA_RT_CALL(hipMalloc(flag, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(flag, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMalloc(flag + 1, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(flag + 1, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMemset(flag[0], 0, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMemset(flag[1], 0, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMalloc(a + dev_id, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(a_new + dev_id, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMemset(a[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMemset(a_new[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(is_top_done_computing_flags + dev_id, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(is_bottom_done_computing_flags + dev_id, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMemset(is_top_done_computing_flags[dev_id], 0, sizeof(int)));
        CUDA_RT_CALL(hipMemset(is_bottom_done_computing_flags[dev_id], 0, sizeof(int)));

        // Calculate local domain boundaries
        int iy_start_global;  // My start index in the global array
        if (dev_id < num_ranks_low) {
            iy_start_global = dev_id * chunk_size_low + 1;
        } else {
            iy_start_global =
                num_ranks_low * chunk_size_low + (dev_id - num_ranks_low) * chunk_size_high + 1;
        }
        int iy_end_global = iy_start_global + chunk_size - 1;  // My last index in the global array

        iy_end[dev_id] = (iy_end_global - iy_start_global + 1) + iy_start;
        int iy_start_bottom = 0;

        // Set dirichlet boundary conditions on left and right border
        initialize_boundaries<<<(ny / num_devices) / 128 + 1, 128>>>(
            a[dev_id], a_new[dev_id], PI, iy_start_global - 1, nx, (chunk_size + 2), ny);

        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipDeviceSynchronize());

        void *kernelArgs[] = {(void *)&a_new[dev_id],
                              (void *)&a[dev_id],
                              (void *)&iy_start,
                              (void *)&iy_end[dev_id],
                              (void *)&nx,
                              (void *)&a_new[top],
                              (void *)&a[top],
                              (void *)&iy_end[top],
                              (void *)&a_new[bottom],
                              (void *)&a[bottom],
                              (void *)&iy_start_bottom,
                              (void *)&iter_max,
                              (void *)&flag[0]};

//        void *kernelArgsBoundary[] = {(void *)&a_new[dev_id],
//                                      (void *)&a[dev_id],
//                                      (void *)&iy_start,
//                                      (void *)&iy_end[dev_id],
//                                      (void *)&nx,
//                                      (void *)&a_new[top],
//                                      (void *)&a[top],
//                                      (void *)&iy_end[top],
//                                      (void *)&a_new[bottom],
//                                      (void *)&a[bottom],
//                                      (void *)&iy_start_bottom,
//                                      (void *)&iter_max,
//                                      (void *)&is_top_done_computing_flags[dev_id],
//                                      (void *)&is_bottom_done_computing_flags[dev_id],
//                                      (void *)&is_bottom_done_computing_flags[top],
//                                      (void *)&is_top_done_computing_flags[bottom],
//                                      (void *)&flag[0],
//                                      (void *)&dev_id
//        };

        hipStream_t inner_domain_stream;
        hipStream_t boundary_sync_stream;

        // Creating streams with priority
        //        CUDA_RT_CALL(hipStreamCreateWithPriority(&inner_domain_stream, hipStreamNonBlocking,
        //                                                  leastPriority));
        //        CUDA_RT_CALL(hipStreamCreateWithPriority(&boundary_sync_stream, hipStreamNonBlocking,
        //                                                  greatestPriority));

        CUDA_RT_CALL(hipStreamCreate(&inner_domain_stream));
        CUDA_RT_CALL(hipStreamCreate(&boundary_sync_stream));

        CUDA_RT_CALL(hipSetDevice(dev_id));

#pragma omp barrier
        // Inner domain
        CUDA_RT_CALL(hipLaunchCooperativeKernel((void *)MultiGPUPeer::jacobi_kernel, dimGrid,
                                                 dimBlock, kernelArgs, 0, inner_domain_stream));

        auto& a_ref = a[dev_id];
        auto& a_new_ref = a_new[dev_id];
        auto& a_top = a[top];
        auto& a_new_top = a_new[top];
        auto& a_bottom = a[bottom];
        auto& a_new_bottom = a_new[bottom];

        for (int iter = 0; iter < iter_max; iter++) {
            // Boundary
            boundary_sync_kernel<<<1, dimBlock, 0, boundary_sync_stream>>>(
                a_new_ref, a_ref, iy_start, iy_end[dev_id], nx, a_new_top, a_top, iy_end[top],
                a_new_bottom, a_bottom, iy_start_bottom, iter, is_top_done_computing_flags[dev_id],
                is_bottom_done_computing_flags[dev_id], is_bottom_done_computing_flags[top],
                is_top_done_computing_flags[bottom], flag[0], dev_id);

            std::swap(a_ref, a_new_ref);
            std::swap(a_top, a_new_top);
            std::swap(a_bottom, a_new_bottom);
        }

        std::cout << "OK" << std::endl;

        CUDA_RT_CALL(hipStreamSynchronize(inner_domain_stream));
    }

    return 0;
};
