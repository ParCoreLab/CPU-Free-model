#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <cmath>
#include <cstdio>
#include <iostream>

#include <hip/hip_cooperative_groups.h>

#include "../../include/common.h"
#include "../../include/single-stream/single-threaded.cuh"

namespace cg = cooperative_groups;

constexpr int MAX_NUM_DEVICES = 32;
typedef float real;

const real PI = 2.0 * std::asin(1.0);

namespace SSSingleThreaded {
    __global__ void initialize_boundaries(real* __restrict__ const a_new, real* __restrict__ const a,
                                          const real pi, const int offset, const int nx,
                                          const int my_ny, const int ny) {
        for (int iy = blockIdx.x * blockDim.x + threadIdx.x; iy < my_ny; iy += blockDim.x * gridDim.x) {
            const real y0 = sin(2.0 * pi * (offset + iy) / (ny - 1));
            a[iy * nx + 0] = y0;
            a[iy * nx + (nx - 1)] = y0;
            a_new[iy * nx + 0] = y0;
            a_new[iy * nx + (nx - 1)] = y0;
        }
    }
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real* __restrict__ a_new, const real* __restrict__ a,
                              const int iy_start, const int iy_end, const int nx,
                              real* __restrict__ const a_new_top, const int top_iy,
                              real* __restrict__ const a_new_bottom, const int bottom_iy,
                              const int iter_max, int* is_top_neigbor_done,
                              int* is_bottom_neigbor_done, int* notify_top_neighbor,
                              int* notify_bottom_neighbor) {
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;

    real local_l2_norm = 0.0;
    int i = 0;

    while (i < iter_max) {
        //    One thread block does communication (and a bit of computation)
        if (blockIdx.x == gridDim.x - 1 && blockIdx.y == gridDim.y - 1) {
            int iy = threadIdx.y + iy_start;
            int ix = threadIdx.x + 1;
            int col = iy * blockDim.x + ix;

            if (col < nx) {
                // Wait until top GPU puts its bottom row as my top halo
                while (!*is_top_neigbor_done) {
                }

                const real first_row_val =
                    0.25 * (a[iy_start * nx + col + 1] + a[iy_start * nx + col - 1] +
                            a[(iy_start + 1) * nx + col] + a[(iy_start - 1) * nx + col]);
                a_new_top[top_iy * nx + col] = first_row_val;

                // Wait until bottom GPU puts its top row as my bottom halo

                while (!*is_bottom_neigbor_done) {
                }

                const real last_row_val =
                    0.25 * (a[(iy_end - 1) * nx + col + 1] + a[(iy_end - 1) * nx + col - 1] +
                            a[(iy_end - 2) * nx + col] + a[(iy_end)*nx + col]);
                a_new_bottom[bottom_iy * nx + col] = last_row_val;
            }

            cta.sync();

            if (threadIdx.x == 0 && threadIdx.y == 0) {
                *notify_bottom_neighbor = 1;
                *notify_top_neighbor = 1;
            }
        } else if (iy > iy_start && iy < iy_end - 1 && ix < (nx - 1)) {
            const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                         a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
            a_new[iy * nx + ix] = new_val;

            real residue = new_val - a[iy * nx + ix];
            local_l2_norm = residue * residue;
        }

        real* temp_pointer = a_new;
        a = a_new;
        a_new = temp_pointer;

        i++;

        if (threadIdx.x == 0 && threadIdx.y == 0) {
            *notify_top_neighbor = 0;
            *notify_bottom_neighbor = 0;
        }

        grid.sync();
    }
}

int SSSingleThreaded::init(int argc, char* argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);

    real* a_new[MAX_NUM_DEVICES];
    real* a[MAX_NUM_DEVICES];

    int iy_start[MAX_NUM_DEVICES];
    int iy_end[MAX_NUM_DEVICES];

    int chunk_size[MAX_NUM_DEVICES];

    int* is_top_done_computing_flags[MAX_NUM_DEVICES];
    int* is_bottom_done_computing_flags[MAX_NUM_DEVICES];

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));

    for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(0));

        int chunk_size_low = (ny - 2) / num_devices;
        int chunk_size_high = chunk_size_low + 1;

        int num_ranks_low = num_devices * chunk_size_low + num_devices - (ny - 2);
        if (dev_id < num_ranks_low)
            chunk_size[dev_id] = chunk_size_low;
        else
            chunk_size[dev_id] = chunk_size_high;

        const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
        const int bottom = (dev_id + 1) % num_devices;

        if (top != dev_id) {
            int canAccessPeer = 0;
            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));
            if (canAccessPeer) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
            } else {
                std::cerr << "P2P access required from " << dev_id << " to " << top << std::endl;
            }
            if (top != bottom) {
                canAccessPeer = 0;
                CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, bottom));
                if (canAccessPeer) {
                    CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom, 0));
                } else {
                    std::cerr << "P2P access required from " << dev_id << " to " << bottom
                              << std::endl;
                }
            }
        }

        CUDA_RT_CALL(hipMalloc(a + dev_id, nx * (chunk_size[dev_id] + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(a_new + dev_id, nx * (chunk_size[dev_id] + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMemset(a[dev_id], 0, nx * (chunk_size[dev_id] + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMemset(a_new[dev_id], 0, nx * (chunk_size[dev_id] + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(is_top_done_computing_flags + dev_id, 1 * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(is_bottom_done_computing_flags + dev_id, 1 * sizeof(int)));

        CUDA_RT_CALL(hipMemset(is_top_done_computing_flags[dev_id], 0, 1 * sizeof(int)));
        CUDA_RT_CALL(hipMemset(is_bottom_done_computing_flags[dev_id], 0, 1 * sizeof(int)));

        // Calculate local domain boundaries
        int iy_start_global;  // My start index in the global array
        if (dev_id < num_ranks_low) {
            iy_start_global = dev_id * chunk_size_low + 1;
        } else {
            iy_start_global =
                num_ranks_low * chunk_size_low + (dev_id - num_ranks_low) * chunk_size_high + 1;
        }

        iy_start[dev_id] = 1;
        iy_end[dev_id] = iy_start[dev_id] + chunk_size[dev_id];

        // Set diriclet boundary conditions on left and right border
        SSSingleThreaded::initialize_boundaries<<<(ny / num_devices) / 128 + 1, 128>>>(
            a[dev_id], a_new[dev_id], PI, iy_start_global - 1, nx, (chunk_size[dev_id] + 2), ny);
        CUDA_RT_CALL(hipGetLastError());

        CUDA_RT_CALL(hipDeviceSynchronize());

        constexpr int dim_block_x = 16;
        constexpr int dim_block_y = 16;

        int* notify_top = dev_id > 0 ? is_bottom_done_computing_flags[dev_id - 1]
                                     : is_bottom_done_computing_flags[num_devices - 1];
        int* notify_bottom = dev_id < num_devices - 1 ? is_top_done_computing_flags[dev_id + 1]
                                                      : is_top_done_computing_flags[0];

        void* kernelArgs[] = {
            (void*)&a_new[dev_id],
            (void*)&a[dev_id],
            (void*)&iy_start,
            (void*)&iy_end[dev_id],
            (void*)&nx,
            (void*)&a_new[top],
            (void*)&iy_end[top],
            (void*)&a_new[bottom],
//            (void*)&iy_start_bottom,
            (void*)&iter_max,
            (void*)&is_top_done_computing_flags[dev_id],
            (void*)&is_bottom_done_computing_flags[dev_id],
            (void*)&notify_top,
            (void*)&notify_bottom,
        };

        hipDeviceProp_t deviceProp{};
        CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, dev_id));
        int numSms = deviceProp.multiProcessorCount;

        constexpr int THREADS_PER_BLOCK = 256;

        int sMemSize = sizeof(double) * ((THREADS_PER_BLOCK / 32) + 1);
        int numBlocksPerSm = 0;
        int numThreads = THREADS_PER_BLOCK;

        CUDA_RT_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &numBlocksPerSm, jacobi_kernel<dim_block_x, dim_block_y>, numThreads, 0));

        int blocks_each = (int)sqrt(numSms * numBlocksPerSm);
        int threads_each = (int)sqrt(THREADS_PER_BLOCK);
        dim3 dimGrid(blocks_each, blocks_each), dimBlock(threads_each, threads_each);

        CUDA_RT_CALL(hipLaunchCooperativeKernel((void*)jacobi_kernel<dim_block_x, dim_block_y>,
                                                 dimGrid, dimBlock, kernelArgs, 0, nullptr));
    }

    for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
        CUDA_RT_CALL(hipGetLastError());

        CUDA_RT_CALL(hipDeviceSynchronize());
    }
}