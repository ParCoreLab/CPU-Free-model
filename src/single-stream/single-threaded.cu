#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <cmath>
#include <cstdio>
#include <iostream>

#include <omp.h>

#include <hip/hip_cooperative_groups.h>

#include "../../include/common.h"
#include "../../include/single-stream/single-threaded.cuh"

namespace cg = cooperative_groups;

namespace SSSingleThreaded {
__global__ void jacobi_kernel(real* a_new, real* a, const int iy_start, const int iy_end,
                              const int nx, real* a_new_top, real* a_top, const int top_iy,
                              real* a_new_bottom, real* a_bottom, const int bottom_iy,
                              const int iter_max,
                              volatile int* local_is_top_neighbor_done_writing_to_me,
                              volatile int* local_is_bottom_neighbor_done_writing_to_me,
                              volatile int* remote_am_done_writing_to_top_neighbor,
                              volatile int* remote_am_done_writing_to_bottom_neighbor) {
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

    unsigned int grid_dim_x = (nx + blockDim.x - 1) / blockDim.x;
    unsigned int block_idx_y = blockIdx.x / grid_dim_x;
    unsigned int block_idx_x = blockIdx.x % grid_dim_x;

    unsigned int iy = block_idx_y * blockDim.y + threadIdx.y + iy_start;
    unsigned int ix = block_idx_x * blockDim.x + threadIdx.x + 1;

    //    real local_l2_norm = 0.0;
    int iter = 0;

    int cur_iter_mod = 0;
    int next_iter_mod = 1;
    int temp_iter_mod = 0;

    while (iter < iter_max) {
        //    One thread block does communication (and a bit of computation)
        if (blockIdx.x == gridDim.x - 1 && blockIdx.y == gridDim.y - 1) {
            unsigned int col = threadIdx.y * blockDim.x + threadIdx.x + 1;

            if (col < nx - 1) {
                // Wait until top GPU puts its bottom row as my top halo
                while (local_is_top_neighbor_done_writing_to_me[cur_iter_mod] != iter) {
                }

                const real first_row_val =
                    0.25 * (a[iy_start * nx + col + 1] + a[iy_start * nx + col - 1] +
                            a[(iy_start + 1) * nx + col] + a[(iy_start - 1) * nx + col]);
                a_new[iy_start * nx + col] = first_row_val;

                while (local_is_bottom_neighbor_done_writing_to_me[cur_iter_mod] != iter) {
                }

                const real last_row_val =
                    0.25 * (a[(iy_end - 1) * nx + col + 1] + a[(iy_end - 1) * nx + col - 1] +
                            a[(iy_end - 2) * nx + col] + a[(iy_end)*nx + col]);
                a_new[(iy_end - 1) * nx + col] = last_row_val;

                //                if (calculate_norm) {
                //                    real first_row_residue = first_row_val - a[iy_start * nx +
                //                    col]; real last_row_residue = last_row_val - a[iy_end * nx +
                //                    col];
                //
                //                    local_l2_norm += first_row_residue * first_row_residue;
                //                    local_l2_norm += last_row_residue * last_row_residue;
                //                }

                // Communication
                a_new_top[top_iy * nx + col] = first_row_val;
                a_new_bottom[bottom_iy * nx + col] = last_row_val;
            }

            cg::sync(cta);

            if (threadIdx.x == 0 && threadIdx.y == 0) {
                remote_am_done_writing_to_top_neighbor[next_iter_mod] = iter + 1;
                remote_am_done_writing_to_bottom_neighbor[next_iter_mod] = iter + 1;
            }
        } else if (iy > iy_start && iy < (iy_end - 1) && ix < (nx - 1)) {
            const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                         a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
            a_new[iy * nx + ix] = new_val;

            //            if (calculate_norm) {
            //                real residue = new_val - a[iy * nx + ix];
            //                local_l2_norm += residue * residue;
            //            }
        }

        real* temp_pointer_first = a_new;
        a_new = a;
        a = temp_pointer_first;

        real* temp_pointer_second = a_new_top;
        a_new_top = a_top;
        a_top = temp_pointer_second;

        real* temp_pointer_third = a_new_bottom;
        a_new_bottom = a_bottom;
        a_bottom = temp_pointer_third;

        iter++;

        temp_iter_mod = cur_iter_mod;
        cur_iter_mod = next_iter_mod;
        next_iter_mod = temp_iter_mod;

        cg::sync(grid);
    }
}
}  // namespace SSSingleThreaded

int SSSingleThreaded::init(int argc, char* argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);

    real* a[MAX_NUM_DEVICES];
    real* a_new[MAX_NUM_DEVICES];

    int iy_start[MAX_NUM_DEVICES];
    int iy_end[MAX_NUM_DEVICES];
    int chunk_size[MAX_NUM_DEVICES];

    int iy_start_bottom = 0;

    real* a_ref_h;
    real* a_h;
    double runtime_serial = 0.0;

    int* is_top_done_computing_flags[MAX_NUM_DEVICES];
    int* is_bottom_done_computing_flags[MAX_NUM_DEVICES];

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));
    //    real l2_norm = 1.0;

    for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(0));

        if (0 == dev_id) {
            CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * sizeof(real)));
            CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * sizeof(real)));
            runtime_serial = single_gpu(nx, ny, iter_max, a_ref_h, 0, true);
        }

        // ny - 2 rows are distributed amongst `size` ranks in such a way
        // that each rank gets either (ny - 2) / size or (ny - 2) / size + 1 rows.
        // This optimizes load balancing when (ny - 2) % size != 0
        int chunk_size_low = (ny - 2) / num_devices;
        int chunk_size_high = chunk_size_low + 1;
        // To calculate the number of ranks that need to compute an extra row,
        // the following formula is derived from this equation:
        // num_ranks_low * chunk_size_low + (size - num_ranks_low) * (chunk_size_low + 1) = ny - 2
        int num_ranks_low = num_devices * chunk_size_low + num_devices -
                            (ny - 2);  // Number of ranks with chunk_size = chunk_size_low
        if (dev_id < num_ranks_low)
            chunk_size[dev_id] = chunk_size_low;
        else
            chunk_size[dev_id] = chunk_size_high;

        CUDA_RT_CALL(hipMalloc(a + dev_id, nx * (chunk_size[dev_id] + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(a_new + dev_id, nx * (chunk_size[dev_id] + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMemset(a[dev_id], 0, nx * (chunk_size[dev_id] + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMemset(a_new[dev_id], 0, nx * (chunk_size[dev_id] + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(is_top_done_computing_flags + dev_id, 2 * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(is_bottom_done_computing_flags + dev_id, 2 * sizeof(int)));

        CUDA_RT_CALL(hipMemset(is_top_done_computing_flags[dev_id], 0, sizeof(int)));
        CUDA_RT_CALL(hipMemset(is_bottom_done_computing_flags[dev_id], 0, sizeof(int)));

        // Calculate local domain boundaries
        int iy_start_global;  // My start index in the global array
        if (dev_id < num_ranks_low) {
            iy_start_global = dev_id * chunk_size_low + 1;
        } else {
            iy_start_global =
                num_ranks_low * chunk_size_low + (dev_id - num_ranks_low) * chunk_size_high + 1;
        }

        iy_start[dev_id] = 1;
        iy_end[dev_id] = iy_start[dev_id] + chunk_size[dev_id];

        // Set diriclet boundary conditions on left and right boarder
        initialize_boundaries<<<(ny / num_devices) / 128 + 1, 128>>>(
            a[dev_id], a_new[dev_id], PI, iy_start_global - 1, nx, (chunk_size[dev_id] + 2), ny);
        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipDeviceSynchronize());

        const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
        int canAccessPeer = 0;
        CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));
        if (canAccessPeer) {
            CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
        }
        const int bottom = (dev_id + 1) % num_devices;
        if (top != bottom) {
            canAccessPeer = 0;
            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, bottom));
            if (canAccessPeer) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom, 0));
            }
        }

        CUDA_RT_CALL(hipDeviceSynchronize());
    }

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    constexpr int num_threads = 1024;

    hipDeviceProp_t deviceProp{};
    CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, dev_id));
    int numSms = deviceProp.multiProcessorCount;

    dim3 dim_grid(numSms, 1, 1);
    dim3 dim_block(dim_block_x, dim_block_y);

    for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipDeviceSynchronize());
    }

    double start = omp_get_wtime();

    for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
        const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
        const int bottom = (dev_id + 1) % num_devices;
        CUDA_RT_CALL(hipSetDevice(dev_id));



        void* kernelArgs[] = {(void*)&a_new[dev_id],
                              (void*)&a[dev_id],
                              (void*)&iy_start,
                              (void*)&iy_end[dev_id],
                              (void*)&nx,
                              (void*)&a_new[top],
                              (void*)&a[top],
                              (void*)&iy_end[top],
                              (void*)&a_new[bottom],
                              (void*)&a[bottom],
                              (void*)&iy_start_bottom,
                              (void*)&iter_max,
                              (void*)&is_top_done_computing_flags[dev_id],
                              (void*)&is_bottom_done_computing_flags[dev_id],
                              (void*)&is_bottom_done_computing_flags[top],
                              (void*)&is_top_done_computing_flags[bottom]};

        CUDA_RT_CALL(hipLaunchCooperativeKernel((void*)SSSingleThreaded::jacobi_kernel, dim_grid,
                                                 dim_block, kernelArgs, 0, nullptr));
    }

    for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipDeviceSynchronize());
    }

    double stop = omp_get_wtime();

    int offset = nx;
    for (int dev_id = 0; dev_id < num_devices; ++dev_id) {
        CUDA_RT_CALL(
            hipMemcpy(a_h + offset, a[dev_id] + nx,
                       std::min((nx * ny) - offset, nx * chunk_size[dev_id]) * sizeof(real),
                       hipMemcpyDeviceToHost));
        offset += std::min(chunk_size[dev_id] * nx, (nx * ny) - offset);
    }

    bool result_correct = true;
    for (int iy = 1; result_correct && (iy < (ny - 1)); ++iy) {
        for (int ix = 1; result_correct && (ix < (nx - 1)); ++ix) {
            if (std::fabs(a_ref_h[iy * nx + ix] - a_h[iy * nx + ix]) > tol) {
                fprintf(stderr,
                        "ERROR: a[%d * %d + %d] = %f does not match %f "
                        "(reference)\n",
                        iy, nx, ix, a_h[iy * nx + ix], a_ref_h[iy * nx + ix]);
                result_correct = false;
            }
        }
    }

    if (result_correct) {
        printf("Num GPUs: %d.\n", num_devices);
        printf(
            "%dx%d: 1 GPU: %8.4f s, %d GPUs: %8.4f s, speedup: %8.2f, "
            "efficiency: %8.2f \n",
            ny, nx, runtime_serial, num_devices, (stop - start), runtime_serial / (stop - start),
            runtime_serial / (num_devices * (stop - start)) * 100);
    }
}