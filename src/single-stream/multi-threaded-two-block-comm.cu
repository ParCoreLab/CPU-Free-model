#include "hip/hip_runtime.h"
/* Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 */
#include <cmath>
#include <cstdio>
#include <iostream>

#include <omp.h>

#include <hip/hip_cooperative_groups.h>

#include "../../include/common.h"
#include "../../include/single-stream/multi-threaded-two-block-comm.cuh"

namespace cg = cooperative_groups;

namespace SSMultiThreadedTwoBlockComm {
    __global__ void __launch_bounds__(1024, 1)
    jacobi_kernel(real *a_new, real *a, const int iy_start, const int iy_end, const int nx,
                  const int tile_size, const int num_tiles_x, const int num_tiles_y,
                  real *a_new_top, real *a_top, const int top_iy, real *a_new_bottom,
                  real *a_bottom, const int bottom_iy, const int iter_max,
                  volatile int *local_is_top_neighbor_done_writing_to_me,
                  volatile int *local_is_bottom_neighbor_done_writing_to_me,
                  volatile int *remote_am_done_writing_to_top_neighbor,
                  volatile int *remote_am_done_writing_to_bottom_neighbor) {
        cg::thread_block cta = cg::this_thread_block();
        cg::grid_group grid = cg::this_grid();

        unsigned int grid_dim_x = (tile_size + blockDim.x - 1) / blockDim.x;
        unsigned int block_idx_y = blockIdx.x / grid_dim_x;
        unsigned int block_idx_x = blockIdx.x % grid_dim_x;

        unsigned int base_iy = block_idx_y * blockDim.y + threadIdx.y + iy_start;
        unsigned int base_ix = block_idx_x * blockDim.x + threadIdx.x + 1;

        int num_flags = 2 * num_tiles_x;

        int iter = 0;

        int cur_iter_mod = 0;
        int next_iter_mod = 1;
        int temp_iter_mod = 0;

        int tile_start_ny = 0;
        int tile_end_ny = 0;
        int tile_start_nx = 0;
        int tile_end_nx = 0;

        int cur_iter_tile_flag_idx;
        int next_iter_tile_flag_idx;

        while (iter < iter_max) {
            for (int tile_idx_y = 0; tile_idx_y < num_tiles_y; tile_idx_y++) {
                unsigned int iy = base_iy + tile_idx_y * tile_size - iy_start * (tile_idx_y != 0);

                tile_start_ny = tile_idx_y * tile_size + iy_start * (tile_idx_y == 0);
                tile_end_ny =
                        (tile_idx_y == num_tiles_y - 1) ? iy_end - 1 : (tile_idx_y + 1) * tile_size;

                for (int tile_idx_x = 0; tile_idx_x < num_tiles_x; tile_idx_x++) {
                    unsigned int ix = base_ix + tile_idx_x * tile_size - (tile_idx_x != 0);

                    tile_start_nx = tile_idx_x * tile_size + (tile_idx_x == 0);
                    tile_end_nx = (tile_idx_x + 1) * tile_size - (tile_idx_x == num_tiles_x - 1);

                    tile_end_nx = min(tile_end_nx, nx - 1);

                    //    One thread block does communication (and a bit of computation)
                    if (blockIdx.x == gridDim.x - 1) {
                        if (tile_idx_y == 0) {
                            unsigned int col = threadIdx.y * blockDim.x + threadIdx.x + tile_start_nx;

                            cur_iter_tile_flag_idx = tile_idx_x + cur_iter_mod * num_flags;
                            next_iter_tile_flag_idx =
                                    (num_tiles_x + tile_idx_x) % num_tiles_x + next_iter_mod * num_flags;

                            if (col <= (tile_end_nx - 1)) {
                                // Wait until top GPU puts its bottom row as my top halo
                                while (
                                        local_is_top_neighbor_done_writing_to_me[cur_iter_tile_flag_idx] !=
                                        iter) {
                                }

                                const real first_row_val =
                                        0.25 *
                                        (a[iy_start * nx + col + 1] + a[iy_start * nx + col - 1] +
                                         a[(iy_start + 1) * nx + col] + a[(iy_start - 1) * nx + col]);
                                a_new[iy_start * nx + col] = first_row_val;

                                // Communication
                                a_new_top[top_iy * nx + col] = first_row_val;
                            }

                            cg::sync(cta);

                            if (threadIdx.x == 0 && threadIdx.y == 0) {
                                remote_am_done_writing_to_top_neighbor[next_iter_tile_flag_idx] =
                                        iter + 1;
                            }
                        }
                    } else if (blockIdx.x == gridDim.x - 2) {
                        if (tile_idx_y == num_tiles_y - 1) {
                            unsigned int col = threadIdx.y * blockDim.x + threadIdx.x + tile_start_nx;

                            cur_iter_tile_flag_idx =
                                    (num_tiles_x + tile_idx_x) % num_tiles_x + cur_iter_mod * num_flags;
                            next_iter_tile_flag_idx = tile_idx_x + next_iter_mod * num_flags;

                            if (col <= (tile_end_nx - 1)) {
                                while (local_is_bottom_neighbor_done_writing_to_me
                                       [cur_iter_tile_flag_idx] != iter) {
                                }

                                const real last_row_val =
                                        0.25 *
                                        (a[(iy_end - 1) * nx + col + 1] + a[(iy_end - 1) * nx + col - 1] +
                                         a[(iy_end - 2) * nx + col] + a[iy_end * nx + col]);
                                a_new[(iy_end - 1) * nx + col] = last_row_val;

                                // Communication
                                a_new_bottom[bottom_iy * nx + col] = last_row_val;
                            }

                            cg::sync(cta);

                            if (threadIdx.x == 0 && threadIdx.y == 0) {
                                remote_am_done_writing_to_bottom_neighbor[next_iter_tile_flag_idx] =
                                        iter + 1;
                            }
                        }
                    } else if (iy >= tile_start_ny && iy <= (tile_end_ny - 1) && ix >= tile_start_nx &&
                               ix <= (tile_end_nx - 1)) {
                        const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                                     a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
                        a_new[iy * nx + ix] = new_val;
                    }
                }
            }

            real *temp_pointer_first = a_new;
            a_new = a;
            a = temp_pointer_first;

            real *temp_pointer_second = a_new_top;
            a_new_top = a_top;
            a_top = temp_pointer_second;

            real *temp_pointer_third = a_new_bottom;
            a_new_bottom = a_bottom;
            a_bottom = temp_pointer_third;

            iter++;

            temp_iter_mod = cur_iter_mod;
            cur_iter_mod = next_iter_mod;
            next_iter_mod = temp_iter_mod;

            cg::sync(grid);
        }
    }
}  // namespace SSMultiThreadedTwoBlockComm

int SSMultiThreadedTwoBlockComm::init(int argc, char *argv[]) {
    const int iter_max = get_argval<int>(argv, argv + argc, "-niter", 1000);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", 16384);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", 16384);
    const bool compare_to_single_gpu = get_arg(argv, argv + argc, "-compare");

    real *a[MAX_NUM_DEVICES];
    real *a_new[MAX_NUM_DEVICES];
    int iy_end[MAX_NUM_DEVICES];

    real *a_ref_h;
    real *a_h;

    double runtime_serial_non_persistent = 0.0;

    int *is_top_done_computing_flags[MAX_NUM_DEVICES];
    int *is_bottom_done_computing_flags[MAX_NUM_DEVICES];

    int num_devices = 0;
    CUDA_RT_CALL(hipGetDeviceCount(&num_devices));
    //    real l2_norm = 1.0;

#pragma omp parallel num_threads(num_devices)
    {
        //        real* l2_norm_d;
        //        real* l2_norm_h;

        int dev_id = omp_get_thread_num();

        CUDA_RT_CALL(hipSetDevice(dev_id));
        CUDA_RT_CALL(hipFree(nullptr));

        if (compare_to_single_gpu && 0 == dev_id) {
            CUDA_RT_CALL(hipHostMalloc(&a_ref_h, nx * ny * sizeof(real)));
            CUDA_RT_CALL(hipHostMalloc(&a_h, nx * ny * sizeof(real)));

            // Passing 0 for nccheck for now
            runtime_serial_non_persistent = single_gpu(nx, ny, iter_max, a_ref_h, 0, true);
        }

#pragma omp barrier

        int chunk_size;
        int chunk_size_low = (ny - 2) / num_devices;
        int chunk_size_high = chunk_size_low + 1;

        // For now, I am assuming the height is exactly divisible by the number of devices
        int height_per_gpu = ny / num_devices;

        // A tile will be TILE_SIZE in both dimensions
        int num_tiles_x = nx / TILE_SIZE + (nx % TILE_SIZE != 0);
        int num_tiles_y = height_per_gpu / TILE_SIZE + (height_per_gpu % TILE_SIZE != 0);
        int num_flags = 4 * num_tiles_x;

        int num_ranks_low = num_devices * chunk_size_low + num_devices - (ny - 2);
        if (dev_id < num_ranks_low)
            chunk_size = chunk_size_low;
        else
            chunk_size = chunk_size_high;

        const int top = dev_id > 0 ? dev_id - 1 : (num_devices - 1);
        const int bottom = (dev_id + 1) % num_devices;

        if (top != dev_id) {
            int canAccessPeer = 0;
            CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, top));
            if (canAccessPeer) {
                CUDA_RT_CALL(hipDeviceEnablePeerAccess(top, 0));
            } else {
                std::cerr << "P2P access required from " << dev_id << " to " << top << std::endl;
            }
            if (top != bottom) {
                canAccessPeer = 0;
                CUDA_RT_CALL(hipDeviceCanAccessPeer(&canAccessPeer, dev_id, bottom));
                if (canAccessPeer) {
                    CUDA_RT_CALL(hipDeviceEnablePeerAccess(bottom, 0));
                } else {
                    std::cerr << "P2P access required from " << dev_id << " to " << bottom
                              << std::endl;
                }
            }
        }

#pragma omp barrier

        CUDA_RT_CALL(hipMalloc(a + dev_id, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMalloc(a_new + dev_id, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMemset(a[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));
        CUDA_RT_CALL(hipMemset(a_new[dev_id], 0, nx * (chunk_size + 2) * sizeof(real)));

        CUDA_RT_CALL(hipMalloc(is_top_done_computing_flags + dev_id, num_flags * sizeof(int)));
        CUDA_RT_CALL(hipMalloc(is_bottom_done_computing_flags + dev_id, num_flags * sizeof(int)));

        CUDA_RT_CALL(hipMemset(is_top_done_computing_flags[dev_id], 0, num_flags * sizeof(int)));
        CUDA_RT_CALL(
                hipMemset(is_bottom_done_computing_flags[dev_id], 0, num_flags * sizeof(int)));

        // Calculate local domain boundaries
        int iy_start_global;  // My start index in the global array
        if (dev_id < num_ranks_low) {
            iy_start_global = dev_id * chunk_size_low + 1;
        } else {
            iy_start_global =
                    num_ranks_low * chunk_size_low + (dev_id - num_ranks_low) * chunk_size_high + 1;
        }
        int iy_end_global = iy_start_global + chunk_size - 1;  // My last index in the global array

        int iy_start = 1;
        iy_end[dev_id] = (iy_end_global - iy_start_global + 1) + iy_start;
        int iy_start_bottom = 0;

        // Set diriclet boundary conditions on left and right border
        initialize_boundaries<<<(ny / num_devices) / 128 + 1, 128>>>(
                a[dev_id], a_new[dev_id], PI, iy_start_global - 1, nx, chunk_size + 2, ny);
        CUDA_RT_CALL(hipGetLastError());

        CUDA_RT_CALL(hipDeviceSynchronize());

        constexpr int dim_block_x = 32;
        constexpr int dim_block_y = 32;

        hipDeviceProp_t deviceProp{};
        CUDA_RT_CALL(hipGetDeviceProperties(&deviceProp, dev_id));
        int numSms = deviceProp.multiProcessorCount;

        dim3 dim_grid(numSms, 1, 1);
        dim3 dim_block(dim_block_x, dim_block_y);

        void *kernelArgs[] = {(void *) &a_new[dev_id],
                              (void *) &a[dev_id],
                              (void *) &iy_start,
                              (void *) &iy_end[dev_id],
                              (void *) &nx,
                              (void *) &TILE_SIZE,
                              (void *) &num_tiles_x,
                              (void *) &num_tiles_y,
                              (void *) &a_new[top],
                              (void *) &a[top],
                              (void *) &iy_end[top],
                              (void *) &a_new[bottom],
                              (void *) &a[bottom],
                              (void *) &iy_start_bottom,
                              (void *) &iter_max,
                              (void *) &is_top_done_computing_flags[dev_id],
                              (void *) &is_bottom_done_computing_flags[dev_id],
                              (void *) &is_bottom_done_computing_flags[top],
                              (void *) &is_top_done_computing_flags[bottom]};

#pragma omp barrier
        double start = omp_get_wtime();

        CUDA_RT_CALL(hipLaunchCooperativeKernel((void *) SSMultiThreadedTwoBlockComm::jacobi_kernel,
                                                 dim_grid, dim_block, kernelArgs, 0, nullptr));

        CUDA_RT_CALL(hipGetLastError());
        CUDA_RT_CALL(hipDeviceSynchronize());

#pragma omp barrier
        double stop = omp_get_wtime();

        if (compare_to_single_gpu) {
            CUDA_RT_CALL(
                    hipMemcpy(a_h + iy_start_global * nx, a[dev_id] + nx,
                               std::min((ny - iy_start_global) * nx, chunk_size * nx) * sizeof(real),
                               hipMemcpyDeviceToHost));
        }

#pragma omp barrier

#pragma omp master
        {
            report_results(ny, nx, a_ref_h, a_h, num_devices, runtime_serial_non_persistent, start,
                           stop, compare_to_single_gpu);
        }

        CUDA_RT_CALL(hipFree(a_new[dev_id]));
        CUDA_RT_CALL(hipFree(a[dev_id]));

        if (compare_to_single_gpu && 0 == dev_id) {
            CUDA_RT_CALL(hipHostFree(a_h));
            CUDA_RT_CALL(hipHostFree(a_ref_h));
        }
    }

    return 0;
}